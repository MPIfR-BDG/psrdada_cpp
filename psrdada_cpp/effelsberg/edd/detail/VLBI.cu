#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/VLBI.cuh"
#include "psrdada_cpp/effelsberg/edd/Packer.cuh"
#include "psrdada_cpp/effelsberg/edd/Tools.cuh"

#include "psrdada_cpp/cuda_utils.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/extrema.h>

#include <cstring>
#include <iostream>
#include <sstream>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {


template <class HandlerType>
VLBI<HandlerType>::VLBI(std::size_t buffer_bytes, std::size_t input_bitDepth,
                        std::size_t speadHeapSize, double sampleRate,
                        double digitizer_threshold,
                        const VDIFHeader &vdifHeader, HandlerType &handler)
    : _buffer_bytes(buffer_bytes), _input_bitDepth(input_bitDepth),
      _sampleRate(sampleRate), _digitizer_threshold(digitizer_threshold),
      _vdifHeader(vdifHeader), _output_bitDepth(2),
      _speadHeapSize(speadHeapSize), _handler(handler), _call_count(0) {

  // Sanity checks
  // check for any device errors
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  BOOST_LOG_TRIVIAL(info) << "Creating new VLBI instance";
  BOOST_LOG_TRIVIAL(info) << "   Output data in VDIF format with "
                          << vlbiHeaderSize << "bytes header info and "
                          << _vdifHeader.getDataFrameLength()
                          << " bytes payload";
  BOOST_LOG_TRIVIAL(debug) << "   Expecting speadheaps of size "
                           << speadHeapSize << "   byte";

  BOOST_LOG_TRIVIAL(debug) << "   Sample rate " << _sampleRate << " Hz";

  std::size_t n64bit_words = _buffer_bytes / sizeof(uint64_t);
  BOOST_LOG_TRIVIAL(debug) << "Allocating memory";
  _raw_voltage_db.resize(n64bit_words);
  BOOST_LOG_TRIVIAL(debug) << "   Input voltages size : "
                           << _raw_voltage_db.size() << " 64-bit words,"
                           << _raw_voltage_db.size() * 64 / 8 << " bytes";

  _packed_voltage.resize(n64bit_words * 64 / input_bitDepth * _output_bitDepth /
                         8);

  _spillOver.reserve(vdifHeader.getDataFrameLength() * 8);
  BOOST_LOG_TRIVIAL(debug) << "   Output voltages size: "
                           << _packed_voltage.size() << " byte";

  CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));

  _unpacker.reset(new Unpacker(_proc_stream));

  _vdifHeader.setBitsPerSample(_output_bitDepth);
  _vdifHeader.setNumberOfChannels(1);
  _vdifHeader.setRealDataType();
} // constructor


template <class HandlerType> VLBI<HandlerType>::~VLBI() {
  BOOST_LOG_TRIVIAL(debug) << "Destroying VLBI";
  hipStreamDestroy(_h2d_stream);
  hipStreamDestroy(_proc_stream);
  hipStreamDestroy(_d2h_stream);
}


template <class HandlerType> void VLBI<HandlerType>::init(RawBytes &block) {
  BOOST_LOG_TRIVIAL(debug) << "VLBI init called";
  std::stringstream headerInfo;
  headerInfo << "\n"
             << "# VLBI parameters: \n";

  size_t bEnd = std::strlen(block.ptr());
  if (bEnd + headerInfo.str().size() < block.total_bytes()) {
    std::strcpy(block.ptr() + bEnd, headerInfo.str().c_str());
  } else {
    BOOST_LOG_TRIVIAL(warning) << "Header of size " << block.total_bytes()
                               << " bytes already contains " << bEnd
                               << "bytes. Cannot add VLBI info of size "
                               << headerInfo.str().size() << " bytes.";
  }

  _baseLineN.resize(array_sum_Nthreads);
  _stdDevN.resize(array_sum_Nthreads);

  _handler.init(block);
}


template <class HandlerType>
bool VLBI<HandlerType>::operator()(RawBytes &block) {
  ++_call_count;
  BOOST_LOG_TRIVIAL(debug) << "VLBI operator() called (count = " << _call_count
                           << ")";
  if (block.used_bytes() != _buffer_bytes) { /* Unexpected buffer size */
    BOOST_LOG_TRIVIAL(error) << "Unexpected Buffer Size - Got "
                             << block.used_bytes() << " byte, expected "
                             << _buffer_bytes << " byte)";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    hipProfilerStop();
    return true;
  }
  ////////////////////////////////////////////////////////////////////////
  // Copy data to device
  CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
  _raw_voltage_db.swap();

  BOOST_LOG_TRIVIAL(debug) << "   block.used_bytes() = " << block.used_bytes()
                           << ", dataBlockBytes = " << _buffer_bytes << "\n";

  CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void *>(_raw_voltage_db.a_ptr()),
                                   static_cast<void *>(block.ptr()),
                                   _buffer_bytes, hipMemcpyHostToDevice,
                                   _h2d_stream));
  if (_call_count == 1) {
    return false;
  }
  ////////////////////////////////////////////////////////////////////////
  // Process data
  _packed_voltage.swap();

  BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
  switch (_input_bitDepth) {
  case 8:
    _unpacker->unpack<8>(_raw_voltage_db.b(), _unpacked_voltage);
    break;
  case 12:
    _unpacker->unpack<12>(_raw_voltage_db.b(), _unpacked_voltage);
    break;
  default:
    throw std::runtime_error("Unsupported number of bits");
  }


  BOOST_LOG_TRIVIAL(debug) << "Calculate baseline";
  psrdada_cpp::effelsberg::edd::
      array_sum<<<64, array_sum_Nthreads, 0, _proc_stream>>>(
          thrust::raw_pointer_cast(_unpacked_voltage.data()),
          _unpacked_voltage.size(),
          thrust::raw_pointer_cast(_baseLineN.data()));
  psrdada_cpp::effelsberg::edd::
      array_sum<<<1, array_sum_Nthreads, 0, _proc_stream>>>(
          thrust::raw_pointer_cast(_baseLineN.data()), _baseLineN.size(),
          thrust::raw_pointer_cast(_baseLineN.data()));

  BOOST_LOG_TRIVIAL(debug) << "Calculate std-dev";
  psrdada_cpp::effelsberg::edd::
      scaled_square_residual_sum<<<64, array_sum_Nthreads, 0, _proc_stream>>>(
          thrust::raw_pointer_cast(_unpacked_voltage.data()),
          _unpacked_voltage.size(), thrust::raw_pointer_cast(_baseLineN.data()),
          thrust::raw_pointer_cast(_stdDevN.data()));
  psrdada_cpp::effelsberg::edd::
      array_sum<<<1, array_sum_Nthreads, 0, _proc_stream>>>(
          thrust::raw_pointer_cast(_stdDevN.data()), _stdDevN.size(),
          thrust::raw_pointer_cast(_stdDevN.data()));


  // non linear packing
  BOOST_LOG_TRIVIAL(debug) << "Packing data with non linear 2-bit packaging "
                              "using levels -v*sigma, 0, v*sigma with v = "
                           << _digitizer_threshold;
  _packed_voltage.b().resize(_unpacked_voltage.size() * 2 / 8);
  BOOST_LOG_TRIVIAL(debug) << "Input size: " << _unpacked_voltage.size()
                           << " elements";
  BOOST_LOG_TRIVIAL(debug) << "Resizing output buffer to "
                           << _packed_voltage.b().size() << " byte";

  pack2bit_nonLinear<<<128, 1024, 0, _proc_stream>>>(
      thrust::raw_pointer_cast(_unpacked_voltage.data()),
      (uint32_t *)thrust::raw_pointer_cast(_packed_voltage.b().data()),
      _unpacked_voltage.size(), _digitizer_threshold,
      thrust::raw_pointer_cast(_stdDevN.data()),
      thrust::raw_pointer_cast(_baseLineN.data()));

  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  BOOST_LOG_TRIVIAL(trace) << " Standard Deviation squared: " << _stdDevN[0]
                           << " "
                           << "Mean Value: "
                           << _baseLineN[0] / _unpacked_voltage.size();

  if ((_call_count == 2)) {
    return false;
  }
  _outputBuffer.swap();

  ////////////////////////////////////////////////////////////////////////
  BOOST_LOG_TRIVIAL(debug) << "Copy Data back to host";
  CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));

  const size_t outputBlockSize = _vdifHeader.getDataFrameLength();

  size_t remainingBytes = outputBlockSize - _spillOver.size();
  size_t numberOfBlocksInOutput =
      (_packed_voltage.size() - remainingBytes) / outputBlockSize;
  BOOST_LOG_TRIVIAL(debug) << "   Number of blocks in output "
                           << numberOfBlocksInOutput;

  _outputBuffer.a().resize((1 + numberOfBlocksInOutput) *
                           (outputBlockSize + vlbiHeaderSize));

  BOOST_LOG_TRIVIAL(debug) << "   Copying " << _spillOver.size()
                           << " bytes spill over";
  // leave room for header and fill first block of output with spill over
  std::copy(_spillOver.begin(), _spillOver.end(),
            _outputBuffer.a().begin() + vlbiHeaderSize);

  BOOST_LOG_TRIVIAL(debug) << "   Copying remaining " << remainingBytes
                           << " bytes for first block";
  // cuda memcopy remainder of first block
  CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void *>(_outputBuffer.a_ptr() + vlbiHeaderSize + _spillOver.size()),
                                   static_cast<void *>(_packed_voltage.a_ptr()),
                                   remainingBytes, hipMemcpyDeviceToHost,
                                   _d2h_stream));

  const size_t dpitch = outputBlockSize + vlbiHeaderSize;
  const size_t spitch = outputBlockSize;
  const size_t width = outputBlockSize;
  size_t height = numberOfBlocksInOutput;

  BOOST_LOG_TRIVIAL(debug) << "   Copying " << height
                           << " blocks a " << outputBlockSize << " bytes";
  // we now have a full first block, pitch copy rest leaving room for the header
  CUDA_ERROR_CHECK(hipMemcpy2DAsync(
      (void *)(_outputBuffer.a_ptr() + outputBlockSize + 2 * vlbiHeaderSize),
      dpitch, (void *)thrust::raw_pointer_cast(_packed_voltage.a_ptr() +
                                               remainingBytes),
      spitch, width, height, hipMemcpyDeviceToHost, _d2h_stream));


  // new spill over
  _spillOver.resize(_packed_voltage.size() - remainingBytes -
                    numberOfBlocksInOutput * outputBlockSize);

  size_t offset = numberOfBlocksInOutput * outputBlockSize + remainingBytes;
  BOOST_LOG_TRIVIAL(debug) << " New spill over size " << _spillOver.size()
                           << " bytes with offset " << offset;

  CUDA_ERROR_CHECK(hipMemcpyAsync(
      static_cast<void *>(thrust::raw_pointer_cast(_spillOver.data())),
      static_cast<void *>(_packed_voltage.a_ptr() + offset),
      _spillOver.size(), hipMemcpyDeviceToHost, _d2h_stream));

  // fill in header data
  const size_t samplesPerDataFrame = outputBlockSize * 8 / _output_bitDepth;
  const size_t dataFramesPerSecond = _sampleRate / samplesPerDataFrame;

  for (size_t i = 0; i < numberOfBlocksInOutput + 1; i++)
  {
    // copy header to correct position
      std::copy(reinterpret_cast<uint8_t *>(_vdifHeader.getData()),
        reinterpret_cast<uint8_t *>(_vdifHeader.getData()) + vlbiHeaderSize,
        _outputBuffer.a().begin() + i * (outputBlockSize + vlbiHeaderSize));
    // update header
    size_t dataFrame = _vdifHeader.getDataFrameNumber();
    if (dataFrame < dataFramesPerSecond)
    {
      _vdifHeader.setDataFrameNumber(dataFrame + 1);
    }
    else
    {
      _vdifHeader.setDataFrameNumber(0);
    _vdifHeader.setSecondsFromReferenceEpoch(_vdifHeader.getSecondsFromReferenceEpoch() + 1);
    }
  }

  if (_call_count == 3) {
    return false;
  }

  // Wrap in a RawBytes object here;
  RawBytes bytes(reinterpret_cast<char *>(_outputBuffer.b_ptr()),
                 _outputBuffer.b().size(), _outputBuffer.b().size());
  BOOST_LOG_TRIVIAL(debug) << "Calling handler, processing "
                           << _outputBuffer.b().size() << " bytes";
  // The handler can't do anything asynchronously without a copy here
  // as it would be unsafe (given that it does not own the memory it
  // is being passed).

  _handler(bytes);
  return false; //
} // operator ()

} // edd
} // effelsberg
} // psrdada_cpp

