#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"
#include "psrdada_cpp/effelsberg/edd/Tools.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/execution_policy.h>

#include <iostream>
#include <iomanip>
#include <cstring>
#include <sstream>
#include <typeinfo>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

// Reduce thread local vatiable v in shared array x, so that x[0] contains sum
template<typename T>
__device__ void sum_reduce(T *x, const T &v)
{
  x[threadIdx.x] = v;
  __syncthreads();
  for(int s = blockDim.x / 2; s > 0; s = s / 2)
  {
    if (threadIdx.x < s)
      x[threadIdx.x] += x[threadIdx.x + s];
    __syncthreads();
  }
}


// If one of the side channel items is lost, then both are considered as lost
// here
__global__ void mergeSideChannels(uint64_t* __restrict__ A, uint64_t*
        __restrict__ B, size_t N);


__global__ void gating(float* __restrict__ G0,
        float* __restrict__ G1,
        const uint64_t* __restrict__ sideChannelData,
        size_t N, size_t heapSize, size_t bitpos,
        size_t noOfSideChannels, size_t selectedSideChannel,
        const float*  __restrict__ _baseLineG0,
        const float*  __restrict__ _baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1);


// Updates the baselines of the gates for the polarization set for the next
// block
// only few output blocks per input block thus execution on only one thread.
// Important is that the execution is async on the GPU.
__global__ void update_baselines(float*  __restrict__ baseLineG0,
        float*  __restrict__ baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1,
        size_t N);


template <class HandlerType, class InputType, class OutputType>
GatedSpectrometer<HandlerType, InputType, OutputType>::GatedSpectrometer(
    const DadaBufferLayout &dadaBufferLayout, std::size_t selectedSideChannel,
    std::size_t selectedBit, std::size_t fft_length, std::size_t naccumulate,
    std::size_t nbits, float input_level, float output_level, HandlerType
    &handler) : _dadaBufferLayout(dadaBufferLayout),
    _selectedSideChannel(selectedSideChannel), _selectedBit(selectedBit),
    _fft_length(fft_length), _naccumulate(naccumulate),
    _handler(handler), _fft_plan(0), _call_count(0), _nsamps_per_heap(4096)
{

  // Sanity checks
  assert(((nbits == 12) || (nbits == 8)));
  assert(_naccumulate > 0);

  // check for any device errors
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  BOOST_LOG_TRIVIAL(info)
      << "Creating new GatedSpectrometer instance with parameters: \n"
      << "  fft_length           " << _fft_length << "\n"
      << "  naccumulate          " << _naccumulate << "\n"
      << "  nSideChannels        " << _dadaBufferLayout.getNSideChannels() << "\n"
      << "  speadHeapSize        " << _dadaBufferLayout.getHeapSize() << " byte\n"
      << "  selectedSideChannel  " << _selectedSideChannel << "\n"
      << "  selectedBit          " << _selectedBit << "\n"
      << "  output bit depth     " << sizeof(IntegratedPowerType) * 8;

  assert((_dadaBufferLayout.getNSideChannels() == 0) ||
         (selectedSideChannel < _dadaBufferLayout.getNSideChannels()));  // Sanity check of side channel value
  assert(selectedBit < 64); // Sanity check of selected bit


  _nchans = _fft_length / 2 + 1;

  // Calculate the scaling parameters for 8 bit output
  float dof = 2 * _naccumulate;
  float scale =
      std::pow(input_level * std::sqrt(static_cast<float>(_nchans)), 2);
  float offset = scale * dof;
  float scaling = scale * std::sqrt(2 * dof) / output_level;
  BOOST_LOG_TRIVIAL(debug)
      << "Correction factors for 8-bit conversion: offset = " << offset
      << ", scaling = " << scaling;

  inputDataStream = new InputType(fft_length, nbits, _dadaBufferLayout);

  //How many output spectra per input block?
  size_t nsamps_per_output_spectra = fft_length * naccumulate;

  size_t nsamps_per_pol = inputDataStream->getSamplesPerInputPolarization();
  if (nsamps_per_output_spectra <= nsamps_per_pol)
  { // one buffer block is used for one or multiple output spectra
    size_t N = nsamps_per_pol / nsamps_per_output_spectra;
    // All data in one block has to be used
    assert(N * nsamps_per_output_spectra == nsamps_per_pol);
    _nBlocks = 1;
  }
  else
  { // multiple blocks are integrated intoone output
    size_t N =  nsamps_per_output_spectra /  nsamps_per_pol;
    // All data in multiple blocks has to be used
    assert(N * nsamps_per_pol == nsamps_per_output_spectra);
    _nBlocks = N;
  }
  BOOST_LOG_TRIVIAL(debug) << "Integrating  " << nsamps_per_output_spectra <<
      " samples from " << _nBlocks << "blocks into one output spectrum.";


  // plan the FFT
  size_t nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;
  int batch = nsamps_per_pol / _fft_length;
  int n[] = {static_cast<int>(_fft_length)};
  BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan: \n"
      << "   fft_length = " << _fft_length << "\n"
      << "   n[0] = " << n[0] << "\n"
      << "   _nchans = " << _nchans << "\n"
      << "   batch = " << batch << "\n";
  CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length, NULL,
                                  1, _nchans, HIPFFT_R2C, batch));


  // We unpack and fft one pol at a time
  _unpacked_voltage_G0.resize(nsamps_per_pol);
  _unpacked_voltage_G1.resize(nsamps_per_pol);
  BOOST_LOG_TRIVIAL(debug) << "  Unpacked voltages size (in samples): " << _unpacked_voltage_G0.size();

  outputDataStream = new OutputType(_nchans, batch / (_naccumulate / _nBlocks));

  CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));
  CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));

  _unpacker.reset(new Unpacker(_proc_stream));
} // constructor


template <class HandlerType, class InputType, class OutputType>
GatedSpectrometer<HandlerType, InputType, OutputType>::~GatedSpectrometer() {
  BOOST_LOG_TRIVIAL(debug) << "Destroying GatedSpectrometer";
  if (_fft_plan)
    hipfftDestroy(_fft_plan);

  delete inputDataStream;
  delete outputDataStream;

  hipStreamDestroy(_h2d_stream);
  hipStreamDestroy(_proc_stream);
  hipStreamDestroy(_d2h_stream);
}


template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::init(RawBytes &block) {
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer init called";
  std::stringstream headerInfo;
  headerInfo << "\n"
      << "# Gated spectrometer parameters: \n"
      << "fft_length               " << _fft_length << "\n"
      << "nchannels                " << _nchans << "\n"
      << "naccumulate              " << _naccumulate << "\n"
      << "selected_side_channel    " << _selectedSideChannel << "\n"
      << "selected_bit             " << _selectedBit << "\n"
      << "output_bit_depth         " << sizeof(IntegratedPowerType) * 8 << "\n"
      << "full_stokes_output       ";
  if (typeid(OutputType) == typeid(GatedFullStokesOutput))
  {
          headerInfo << "yes\n";
  }
  else
  {
          headerInfo << "no\n";
  }

  size_t bEnd = std::strlen(block.ptr());
  if (bEnd + headerInfo.str().size() < block.total_bytes())
  {
    std::strcpy(block.ptr() + bEnd, headerInfo.str().c_str());
  }
  else
  {
    BOOST_LOG_TRIVIAL(warning) << "Header of size " << block.total_bytes()
      << " bytes already contains " << bEnd
      << "bytes. Cannot add gated spectrometer info of size "
      << headerInfo.str().size() << " bytes.";
  }

  _handler.init(block);
}



template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::gated_fft(
  PolarizationData &data,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G0,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G1
        )
{
  BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
  switch (data._nbits) {
  case 8:
    _unpacker->unpack<8>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  case 12:
    _unpacker->unpack<12>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  default:
    throw std::runtime_error("Unsupported number of bits");
  }

  // Loop over outputblocks, for case of multiple output blocks per input block
  int step = data._sideChannelData.b().size() / _noOfBitSetsIn_G0.size();

  for (size_t i = 0; i < _noOfBitSetsIn_G0.size(); i++)
  { // ToDo: Should be in one kernel call
  gating<<<1024, 1024, 0, _proc_stream>>>(
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(_unpacked_voltage_G1.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(data._sideChannelData.b().data() + i * step),
      _unpacked_voltage_G0.size() / _noOfBitSetsIn_G0.size(),
      _dadaBufferLayout.getHeapSize(),
      _selectedBit,
      _dadaBufferLayout.getNSideChannels(),
      _selectedSideChannel,
      thrust::raw_pointer_cast(data._baseLineG0.data()),
      thrust::raw_pointer_cast(data._baseLineG1.data()),
      thrust::raw_pointer_cast(data._baseLineG0_update.data()),
      thrust::raw_pointer_cast(data._baseLineG1_update.data()),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data() + i),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data() + i)
      );
  }

    // only few output blocks per input block thus execution on only one thread.
    // Important is that the execution is async on the GPU.
    update_baselines<<<1,1,0, _proc_stream>>>(
        thrust::raw_pointer_cast(data._baseLineG0.data()),
        thrust::raw_pointer_cast(data._baseLineG1.data()),
        thrust::raw_pointer_cast(data._baseLineG0_update.data()),
        thrust::raw_pointer_cast(data._baseLineG1_update.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data()),
        _noOfBitSetsIn_G0.size()
            );

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 1";
  BOOST_LOG_TRIVIAL(debug) << "Accessing unpacked voltage";
  UnpackedVoltageType *_unpacked_voltage_ptr =
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data());
  BOOST_LOG_TRIVIAL(debug) << "Accessing channelized voltage";
  ChannelisedVoltageType *_channelised_voltage_ptr =
      thrust::raw_pointer_cast(data._channelised_voltage_G0.data());

  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 2";
  _unpacked_voltage_ptr = thrust::raw_pointer_cast(_unpacked_voltage_G1.data());
  _channelised_voltage_ptr = thrust::raw_pointer_cast(data._channelised_voltage_G1.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

//  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
//  BOOST_LOG_TRIVIAL(debug) << "Exit processing";
} // process






template <class HandlerType, class InputType, class OutputType>
bool GatedSpectrometer<HandlerType, InputType, OutputType>::operator()(RawBytes &block) {
  ++_call_count;
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer operator() called (count = "
                           << _call_count << ")";
  if (block.used_bytes() != _dadaBufferLayout.getBufferSize()) { /* Unexpected buffer size */
    BOOST_LOG_TRIVIAL(error) << "Unexpected Buffer Size - Got "
                             << block.used_bytes() << " byte, expected "
                             << _dadaBufferLayout.getBufferSize() << " byte)";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    hipProfilerStop();
    return true;
  }

  // Copy data to device
  CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
  inputDataStream->swap();
  inputDataStream->getFromBlock(block, _h2d_stream);


  if (_call_count == 1) {
    return false;
  }
  // process data

  // check if new outblock is started:  _call_count -1 because this is the block number on the device
  bool newBlock = (((_call_count-1)  % (_nBlocks)) == 0);

  // only if  a newblock is started the output buffer is swapped. Otherwise the
  // new data is added to it
  if (newBlock)
  {
    BOOST_LOG_TRIVIAL(debug) << "Starting new output block.";
    CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));
    outputDataStream->swap(_proc_stream);
  }

  BOOST_LOG_TRIVIAL(debug) << "Processing block.";
  process(inputDataStream, outputDataStream);
  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  BOOST_LOG_TRIVIAL(debug) << "Processing block finished.";
  /// For one pol input and power out
  /// ToDo: For two pol input and power out
  /// ToDo: For two pol input and stokes out


  if ((_call_count == 2) || (!newBlock)) {
    return false;
  }

  outputDataStream->data2Host(_d2h_stream);
  if (_call_count == 3) {
    return false;
  }

  // Wrap in a RawBytes object here;
  RawBytes bytes(reinterpret_cast<char *>(outputDataStream->_host_power.b_ptr()),
                 outputDataStream->_host_power.size(),
                 outputDataStream->_host_power.size());
  BOOST_LOG_TRIVIAL(debug) << "Calling handler";
  // The handler can't do anything asynchronously without a copy here
  // as it would be unsafe (given that it does not own the memory it
  // is being passed).

  _handler(bytes);
  return false; //
} // operator ()



template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::process(SinglePolarizationInput *inputDataStream, GatedPowerSpectrumOutput *outputDataStream)
{
  gated_fft(*inputDataStream, outputDataStream->G0._noOfBitSets.a(), outputDataStream->G1._noOfBitSets.a());



  kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
            thrust::raw_pointer_cast(inputDataStream->_channelised_voltage_G0.data()),
            thrust::raw_pointer_cast(outputDataStream->G0.data.a().data()),
            _nchans,
            inputDataStream->_channelised_voltage_G0.size() / _nchans,
            _naccumulate / _nBlocks,
            1, 0., 1, 0);

  kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
            thrust::raw_pointer_cast(inputDataStream->_channelised_voltage_G1.data()),
            thrust::raw_pointer_cast(outputDataStream->G1.data.a().data()),
            _nchans,
            inputDataStream->_channelised_voltage_G1.size() / _nchans,
            _naccumulate / _nBlocks,
            1, 0., 1, 0);

}


template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::process(DualPolarizationInput *inputDataStream, GatedFullStokesOutput *outputDataStream)
{
  mergeSideChannels<<<1024, 1024, 0, _proc_stream>>>(thrust::raw_pointer_cast(inputDataStream->polarization0._sideChannelData.a().data()),
          thrust::raw_pointer_cast(inputDataStream->polarization1._sideChannelData.a().data()), inputDataStream->polarization1._sideChannelData.a().size());

  gated_fft(inputDataStream->polarization0, outputDataStream->G0._noOfBitSets.a(), outputDataStream->G1._noOfBitSets.a());
  gated_fft(inputDataStream->polarization1, outputDataStream->G0._noOfBitSets.a(), outputDataStream->G1._noOfBitSets.a());

  for(int output_block_number = 0; output_block_number < outputDataStream->G0._noOfBitSets.size(); output_block_number++)
  {
      size_t input_offset = output_block_number * inputDataStream->polarization0._channelised_voltage_G0.size() / outputDataStream->G0._noOfBitSets.size();
      size_t output_offset = output_block_number * outputDataStream->G0.I.a().size() / outputDataStream->G0._noOfBitSets.size();
      BOOST_LOG_TRIVIAL(debug) << "Accumulating data for output block " << output_block_number << " with input offset " << input_offset << " and output_offset " << output_offset;
      stokes_accumulate<<<1024, 1024, 0, _proc_stream>>>(
              thrust::raw_pointer_cast(inputDataStream->polarization0._channelised_voltage_G0.data() + input_offset),
              thrust::raw_pointer_cast(inputDataStream->polarization1._channelised_voltage_G0.data() + input_offset),
              thrust::raw_pointer_cast(outputDataStream->G0.I.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G0.Q.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G0.U.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G0.V.a().data() + output_offset),
              _nchans, _naccumulate / _nBlocks
              );

      stokes_accumulate<<<1024, 1024, 0, _proc_stream>>>(
              thrust::raw_pointer_cast(inputDataStream->polarization0._channelised_voltage_G1.data() + input_offset),
              thrust::raw_pointer_cast(inputDataStream->polarization1._channelised_voltage_G1.data() + input_offset),
              thrust::raw_pointer_cast(outputDataStream->G1.I.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G1.Q.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G1.U.a().data() + output_offset),
              thrust::raw_pointer_cast(outputDataStream->G1.V.a().data() + output_offset),
              _nchans, _naccumulate / _nBlocks
              );
  }
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G0.I.a().begin(), outputDataStream->G0.I.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G0.Q.a().begin(), outputDataStream->G0.Q.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G0.U.a().begin(), outputDataStream->G0.U.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G0.V.a().begin(), outputDataStream->G0.V.a().end(), _call_count);


 // thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G1.I.a().begin(), outputDataStream->G1.I.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G1.Q.a().begin(), outputDataStream->G1.Q.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G1.U.a().begin(), outputDataStream->G1.U.a().end(), _call_count);
  //thrust::fill(thrust::cuda::par.on(_proc_stream),outputDataStream->G1.V.a().begin(), outputDataStream->G1.V.a().end(), _call_count);

  //  hipDeviceSynchronize();
}

} // edd
} // effelsberg
} // psrdada_cpp

