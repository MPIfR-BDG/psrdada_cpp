#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"
#include "psrdada_cpp/effelsberg/edd/Tools.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/execution_policy.h>

#include <iostream>
#include <iomanip>
#include <cstring>
#include <sstream>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

// Reduce thread local vatiable v in shared array x, so that x[0]
template<typename T>
__device__ void sum_reduce(T *x, const T &v)
{
  x[threadIdx.x] = v;
  __syncthreads();
  for(int s = blockDim.x / 2; s > 0; s = s / 2)
  {
    if (threadIdx.x < s)
      x[threadIdx.x] += x[threadIdx.x + s];
    __syncthreads();
  }
}


// If one of the side channel items is lsot, then both are considered as lost
// here
__global__ void mergeSideChannels(uint64_t* __restrict__ A, uint64_t* __restrict__ B, size_t N)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x)
  {
    uint64_t v = A[i] || B[i];
    A[i] = v;
    B[i] = v;
  }
}


__global__ void gating(float* __restrict__ G0,
        float* __restrict__ G1,
        const uint64_t* __restrict__ sideChannelData,
        size_t N, size_t heapSize, size_t bitpos,
        size_t noOfSideChannels, size_t selectedSideChannel,
        const float*  __restrict__ _baseLineG0,
        const float*  __restrict__ _baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1) {
  // statistics values for samopels to G0, G1
  uint32_t _G0stats = 0;
  uint32_t _G1stats = 0;

  const float baseLineG0 = _baseLineG0[0];
  const float baseLineG1 = _baseLineG1[0];

  float baselineUpdateG0 = 0;
  float baselineUpdateG1 = 0;

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    const float v = G0[i];

    const uint64_t sideChannelItem = sideChannelData[((i / heapSize) * (noOfSideChannels)) +
                        selectedSideChannel];

    const unsigned int bit_set = TEST_BIT(sideChannelItem, bitpos);
    const unsigned int heap_lost = TEST_BIT(sideChannelItem, 63);
    G1[i] = (v - baseLineG1) * bit_set * (!heap_lost) + baseLineG1;
    G0[i] = (v - baseLineG0) * (!bit_set) *(!heap_lost) + baseLineG0;

    _G0stats += (!bit_set) *(!heap_lost);
    _G1stats += bit_set * (!heap_lost);

    baselineUpdateG1 += v * bit_set * (!heap_lost);
    baselineUpdateG0 += v * (!bit_set) *(!heap_lost);
  }

  __shared__ uint32_t x[1024];

  // Reduce G0, G1
  sum_reduce<uint32_t>(x, _G0stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G0,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<uint32_t>(x, _G1stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G1,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  //reuse shared array
  float *y = (float*) x;
  //update the baseline array
  sum_reduce<float>(y, baselineUpdateG0);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG0, y[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<float>(y, baselineUpdateG1);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG1, y[threadIdx.x]);
  }
  __syncthreads();
}



// Updates the baselines of the gates for the polarization set for the next
// block
// only few output blocks per input block thus execution on only one thread.
// Important is that the execution is async on the GPU.
__global__ void update_baselines(float*  __restrict__ baseLineG0,
        float*  __restrict__ baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1,
        size_t N)
{
    size_t NG0 = 0;
    size_t NG1 = 0;

    for (size_t i =0; i < N; i++)
    {
       NG0 += stats_G0[i];
       NG1 += stats_G1[i];
    }

    baseLineG0[0] = baseLineNG0[0] / NG0;
    baseLineG1[0] = baseLineNG1[0] / NG1;
    baseLineNG0[0] = 0;
    baseLineNG1[0] = 0;
}








template <class HandlerType>
GatedSpectrometer<HandlerType>::GatedSpectrometer(
    const DadaBufferLayout &dadaBufferLayout,
    std::size_t selectedSideChannel, std::size_t selectedBit, std::size_t fft_length, std::size_t naccumulate,
    std::size_t nbits, float input_level, float output_level,
    HandlerType &handler) : _dadaBufferLayout(dadaBufferLayout),
      _selectedSideChannel(selectedSideChannel), _selectedBit(selectedBit),
      _fft_length(fft_length),
      _naccumulate(naccumulate), _nbits(nbits), _handler(handler), _fft_plan(0),
      _call_count(0), _nsamps_per_heap(4096), _processing_efficiency(0.){

  // Sanity checks
  assert(((_nbits == 12) || (_nbits == 8)));
  assert(_naccumulate > 0);

  // check for any device errors
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  BOOST_LOG_TRIVIAL(info)
      << "Creating new GatedSpectrometer instance with parameters: \n"
      << "  fft_length           " << _fft_length << "\n"
      << "  naccumulate          " << _naccumulate << "\n"
      << "  nSideChannels        " << _dadaBufferLayout.getNSideChannels() << "\n"
      << "  speadHeapSize        " << _dadaBufferLayout.getHeapSize() << " byte\n"
      << "  selectedSideChannel  " << _selectedSideChannel << "\n"
      << "  selectedBit          " << _selectedBit << "\n"
      << "  output bit depth     " << sizeof(IntegratedPowerType) * 8;

  assert((_dadaBufferLayout.getNSideChannels() == 0) ||
         (selectedSideChannel < _dadaBufferLayout.getNSideChannels()));  // Sanity check of side channel value
  assert(selectedBit < 64); // Sanity check of selected bit

   _nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;

  _nsamps_per_output_spectra = fft_length * naccumulate;
  if (_nsamps_per_output_spectra <= _nsamps_per_buffer)
  { // one buffer block is used for one or multiple output spectra
    size_t N = _nsamps_per_buffer / _nsamps_per_output_spectra;
    // All data in one block has to be used
    assert(N * _nsamps_per_output_spectra == _nsamps_per_buffer);
    _nBlocks = 1;
  }
  else
  { // multiple blocks are integrated intoone output
    size_t N =  _nsamps_per_output_spectra /  _nsamps_per_buffer;
    // All data in multiple blocks has to be used
    assert(N * _nsamps_per_buffer == _nsamps_per_output_spectra);
    _nBlocks = N;
  }
  BOOST_LOG_TRIVIAL(debug) << "Integrating  " << _nsamps_per_output_spectra << " samples from " << _nBlocks << " into one spectra.";

  _nchans = _fft_length / 2 + 1;
  int batch = _nsamps_per_buffer / _fft_length;
  float dof = 2 * _naccumulate;
  float scale =
      std::pow(input_level * std::sqrt(static_cast<float>(_nchans)), 2);
  float offset = scale * dof;
  float scaling = scale * std::sqrt(2 * dof) / output_level;
  BOOST_LOG_TRIVIAL(debug)
      << "Correction factors for 8-bit conversion: offset = " << offset
      << ", scaling = " << scaling;

  BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan";
  int n[] = {static_cast<int>(_fft_length)};
  CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length, NULL,
                                  1, _nchans, HIPFFT_R2C, batch));
  hipfftSetStream(_fft_plan, _proc_stream);

  BOOST_LOG_TRIVIAL(debug) << "Allocating memory";

  // if singlePol
  inputDataStream = new PolarizationData();
  inputDataStream->resize(_nchans, batch, _dadaBufferLayout);

  _unpacked_voltage_G0.resize(_nsamps_per_buffer);
  _unpacked_voltage_G1.resize(_nsamps_per_buffer);
  BOOST_LOG_TRIVIAL(debug) << "  Unpacked voltages size (in samples): "
                           << _unpacked_voltage_G0.size();

  outputDataStream = new GatedPowerSpectrumOutput();
  outputDataStream->resize(_nchans, batch / (_naccumulate / _nBlocks));



  CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));
  CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));

  _unpacker.reset(new Unpacker(_proc_stream));
  _detector.reset(new DetectorAccumulator<IntegratedPowerType>(_nchans, _naccumulate / _nBlocks, scaling,
                                          offset, _proc_stream));
} // constructor


template <class HandlerType>
GatedSpectrometer<HandlerType>::~GatedSpectrometer() {
  BOOST_LOG_TRIVIAL(debug) << "Destroying GatedSpectrometer";
  if (!_fft_plan)
    hipfftDestroy(_fft_plan);
  hipStreamDestroy(_h2d_stream);
  hipStreamDestroy(_proc_stream);
  hipStreamDestroy(_d2h_stream);
}


template <class HandlerType>
void GatedSpectrometer<HandlerType>::init(RawBytes &block) {
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer init called";
  std::stringstream headerInfo;
  headerInfo << "\n"
      << "# Gated spectrometer parameters: \n"
      << "fft_length               " << _fft_length << "\n"
      << "nchannels                " << _fft_length << "\n"
      << "naccumulate              " << _naccumulate << "\n"
      << "selected_side_channel    " << _selectedSideChannel << "\n"
      << "selected_bit             " << _selectedBit << "\n"
      << "output_bit_depth         " << sizeof(IntegratedPowerType) * 8;

  size_t bEnd = std::strlen(block.ptr());
  if (bEnd + headerInfo.str().size() < block.total_bytes())
  {
    std::strcpy(block.ptr() + bEnd, headerInfo.str().c_str());
  }
  else
  {
    BOOST_LOG_TRIVIAL(warning) << "Header of size " << block.total_bytes()
      << " bytes already contains " << bEnd
      << "bytes. Cannot add gated spectrometer info of size "
      << headerInfo.str().size() << " bytes.";
  }

  _handler.init(block);
}



template <class HandlerType>
void GatedSpectrometer<HandlerType>::gated_fft(
  PolarizationData &data,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G0,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G1
        )
{
  BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
  switch (_nbits) {
  case 8:
    _unpacker->unpack<8>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  case 12:
    _unpacker->unpack<12>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  default:
    throw std::runtime_error("Unsupported number of bits");
  }

  // Loop over outputblocks, for case of multiple output blocks per input block
  int step = data._sideChannelData.b().size() / _noOfBitSetsIn_G0.size();

  for (size_t i = 0; i < _noOfBitSetsIn_G0.size(); i++)
  { // ToDo: Should be in one kernel call
  gating<<<1024, 1024, 0, _proc_stream>>>(
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(_unpacked_voltage_G1.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(data._sideChannelData.b().data() + i * step),
      _unpacked_voltage_G0.size() / _noOfBitSetsIn_G0.size(),
      _dadaBufferLayout.getHeapSize(),
      _selectedBit,
      _dadaBufferLayout.getNSideChannels(),
      _selectedSideChannel,
      thrust::raw_pointer_cast(data._baseLineG0.data()),
      thrust::raw_pointer_cast(data._baseLineG1.data()),
      thrust::raw_pointer_cast(data._baseLineG0_update.data()),
      thrust::raw_pointer_cast(data._baseLineG1_update.data()),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data() + i),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data() + i)
      );
  }

    // only few output blocks per input block thus execution on only one thread.
    // Important is that the execution is async on the GPU.
    update_baselines<<<1,1,0, _proc_stream>>>(
        thrust::raw_pointer_cast(data._baseLineG0.data()),
        thrust::raw_pointer_cast(data._baseLineG1.data()),
        thrust::raw_pointer_cast(data._baseLineG0_update.data()),
        thrust::raw_pointer_cast(data._baseLineG1_update.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data()),
        _noOfBitSetsIn_G0.size()
            );

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 1";
  UnpackedVoltageType *_unpacked_voltage_ptr =
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data());
  ChannelisedVoltageType *_channelised_voltage_ptr =
      thrust::raw_pointer_cast(data._channelised_voltage_G0.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 2";
  _unpacked_voltage_ptr = thrust::raw_pointer_cast(_unpacked_voltage_G1.data());
  _channelised_voltage_ptr = thrust::raw_pointer_cast(data._channelised_voltage_G1.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  BOOST_LOG_TRIVIAL(debug) << "Exit processing";
} // process






template <class HandlerType>
bool GatedSpectrometer<HandlerType>::operator()(RawBytes &block) {
  ++_call_count;
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer operator() called (count = "
                           << _call_count << ")";
  if (block.used_bytes() != _dadaBufferLayout.getBufferSize()) { /* Unexpected buffer size */
    BOOST_LOG_TRIVIAL(error) << "Unexpected Buffer Size - Got "
                             << block.used_bytes() << " byte, expected "
                             << _dadaBufferLayout.getBufferSize() << " byte)";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    hipProfilerStop();
    return true;
  }

  // Copy data to device
  CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
  inputDataStream->swap();
  inputDataStream->getFromBlock(block, _dadaBufferLayout, _h2d_stream);


  if (_call_count == 1) {
    return false;
  }
  // process data

  // check if new outblock is started:  _call_count -1 because this is the block number on the device
  bool newBlock = (((_call_count-1) * _nsamps_per_buffer) % _nsamps_per_output_spectra == 0);

  // only if  a newblock is started the output buffer is swapped. Otherwise the
  // new data is added to it
  if (newBlock)
  {
    BOOST_LOG_TRIVIAL(debug) << "Starting new output block.";
    outputDataStream->swap();
    outputDataStream->reset(_proc_stream);
  }

  /// For one pol input and power out
  /// ToDo: For two pol input and power out
  /// ToDo: For two pol input and stokes out
  PolarizationData *polData = dynamic_cast<PolarizationData*>(inputDataStream);
  GatedPowerSpectrumOutput *powOut = dynamic_cast<GatedPowerSpectrumOutput*>(outputDataStream);
  gated_fft(*polData, powOut->G0._noOfBitSets.a(), powOut->G1._noOfBitSets.a());


//  float2 const* input_ptr;
//  IntegratedPowerType * output_ptr;
//  = thrust::raw_pointer_cast(input.data());
//      T * output_ptr = thrust::raw_pointer_cast(output.data());
//    input_ptr =
      kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
              thrust::raw_pointer_cast(polData->_channelised_voltage_G0.data()),
              thrust::raw_pointer_cast(powOut->G0.data.a().data()),
              _nchans,
              polData->_channelised_voltage_G0.size() / _nchans,
              _naccumulate / _nBlocks,
              1, 0., 1, 0);

    kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
              thrust::raw_pointer_cast(polData->_channelised_voltage_G1.data()),
              thrust::raw_pointer_cast(powOut->G1.data.a().data()),
              _nchans,
              polData->_channelised_voltage_G1.size() / _nchans,
              _naccumulate / _nBlocks,
              1, 0., 1, 0);


  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));

  if ((_call_count == 2) || (!newBlock)) {
    return false;
  }

    powOut->data2Host(_d2h_stream);
  if (_call_count == 3) {
    return false;
  }

//  // calculate off value
//  BOOST_LOG_TRIVIAL(info) << "Buffer block: " << _call_count-3 << " with " << _noOfBitSetsIn_G0.size() << "x2 output heaps:";
//  size_t total_samples_lost = 0;
//  for (size_t i = 0; i < _noOfBitSetsIn_G0.size(); i++)
//  {
//    size_t memOffset = 2 * i * (_nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
//
//    size_t* on_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType));
//    size_t* off_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
//
//    size_t samples_lost = _nsamps_per_output_spectra - (*on_values) - (*off_values);
//    total_samples_lost += samples_lost;
//
//    BOOST_LOG_TRIVIAL(info) << "    Heap " << i << ":\n"
//      <<"                            Samples with  bit set  : " << *on_values << std::endl
//      <<"                            Samples without bit set: " << *off_values << std::endl
//      <<"                            Samples lost           : " << samples_lost << " out of " << _nsamps_per_output_spectra << std::endl;
//  }
//  double efficiency = 1. - double(total_samples_lost) / (_nsamps_per_output_spectra * _noOfBitSetsIn_G0.size());
//  double prev_average = _processing_efficiency / (_call_count- 3 - 1);
//  _processing_efficiency += efficiency;
//  double average = _processing_efficiency / (_call_count-3);
//  BOOST_LOG_TRIVIAL(info) << "Total processing efficiency of this buffer block:" << std::setprecision(6) << efficiency << ". Run average: " << average << " (Trend: " << std::showpos << (average - prev_average) << ")";
//
//  // Wrap in a RawBytes object here;
  RawBytes bytes(reinterpret_cast<char *>(powOut->_host_power.b_ptr()),
                 powOut->_host_power.size(),
                 powOut->_host_power.size());
  BOOST_LOG_TRIVIAL(debug) << "Calling handler";
  // The handler can't do anything asynchronously without a copy here
  // as it would be unsafe (given that it does not own the memory it
  // is being passed).

  _handler(bytes);
  return false; //
} // operator ()

} // edd
} // effelsberg
} // psrdada_cpp

