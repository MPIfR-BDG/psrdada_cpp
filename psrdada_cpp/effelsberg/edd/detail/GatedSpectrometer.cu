#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/execution_policy.h>

#include <iostream>
#include <iomanip>
#include <cstring>
#include <sstream>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

__global__ void gating(float* __restrict__ G0, float* __restrict__ G1, const uint64_t* __restrict__ sideChannelData,
                       size_t N, size_t heapSize, size_t bitpos,
                       size_t noOfSideChannels, size_t selectedSideChannel, const float* __restrict__ _baseLineN) {
  float baseLine = (*_baseLineN) / N;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    const float w = G0[i] - baseLine;
    const uint64_t sideChannelItem =
        sideChannelData[((i / heapSize) * (noOfSideChannels)) +
                        selectedSideChannel]; // Probably not optimal access as
                                              // same data is copied for several
                                              // threads, but maybe efficiently
                                              // handled by cache?

    const int bit_set = TEST_BIT(sideChannelItem, bitpos);
    G1[i] = w * bit_set + baseLine;
    G0[i] = w * (!bit_set) + baseLine;
  }
}


__global__ void countBitSet(const uint64_t *sideChannelData, size_t N, size_t
    bitpos, size_t noOfSideChannels, size_t selectedSideChannel, size_t
    *nBitsSet)
{
  // really not optimized reduction, but here only trivial array sizes.
  // run only in one block!
  __shared__ size_t x[1024];
  size_t ls = 0;

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    ls += TEST_BIT(sideChannelData[i * noOfSideChannels + selectedSideChannel], bitpos);
  }
  x[threadIdx.x] = ls;

  __syncthreads();
  for(int s = blockDim.x / 2; s > 0; s = s / 2)
  {
    if (threadIdx.x < s)
      x[threadIdx.x] += x[threadIdx.x + s];
    __syncthreads();
  }

  if(threadIdx.x == 0)
   nBitsSet[0] += x[threadIdx.x];
}


// blocksize for the array sum kernel
#define array_sum_Nthreads 1024

__global__ void array_sum(float *in, size_t N, float *out) {
  extern __shared__ float data[];

  size_t tid = threadIdx.x;

  float ls = 0;

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    ls += in[i]; // + in[i + blockDim.x];   // loading two elements increase the used bandwidth by ~10% but requires matching blocksize and size of input array
  }

  data[tid] = ls;
  __syncthreads();

  for (size_t i = blockDim.x / 2; i > 0; i /= 2) {
    if (tid < i) {
      data[tid] += data[tid + i];
    }
    __syncthreads();
  }

  // unroll last warp
  // if (tid < 32)
  //{
  //  warpReduce(data, tid);
  //}

  if (tid == 0) {
    out[blockIdx.x] = data[0];
  }
}


template <class HandlerType, typename IntegratedPowerType>
GatedSpectrometer<HandlerType, IntegratedPowerType>::GatedSpectrometer(
    const DadaBufferLayout &dadaBufferLayout,
    std::size_t selectedSideChannel, std::size_t selectedBit, std::size_t fft_length, std::size_t naccumulate,
    std::size_t nbits, float input_level, float output_level,
    HandlerType &handler) : _dadaBufferLayout(dadaBufferLayout),
      _selectedSideChannel(selectedSideChannel), _selectedBit(selectedBit),
      _fft_length(fft_length),
      _naccumulate(naccumulate), _nbits(nbits), _handler(handler), _fft_plan(0),
      _call_count(0), _nsamps_per_heap(4096) {

  // Sanity checks
  assert(((_nbits == 12) || (_nbits == 8)));
  assert(_naccumulate > 0);

  // check for any device errors
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  BOOST_LOG_TRIVIAL(info)
      << "Creating new GatedSpectrometer instance with parameters: \n"
      << "  fft_length           " << _fft_length << "\n"
      << "  naccumulate          " << _naccumulate << "\n"
      << "  nSideChannels        " << _dadaBufferLayout.getNSideChannels() << "\n"
      << "  speadHeapSize        " << _dadaBufferLayout.getHeapSize() << " byte\n"
      << "  selectedSideChannel  " << _selectedSideChannel << "\n"
      << "  selectedBit          " << _selectedBit << "\n"
      << "  output bit depth     " << sizeof(IntegratedPowerType) * 8;

  assert((_dadaBufferLayout.getNSideChannels() == 0) ||
         (selectedSideChannel < _dadaBufferLayout.getNSideChannels()));  // Sanity check of side channel value
  assert(selectedBit < 64); // Sanity check of selected bit

   _nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;

  _nsamps_per_output_spectra = fft_length * naccumulate;
  int nBlocks;
  if (_nsamps_per_output_spectra <= _nsamps_per_buffer)
  { // one buffer block is used for one or multiple output spectra
    size_t N = _nsamps_per_buffer / _nsamps_per_output_spectra;
    // All data in one block has to be used
    assert(N * _nsamps_per_output_spectra == _nsamps_per_buffer);
    nBlocks = 1;
  }
  else
  { // multiple blocks are integrated intoone output
    size_t N =  _nsamps_per_output_spectra /  _nsamps_per_buffer;
    // All data in multiple blocks has to be used
    assert(N * _nsamps_per_buffer == _nsamps_per_output_spectra);
    nBlocks = N;
  }
  BOOST_LOG_TRIVIAL(debug) << "Integrating  " << _nsamps_per_output_spectra << " samples from " << nBlocks << " into one spectra.";

  _nchans = _fft_length / 2 + 1;
  int batch = _nsamps_per_buffer / _fft_length;
  float dof = 2 * _naccumulate;
  float scale =
      std::pow(input_level * std::sqrt(static_cast<float>(_nchans)), 2);
  float offset = scale * dof;
  float scaling = scale * std::sqrt(2 * dof) / output_level;
  BOOST_LOG_TRIVIAL(debug)
      << "Correction factors for 8-bit conversion: offset = " << offset
      << ", scaling = " << scaling;

  BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan";
  int n[] = {static_cast<int>(_fft_length)};
  CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length, NULL,
                                  1, _nchans, HIPFFT_R2C, batch));
  hipfftSetStream(_fft_plan, _proc_stream);

  BOOST_LOG_TRIVIAL(debug) << "Allocating memory";
  _raw_voltage_db.resize(_dadaBufferLayout.sizeOfData() / sizeof(uint64_t));
  _sideChannelData_db.resize(_dadaBufferLayout.getNSideChannels() * _dadaBufferLayout.getNHeaps());
  BOOST_LOG_TRIVIAL(debug) << "  Input voltages size (in 64-bit words): "
                           << _raw_voltage_db.size();
  _unpacked_voltage_G0.resize(_nsamps_per_buffer);
  _unpacked_voltage_G1.resize(_nsamps_per_buffer);

  _baseLineN.resize(array_sum_Nthreads);
  BOOST_LOG_TRIVIAL(debug) << "  Unpacked voltages size (in samples): "
                           << _unpacked_voltage_G0.size();
  _channelised_voltage.resize(_nchans * batch);
  BOOST_LOG_TRIVIAL(debug) << "  Channelised voltages size: "
                           << _channelised_voltage.size();
  _power_db.resize(_nchans * batch / (_naccumulate / nBlocks) * 2);  // hold on and off spectra to simplify output
  thrust::fill(_power_db.a().begin(), _power_db.a().end(), 0.);
  thrust::fill(_power_db.b().begin(), _power_db.b().end(), 0.);
  BOOST_LOG_TRIVIAL(debug) << "  Powers size: " << _power_db.size() / 2;

  _noOfBitSetsInSideChannel.resize( batch / (_naccumulate / nBlocks));
  thrust::fill(_noOfBitSetsInSideChannel.a().begin(), _noOfBitSetsInSideChannel.a().end(), 0L);
  thrust::fill(_noOfBitSetsInSideChannel.b().begin(), _noOfBitSetsInSideChannel.b().end(), 0L);
  BOOST_LOG_TRIVIAL(debug) << "  Bit set counrer size: " << _noOfBitSetsInSideChannel.size();

  // on the host both power are stored in the same data buffer together with
  // the number of bit sets
  _host_power_db.resize( _power_db.size() * sizeof(IntegratedPowerType) + 2 * sizeof(size_t) * _noOfBitSetsInSideChannel.size());

  CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));
  CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));

  _unpacker.reset(new Unpacker(_proc_stream));
  _detector.reset(new DetectorAccumulator<IntegratedPowerType>(_nchans, _naccumulate / nBlocks, scaling,
                                          offset, _proc_stream));
} // constructor


template <class HandlerType, typename IntegratedPowerType>
GatedSpectrometer<HandlerType, IntegratedPowerType>::~GatedSpectrometer() {
  BOOST_LOG_TRIVIAL(debug) << "Destroying GatedSpectrometer";
  if (!_fft_plan)
    hipfftDestroy(_fft_plan);
  hipStreamDestroy(_h2d_stream);
  hipStreamDestroy(_proc_stream);
  hipStreamDestroy(_d2h_stream);
}


template <class HandlerType, typename IntegratedPowerType>
void GatedSpectrometer<HandlerType, IntegratedPowerType>::init(RawBytes &block) {
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer init called";
  std::stringstream headerInfo;
  headerInfo << "\n"
      << "# Gated spectrometer parameters: \n"
      << "fft_length               " << _fft_length << "\n"
      << "nchannels                " << _fft_length << "\n"
      << "naccumulate              " << _naccumulate << "\n"
      << "selected_side_channel    " << _selectedSideChannel << "\n"
      << "selected_bit             " << _selectedBit << "\n"
      << "output_bit_depth         " << sizeof(IntegratedPowerType) * 8;

  size_t bEnd = std::strlen(block.ptr());
  if (bEnd + headerInfo.str().size() < block.total_bytes())
  {
    std::strcpy(block.ptr() + bEnd, headerInfo.str().c_str());
  }
  else
  {
    BOOST_LOG_TRIVIAL(warning) << "Header of size " << block.total_bytes()
      << " bytes already contains " << bEnd
      << "bytes. Cannot add gated spectrometer info of size "
      << headerInfo.str().size() << " bytes.";
  }

  _handler.init(block);
}


template <class HandlerType, typename IntegratedPowerType>
void GatedSpectrometer<HandlerType, IntegratedPowerType>::process(
    thrust::device_vector<RawVoltageType> const &digitiser_raw,
    thrust::device_vector<uint64_t> const &sideChannelData,
    thrust::device_vector<IntegratedPowerType> &detected, thrust::device_vector<size_t> &noOfBitSet) {
  BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
  switch (_nbits) {
  case 8:
    _unpacker->unpack<8>(digitiser_raw, _unpacked_voltage_G0);
    break;
  case 12:
    _unpacker->unpack<12>(digitiser_raw, _unpacked_voltage_G0);
    break;
  default:
    throw std::runtime_error("Unsupported number of bits");
  }
  BOOST_LOG_TRIVIAL(debug) << "Calculate baseline";
  psrdada_cpp::effelsberg::edd::array_sum<<<64, array_sum_Nthreads, array_sum_Nthreads * sizeof(float), _proc_stream>>>(thrust::raw_pointer_cast(_unpacked_voltage_G0.data()), _unpacked_voltage_G0.size(), thrust::raw_pointer_cast(_baseLineN.data()));
  psrdada_cpp::effelsberg::edd::array_sum<<<1, array_sum_Nthreads, array_sum_Nthreads * sizeof(float), _proc_stream>>>(thrust::raw_pointer_cast(_baseLineN.data()), _baseLineN.size(), thrust::raw_pointer_cast(_baseLineN.data()));

  BOOST_LOG_TRIVIAL(debug) << "Perform gating";
  gating<<<1024, 1024, 0, _proc_stream>>>(
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data()),
      thrust::raw_pointer_cast(_unpacked_voltage_G1.data()),
      thrust::raw_pointer_cast(sideChannelData.data()),
      _unpacked_voltage_G0.size(), _dadaBufferLayout.getHeapSize(), _selectedBit, _dadaBufferLayout.getNSideChannels(),
      _selectedSideChannel, thrust::raw_pointer_cast(_baseLineN.data()));

  for (size_t i = 0; i < _noOfBitSetsInSideChannel.size(); i++)
  { // ToDo: Should be in one kernel call
    countBitSet<<<1, 1024, 0, _proc_stream>>>(thrust::raw_pointer_cast(sideChannelData.data() + i * sideChannelData.size() / _noOfBitSetsInSideChannel.size() ),
          sideChannelData.size() / _noOfBitSetsInSideChannel.size(), _selectedBit,
          _dadaBufferLayout.getNSideChannels(), _selectedBit,
          thrust::raw_pointer_cast(noOfBitSet.data() + i));
    
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  }

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 1";
  UnpackedVoltageType *_unpacked_voltage_ptr =
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data());
  ChannelisedVoltageType *_channelised_voltage_ptr =
      thrust::raw_pointer_cast(_channelised_voltage.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));
  _detector->detect(_channelised_voltage, detected, 2, 0);

  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 2";
  _unpacked_voltage_ptr = thrust::raw_pointer_cast(_unpacked_voltage_G1.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

  _detector->detect(_channelised_voltage, detected, 2, 1);
  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  BOOST_LOG_TRIVIAL(debug) << "Exit processing";
} // process


template <class HandlerType, typename IntegratedPowerType>
bool GatedSpectrometer<HandlerType, IntegratedPowerType>::operator()(RawBytes &block) {
  ++_call_count;
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer operator() called (count = "
                           << _call_count << ")";
  if (block.used_bytes() != _dadaBufferLayout.getBufferSize()) { /* Unexpected buffer size */
    BOOST_LOG_TRIVIAL(error) << "Unexpected Buffer Size - Got "
                             << block.used_bytes() << " byte, expected "
                             << _dadaBufferLayout.getBufferSize() << " byte)";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    hipProfilerStop();
    return true;
  }

  // Copy data to device
  CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
  _raw_voltage_db.swap();
  _sideChannelData_db.swap();

  BOOST_LOG_TRIVIAL(debug) << "   block.used_bytes() = " << block.used_bytes()
                           << ", dataBlockBytes = " << _dadaBufferLayout.sizeOfData() << "\n";

  CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void *>(_raw_voltage_db.a_ptr()),
                                   static_cast<void *>(block.ptr()),
                                   _dadaBufferLayout.sizeOfData() , hipMemcpyHostToDevice,
                                   _h2d_stream));
  CUDA_ERROR_CHECK(hipMemcpyAsync(
      static_cast<void *>(_sideChannelData_db.a_ptr()),
      static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()),
      _dadaBufferLayout.sizeOfSideChannelData(), hipMemcpyHostToDevice, _h2d_stream));
  BOOST_LOG_TRIVIAL(debug) << "First side channel item: 0x" <<   std::setw(12) << std::setfill('0') << std::hex <<  (reinterpret_cast<uint64_t*>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()))[0] << std::dec;


  if (_call_count == 1) {
    return false;
  }
  // process data

  // only if  a newblock is started the output buffer is swapped. Otherwise the
  // new data is added to it
  bool newBlock = false;
  if (((_call_count-1) * _nsamps_per_buffer) % _nsamps_per_output_spectra == 0) // _call_count -1 because this is the block number on the device
  {
    BOOST_LOG_TRIVIAL(debug) << "Starting new output block.";
    newBlock = true;
    _power_db.swap();
    _noOfBitSetsInSideChannel.swap();
    // move to specific stream!
    thrust::fill(thrust::cuda::par.on(_proc_stream),_power_db.a().begin(), _power_db.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), _noOfBitSetsInSideChannel.a().begin(), _noOfBitSetsInSideChannel.a().end(), 0L);
  }

  process(_raw_voltage_db.b(), _sideChannelData_db.b(), _power_db.a(), _noOfBitSetsInSideChannel.a());
  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));

  if ((_call_count == 2) || (!newBlock)) {
    return false;
  }

  // copy data to host if block is finished
  CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));
  _host_power_db.swap();

  for (size_t i = 0; i < _noOfBitSetsInSideChannel.size(); i++)
  {
    size_t memOffset = 2 * i * (_nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
    // copy 2x channel data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power_db.a_ptr() + memOffset) ,
                        static_cast<void *>(_power_db.b_ptr() + 2 * i * _nchans),
                        2 * _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));
    // copy noOf bit set data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power_db.a_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType)),
          static_cast<void *>(_noOfBitSetsInSideChannel.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    BOOST_LOG_TRIVIAL(info) << " TOBR NOF BITS SET: " << _noOfBitSetsInSideChannel.b()[i]; 
  }

  BOOST_LOG_TRIVIAL(debug) << "Copy Data back to host";

  if (_call_count == 3) {
    return false;
  }

  // calculate off value
  BOOST_LOG_TRIVIAL(info) << "Buffer block: " << _call_count << " with " << _noOfBitSetsInSideChannel.size() << " output heaps:";
  for (size_t i = 0; i < _noOfBitSetsInSideChannel.size(); i++)
  {
    size_t memOffset = 2 * i * (_nchans * sizeof(IntegratedPowerType) + sizeof(size_t));

    size_t* on_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType));
    *on_values *= _nsamps_per_heap;
    size_t* off_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
    *off_values =  _nsamps_per_output_spectra - (*on_values);

    BOOST_LOG_TRIVIAL(info) << "    " << i << ": No of samples wo/w. bit set in side channel: " << *on_values << " / " << *off_values << std::endl;
  }

  // Wrap in a RawBytes object here;
  RawBytes bytes(reinterpret_cast<char *>(_host_power_db.b_ptr()),
                 _host_power_db.size(),
                 _host_power_db.size());
  BOOST_LOG_TRIVIAL(debug) << "Calling handler";
  // The handler can't do anything asynchronously without a copy here
  // as it would be unsafe (given that it does not own the memory it
  // is being passed).

  _handler(bytes);
  return false; //
} // operator ()

} // edd
} // effelsberg
} // psrdada_cpp

