#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"
#include "psrdada_cpp/effelsberg/edd/Tools.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/execution_policy.h>

#include <iostream>
#include <iomanip>
#include <cstring>
#include <sstream>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

// Reduce thread local vatiable v in shared array x, so that x[0] contains sum
template<typename T>
__device__ void sum_reduce(T *x, const T &v)
{
  x[threadIdx.x] = v;
  __syncthreads();
  for(int s = blockDim.x / 2; s > 0; s = s / 2)
  {
    if (threadIdx.x < s)
      x[threadIdx.x] += x[threadIdx.x + s];
    __syncthreads();
  }
}


// If one of the side channel items is lost, then both are considered as lost
// here
__global__ void mergeSideChannels(uint64_t* __restrict__ A, uint64_t*
        __restrict__ B, size_t N);


__global__ void gating(float* __restrict__ G0,
        float* __restrict__ G1,
        const uint64_t* __restrict__ sideChannelData,
        size_t N, size_t heapSize, size_t bitpos,
        size_t noOfSideChannels, size_t selectedSideChannel,
        const float*  __restrict__ _baseLineG0,
        const float*  __restrict__ _baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1);


// Updates the baselines of the gates for the polarization set for the next
// block
// only few output blocks per input block thus execution on only one thread.
// Important is that the execution is async on the GPU.
__global__ void update_baselines(float*  __restrict__ baseLineG0,
        float*  __restrict__ baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1,
        size_t N);


template <class HandlerType, class InputType, class OutputType>
GatedSpectrometer<HandlerType, InputType, OutputType>::GatedSpectrometer(
    const DadaBufferLayout &dadaBufferLayout, std::size_t selectedSideChannel,
    std::size_t selectedBit, std::size_t fft_length, std::size_t naccumulate,
    std::size_t nbits, float input_level, float output_level, HandlerType
    &handler) : _dadaBufferLayout(dadaBufferLayout),
    _selectedSideChannel(selectedSideChannel), _selectedBit(selectedBit),
    _fft_length(fft_length), _naccumulate(naccumulate), _nbits(nbits),
    _handler(handler), _fft_plan(0), _call_count(0), _nsamps_per_heap(4096)
{

  // Sanity checks
  assert(((_nbits == 12) || (_nbits == 8)));
  assert(_naccumulate > 0);

  // check for any device errors
  CUDA_ERROR_CHECK(hipDeviceSynchronize());

  BOOST_LOG_TRIVIAL(info)
      << "Creating new GatedSpectrometer instance with parameters: \n"
      << "  fft_length           " << _fft_length << "\n"
      << "  naccumulate          " << _naccumulate << "\n"
      << "  nSideChannels        " << _dadaBufferLayout.getNSideChannels() << "\n"
      << "  speadHeapSize        " << _dadaBufferLayout.getHeapSize() << " byte\n"
      << "  selectedSideChannel  " << _selectedSideChannel << "\n"
      << "  selectedBit          " << _selectedBit << "\n"
      << "  output bit depth     " << sizeof(IntegratedPowerType) * 8;

  assert((_dadaBufferLayout.getNSideChannels() == 0) ||
         (selectedSideChannel < _dadaBufferLayout.getNSideChannels()));  // Sanity check of side channel value
  assert(selectedBit < 64); // Sanity check of selected bit

   _nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;
  _nsamps_per_output_spectra = fft_length * naccumulate;
  if (_nsamps_per_output_spectra <= _nsamps_per_buffer)
  { // one buffer block is used for one or multiple output spectra
    size_t N = _nsamps_per_buffer / _nsamps_per_output_spectra;
    // All data in one block has to be used
    assert(N * _nsamps_per_output_spectra == _nsamps_per_buffer);
    _nBlocks = 1;
  }
  else
  { // multiple blocks are integrated intoone output
    size_t N =  _nsamps_per_output_spectra /  _nsamps_per_buffer;
    // All data in multiple blocks has to be used
    assert(N * _nsamps_per_buffer == _nsamps_per_output_spectra);
    _nBlocks = N;
  }
  BOOST_LOG_TRIVIAL(debug) << "Integrating  " << _nsamps_per_output_spectra <<
      " samples from " << _nBlocks << " into one output spectrum.";

  _nchans = _fft_length / 2 + 1;
  int batch = _nsamps_per_buffer / _fft_length;

  float dof = 2 * _naccumulate;
  float scale =
      std::pow(input_level * std::sqrt(static_cast<float>(_nchans)), 2);
  float offset = scale * dof;
  float scaling = scale * std::sqrt(2 * dof) / output_level;
  BOOST_LOG_TRIVIAL(debug)
      << "Correction factors for 8-bit conversion: offset = " << offset
      << ", scaling = " << scaling;

  int n[] = {static_cast<int>(_fft_length)};
  BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan: \n"
      << "   fft_length = " << _fft_length << "\n"
      << "   n[0] = " << n[0] << "\n"
      << "   _nchans = " << _nchans << "\n"
      << "   batch = " << batch << "\n";


      ;
  CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length, NULL,
                                  1, _nchans, HIPFFT_R2C, batch));

  BOOST_LOG_TRIVIAL(debug) << "Allocating memory";

  inputDataStream = new InputType(fft_length, batch, _dadaBufferLayout);

  _unpacked_voltage_G0.resize(_nsamps_per_buffer);
  _unpacked_voltage_G1.resize(_nsamps_per_buffer);
  BOOST_LOG_TRIVIAL(debug) << "  Unpacked voltages size (in samples): "
                           << _unpacked_voltage_G0.size();

  outputDataStream = new OutputType(_nchans, batch / (_naccumulate / _nBlocks));

  CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
  CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));
  CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));

  _unpacker.reset(new Unpacker(_proc_stream));
} // constructor


template <class HandlerType, class InputType, class OutputType>
GatedSpectrometer<HandlerType, InputType, OutputType>::~GatedSpectrometer() {
  BOOST_LOG_TRIVIAL(debug) << "Destroying GatedSpectrometer";
  hipDeviceSynchronize();
  if (!_fft_plan)
    hipfftDestroy(_fft_plan);
  hipStreamDestroy(_h2d_stream);
  hipStreamDestroy(_proc_stream);
  hipStreamDestroy(_d2h_stream);
}


template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::init(RawBytes &block) {
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer init called";
  std::stringstream headerInfo;
  headerInfo << "\n"
      << "# Gated spectrometer parameters: \n"
      << "fft_length               " << _fft_length << "\n"
      << "nchannels                " << _fft_length /2 + 1 << "\n"
      << "naccumulate              " << _naccumulate << "\n"
      << "selected_side_channel    " << _selectedSideChannel << "\n"
      << "selected_bit             " << _selectedBit << "\n"
      << "output_bit_depth         " << sizeof(IntegratedPowerType) * 8;

  size_t bEnd = std::strlen(block.ptr());
  if (bEnd + headerInfo.str().size() < block.total_bytes())
  {
    std::strcpy(block.ptr() + bEnd, headerInfo.str().c_str());
  }
  else
  {
    BOOST_LOG_TRIVIAL(warning) << "Header of size " << block.total_bytes()
      << " bytes already contains " << bEnd
      << "bytes. Cannot add gated spectrometer info of size "
      << headerInfo.str().size() << " bytes.";
  }

  _handler.init(block);
}



template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::gated_fft(
  PolarizationData &data,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G0,
  thrust::device_vector<uint64_cu> &_noOfBitSetsIn_G1
        )
{
  BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
  switch (_nbits) {
  case 8:
    _unpacker->unpack<8>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  case 12:
    _unpacker->unpack<12>(data._raw_voltage.b(), _unpacked_voltage_G0);
    break;
  default:
    throw std::runtime_error("Unsupported number of bits");
  }

  // Loop over outputblocks, for case of multiple output blocks per input block
  int step = data._sideChannelData.b().size() / _noOfBitSetsIn_G0.size();

  for (size_t i = 0; i < _noOfBitSetsIn_G0.size(); i++)
  { // ToDo: Should be in one kernel call
  gating<<<1024, 1024, 0, _proc_stream>>>(
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(_unpacked_voltage_G1.data() + i * step * _nsamps_per_heap),
      thrust::raw_pointer_cast(data._sideChannelData.b().data() + i * step),
      _unpacked_voltage_G0.size() / _noOfBitSetsIn_G0.size(),
      _dadaBufferLayout.getHeapSize(),
      _selectedBit,
      _dadaBufferLayout.getNSideChannels(),
      _selectedSideChannel,
      thrust::raw_pointer_cast(data._baseLineG0.data()),
      thrust::raw_pointer_cast(data._baseLineG1.data()),
      thrust::raw_pointer_cast(data._baseLineG0_update.data()),
      thrust::raw_pointer_cast(data._baseLineG1_update.data()),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data() + i),
      thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data() + i)
      );
  }

    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    // only few output blocks per input block thus execution on only one thread.
    // Important is that the execution is async on the GPU.
    update_baselines<<<1,1,0, _proc_stream>>>(
        thrust::raw_pointer_cast(data._baseLineG0.data()),
        thrust::raw_pointer_cast(data._baseLineG1.data()),
        thrust::raw_pointer_cast(data._baseLineG0_update.data()),
        thrust::raw_pointer_cast(data._baseLineG1_update.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G0.data()),
        thrust::raw_pointer_cast(_noOfBitSetsIn_G1.data()),
        _noOfBitSetsIn_G0.size()
            );

    CUDA_ERROR_CHECK(hipDeviceSynchronize());
  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 1";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
  BOOST_LOG_TRIVIAL(debug) << "Accessing unpacked voltage";
  UnpackedVoltageType *_unpacked_voltage_ptr =
      thrust::raw_pointer_cast(_unpacked_voltage_G0.data());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
  BOOST_LOG_TRIVIAL(debug) << "Accessing channelized voltage";
  ChannelisedVoltageType *_channelised_voltage_ptr =
      thrust::raw_pointer_cast(data._channelised_voltage_G0.data());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

    CUDA_ERROR_CHECK(hipDeviceSynchronize());
  BOOST_LOG_TRIVIAL(debug) << "Performing FFT 2";
  _unpacked_voltage_ptr = thrust::raw_pointer_cast(_unpacked_voltage_G1.data());
  _channelised_voltage_ptr = thrust::raw_pointer_cast(data._channelised_voltage_G1.data());
  CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal *)_unpacked_voltage_ptr,
                                 (hipfftComplex *)_channelised_voltage_ptr));

    CUDA_ERROR_CHECK(hipDeviceSynchronize());
  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
  BOOST_LOG_TRIVIAL(debug) << "Exit processing";
} // process






template <class HandlerType, class InputType, class OutputType>
bool GatedSpectrometer<HandlerType, InputType, OutputType>::operator()(RawBytes &block) {
  ++_call_count;
  BOOST_LOG_TRIVIAL(debug) << "GatedSpectrometer operator() called (count = "
                           << _call_count << ")";
  if (block.used_bytes() != _dadaBufferLayout.getBufferSize()) { /* Unexpected buffer size */
    BOOST_LOG_TRIVIAL(error) << "Unexpected Buffer Size - Got "
                             << block.used_bytes() << " byte, expected "
                             << _dadaBufferLayout.getBufferSize() << " byte)";
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    hipProfilerStop();
    return true;
  }

  // Copy data to device
  CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
  inputDataStream->swap();
  inputDataStream->getFromBlock(block, _h2d_stream);


  if (_call_count == 1) {
    return false;
  }
  // process data

  // check if new outblock is started:  _call_count -1 because this is the block number on the device
  bool newBlock = (((_call_count-1) * _nsamps_per_buffer) % _nsamps_per_output_spectra == 0);

  // only if  a newblock is started the output buffer is swapped. Otherwise the
  // new data is added to it
  if (newBlock)
  {
    BOOST_LOG_TRIVIAL(debug) << "Starting new output block.";
    outputDataStream->swap();
    outputDataStream->reset(_proc_stream);
  }

  BOOST_LOG_TRIVIAL(debug) << "Processing block.";
  hipDeviceSynchronize();
  process(inputDataStream, outputDataStream);
  hipDeviceSynchronize();
  BOOST_LOG_TRIVIAL(debug) << "Processing block finished.";
  /// For one pol input and power out
  /// ToDo: For two pol input and power out
  /// ToDo: For two pol input and stokes out


  if ((_call_count == 2) || (!newBlock)) {
    return false;
  }

    outputDataStream->data2Host(_d2h_stream);
  if (_call_count == 3) {
    return false;
  }

//  // calculate off value
//  BOOST_LOG_TRIVIAL(info) << "Buffer block: " << _call_count-3 << " with " << _noOfBitSetsIn_G0.size() << "x2 output heaps:";
//  size_t total_samples_lost = 0;
//  for (size_t i = 0; i < _noOfBitSetsIn_G0.size(); i++)
//  {
//    size_t memOffset = 2 * i * (_nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
//
//    size_t* on_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType));
//    size_t* off_values = reinterpret_cast<size_t*> (_host_power_db.b_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType) + sizeof(size_t));
//
//    size_t samples_lost = _nsamps_per_output_spectra - (*on_values) - (*off_values);
//    total_samples_lost += samples_lost;
//
//    BOOST_LOG_TRIVIAL(info) << "    Heap " << i << ":\n"
//      <<"                            Samples with  bit set  : " << *on_values << std::endl
//      <<"                            Samples without bit set: " << *off_values << std::endl
//      <<"                            Samples lost           : " << samples_lost << " out of " << _nsamps_per_output_spectra << std::endl;
//  }
//  double efficiency = 1. - double(total_samples_lost) / (_nsamps_per_output_spectra * _noOfBitSetsIn_G0.size());
//  double prev_average = _processing_efficiency / (_call_count- 3 - 1);
//  _processing_efficiency += efficiency;
//  double average = _processing_efficiency / (_call_count-3);
//  BOOST_LOG_TRIVIAL(info) << "Total processing efficiency of this buffer block:" << std::setprecision(6) << efficiency << ". Run average: " << average << " (Trend: " << std::showpos << (average - prev_average) << ")";
//
//  // Wrap in a RawBytes object here;
  RawBytes bytes(reinterpret_cast<char *>(outputDataStream->_host_power.b_ptr()),
                 outputDataStream->_host_power.size(),
                 outputDataStream->_host_power.size());
  BOOST_LOG_TRIVIAL(debug) << "Calling handler";
  // The handler can't do anything asynchronously without a copy here
  // as it would be unsafe (given that it does not own the memory it
  // is being passed).

  _handler(bytes);
  return false; //
} // operator ()



template <class HandlerType, class InputType, class OutputType>
void GatedSpectrometer<HandlerType, InputType, OutputType>::process(PolarizationData *inputDataStream, GatedPowerSpectrumOutput *outputDataStream)
{
  gated_fft(*inputDataStream, outputDataStream->G0._noOfBitSets.a(), outputDataStream->G1._noOfBitSets.a());

  kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
            thrust::raw_pointer_cast(inputDataStream->_channelised_voltage_G0.data()),
            thrust::raw_pointer_cast(outputDataStream->G0.data.a().data()),
            _nchans,
            inputDataStream->_channelised_voltage_G0.size() / _nchans,
            _naccumulate / _nBlocks,
            1, 0., 1, 0);

  kernels::detect_and_accumulate<IntegratedPowerType> <<<1024, 1024, 0, _proc_stream>>>(
            thrust::raw_pointer_cast(inputDataStream->_channelised_voltage_G1.data()),
            thrust::raw_pointer_cast(outputDataStream->G1.data.a().data()),
            _nchans,
            inputDataStream->_channelised_voltage_G1.size() / _nchans,
            _naccumulate / _nBlocks,
            1, 0., 1, 0);

  CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
}



} // edd
} // effelsberg
} // psrdada_cpp

