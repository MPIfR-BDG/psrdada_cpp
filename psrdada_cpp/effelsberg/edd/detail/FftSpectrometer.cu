#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/FftSpectrometer.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"
#include <hip/hip_runtime.h>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

template <class HandlerType>
FftSpectrometer<HandlerType>::FftSpectrometer(
    std::size_t buffer_bytes,
    std::size_t fft_length,
    std::size_t naccumulate,
    std::size_t nbits,
    float input_level,
    float output_level,
    HandlerType& handler)
    : _buffer_bytes(buffer_bytes)
    , _fft_length(fft_length)
    , _naccumulate(naccumulate)
    , _nbits(nbits)
    , _handler(handler)
    , _fft_plan(0)
    , _call_count(0)
{

    assert(((_nbits == 12) || (_nbits == 8)));
    BOOST_LOG_TRIVIAL(debug)
    << "Creating new FftSpectrometer instance with parameters: \n"
    << "fft_length = " << _fft_length << "\n"
    << "naccumulate = " << _naccumulate;
    std::size_t nsamps_per_buffer = buffer_bytes * 8 / nbits;
    assert(nsamps_per_buffer % _fft_length == 0 /*Number of samples is not multiple of FFT size*/);
    std::size_t n64bit_words = buffer_bytes / sizeof(uint64_t);
    _nchans = _fft_length / 2 + 1;
    int batch = nsamps_per_buffer/_fft_length;
    BOOST_LOG_TRIVIAL(debug) << "Calculating scales and offsets";
    float dof = 2 * _naccumulate;
    float scale = std::pow(input_level * std::sqrt(static_cast<float>(_nchans)), 2);
    float offset = scale * dof;
    float scaling = scale * std::sqrt(2 * dof) / output_level;
    BOOST_LOG_TRIVIAL(debug) << "Correction factors for 8-bit conversion: offset = " << offset << ", scaling = " << scaling;
    BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan";
    int n[] = {static_cast<int>(_fft_length)};
    CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length,
        NULL, 1, _fft_length/2 + 1, HIPFFT_R2C, batch));
    hipfftSetStream(_fft_plan, _proc_stream);
    BOOST_LOG_TRIVIAL(debug) << "Allocating memory";
    _raw_voltage_db.resize(n64bit_words);
    BOOST_LOG_TRIVIAL(debug) << "Input voltages size (in 64-bit words): " << _raw_voltage_db.size();
    _unpacked_voltage.resize(nsamps_per_buffer);
    BOOST_LOG_TRIVIAL(debug) << "Unpacked voltages size (in samples): " << _unpacked_voltage.size();
    _channelised_voltage.resize(_nchans * batch);
    BOOST_LOG_TRIVIAL(debug) << "Channelised voltages size: " << _channelised_voltage.size();
    _power_db.resize(_nchans * batch / _naccumulate);
    BOOST_LOG_TRIVIAL(debug) << "Powers size: " << _power_db.size();
    _host_power_db.resize(_power_db.size());
    CUDA_ERROR_CHECK(hipStreamCreate(&_h2d_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_d2h_stream));
    CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));
    _unpacker.reset(new Unpacker(_proc_stream));
    _detector.reset(new DetectorAccumulator(_nchans, _naccumulate,
        scaling, offset, _proc_stream));
}

template <class HandlerType>
FftSpectrometer<HandlerType>::~FftSpectrometer()
{
    BOOST_LOG_TRIVIAL(debug) << "Destroying FftSpectrometer";
    if (!_fft_plan)
        hipfftDestroy(_fft_plan);
    hipStreamDestroy(_h2d_stream);
    hipStreamDestroy(_proc_stream);
    hipStreamDestroy(_d2h_stream);
}

template <class HandlerType>
void FftSpectrometer<HandlerType>::init(RawBytes& block)
{
    BOOST_LOG_TRIVIAL(debug) << "FftSpectrometer init called";
    _handler.init(block);
}

template <class HandlerType>
void FftSpectrometer<HandlerType>::process(
    thrust::device_vector<RawVoltageType> const& digitiser_raw,
    thrust::device_vector<IntegratedPowerType>& detected)
{
    BOOST_LOG_TRIVIAL(debug) << "Unpacking raw voltages";
    switch (_nbits)
    {
        case 8:  _unpacker->unpack<8>(digitiser_raw, _unpacked_voltage); break;
        case 12: _unpacker->unpack<12>(digitiser_raw, _unpacked_voltage); break;
        default: throw std::runtime_error("Unsupported number of bits");
    }
    BOOST_LOG_TRIVIAL(debug) << "Performing FFT";
    UnpackedVoltageType* _unpacked_voltage_ptr = thrust::raw_pointer_cast(_unpacked_voltage.data());
    ChannelisedVoltageType* _channelised_voltage_ptr = thrust::raw_pointer_cast(_channelised_voltage.data());
    CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan,
        (hipfftReal*) _unpacked_voltage_ptr,
        (hipfftComplex*) _channelised_voltage_ptr));
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
    _detector->detect(_channelised_voltage, detected);
}

template <class HandlerType>
bool FftSpectrometer<HandlerType>::operator()(RawBytes& block)
{
    ++_call_count;
    BOOST_LOG_TRIVIAL(debug) << "FftSpectrometer operator() called (count = " << _call_count << ")";
    assert(block.used_bytes() == _buffer_bytes /* Unexpected buffer size */);

    CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
    _raw_voltage_db.swap();

    CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void*>(_raw_voltage_db.a_ptr()),
        static_cast<void*>(block.ptr()), block.used_bytes(),
        hipMemcpyHostToDevice, _h2d_stream));

    if (_call_count == 1)
    {
        return false;
    }

    // Synchronize all streams
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
    _power_db.swap();
    process(_raw_voltage_db.b(), _power_db.a());

    if (_call_count == 2)
    {
        return false;
    }

    CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));
    _host_power_db.swap();
    CUDA_ERROR_CHECK(hipMemcpyAsync(
        static_cast<void*>(_host_power_db.a_ptr()),
        static_cast<void*>(_power_db.b_ptr()),
        _power_db.size() * sizeof(IntegratedPowerType),
        hipMemcpyDeviceToHost,
        _d2h_stream));
    
    if (_call_count == 3)
    {
        return false;
    }   

    //Wrap _detected_host_previous in a RawBytes object here;
    RawBytes bytes(reinterpret_cast<char*>(_host_power_db.b_ptr()),
        _host_power_db.size() * sizeof(IntegratedPowerType),
        _host_power_db.size() * sizeof(IntegratedPowerType));
    BOOST_LOG_TRIVIAL(debug) << "Calling handler";
    // The handler can't do anything asynchronously without a copy here 
    // as it would be unsafe (given that it does not own the memory it 
    // is being passed).
    return _handler(bytes);
}

} //edd
} //effelsberg
} //psrdada_cpp


