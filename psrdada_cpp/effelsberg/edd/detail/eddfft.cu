#include "psrdada_cpp/effelsberg/edd/eddfft.cuh"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/raw_bytes.hpp"
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <hip/hip_runtime.h>

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

template <class HandlerType>
SimpleFFTSpectrometer<HandlerType>::SimpleFFTSpectrometer(
    int nsamps_per_block,
    int fft_length,
    int naccumulate,
    int nbits,
    HandlerType& handler)
    : _nsamps(nsamps_per_block)
    , _fft_length(fft_length)
    , _naccumulate(naccumulate)
    , _nbits(nbits)
    , _handler(handler)
    , _fft_plan(0)
    , _first(true)
    , _second(true)
    , _third(true)
{
    BOOST_LOG_TRIVIAL(debug)
    << "Creating new SimpleFFTSpectrometer instance with parameters: \n"
    << "fft_length = " << _fft_length << "\n"
    << "naccumulate = " << _naccumulate;

    if (_nsamps % _fft_length != 0)
    {
        throw std::runtime_error("Number of samples is not multiple of FFT size");
    }

    if (_nbits != 12)
    {
        throw std::runtime_error("Only 12-bit mode is supported");
    }

    hipStreamCreate(&_h2d_stream);
    hipStreamCreate(&_proc_stream);
    hipStreamCreate(&_d2h_stream);

    _nchans = _fft_length / 2 + 1;

    int n64bit_words = 3 * _nsamps / 16;
    int batch = _nsamps/_fft_length;

    BOOST_LOG_TRIVIAL(debug) << "Generating FFT plan";
    int n[] = {_fft_length};
    CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, NULL, 1, _fft_length,
        NULL, 1, _fft_length/2 + 1, HIPFFT_R2C, batch));
    hipfftSetStream(_fft_plan, _proc_stream);

    BOOST_LOG_TRIVIAL(debug) << "Allocating memory";
    _edd_raw_a.resize(n64bit_words);
    _edd_raw_b.resize(n64bit_words);
    _edd_raw_current  = &_edd_raw_a;
    _edd_raw_previous = &_edd_raw_b;

    _edd_unpacked.resize(_nsamps);
    _channelised.resize(_nchans * batch);

    _detected_a.resize(_nchans * batch / _naccumulate);
    _detected_b.resize(_nchans * batch / _naccumulate);
    _detected_current = &_detected_a;
    _detected_previous = &_detected_b;

    _detected_host_a.resize(_nchans * batch / _naccumulate);
    _detected_host_b.resize(_nchans * batch / _naccumulate);
    _detected_host_current = &_detected_host_a;
    _detected_host_previous = &_detected_host_b;

}

template <class HandlerType>
SimpleFFTSpectrometer<HandlerType>::~SimpleFFTSpectrometer()
{
    BOOST_LOG_TRIVIAL(debug) << "Destroying SimpleFFTSpectrometer";
    if (!_fft_plan)
        hipfftDestroy(_fft_plan);
    hipStreamDestroy(_h2d_stream);
    hipStreamDestroy(_proc_stream);
    hipStreamDestroy(_d2h_stream);
}

template <class HandlerType>
void SimpleFFTSpectrometer<HandlerType>::init(RawBytes& block)
{
    BOOST_LOG_TRIVIAL(debug) << "SimpleFFTSpectrometer init called";
    _handler.init(block);
}


template <class HandlerType>
void SimpleFFTSpectrometer<HandlerType>::process(
    thrust::device_vector<uint64_t>* digitiser_raw,
    thrust::device_vector<float>* detected)
{

    uint64_t* digitiser_raw_ptr = thrust::raw_pointer_cast(digitiser_raw->data());
    float* digitiser_unpacked_ptr = thrust::raw_pointer_cast(_edd_unpacked.data());
    hipfftComplex* channelised_ptr = thrust::raw_pointer_cast(_channelised.data());
    float* detected_ptr = thrust::raw_pointer_cast(detected->data());

    BOOST_LOG_TRIVIAL(debug) << "Unpacking 12-bit data";
    int nblocks = digitiser_raw->size() / NTHREADS_UNPACK;
    kernels::unpack_edd_12bit_to_float32<<< nblocks, NTHREADS_UNPACK, 0, _proc_stream>>>(
        digitiser_raw_ptr, digitiser_unpacked_ptr, digitiser_raw->size());

    BOOST_LOG_TRIVIAL(debug) << "Performing FFT";
    CUFFT_ERROR_CHECK(hipfftExecR2C(_fft_plan, (hipfftReal*) digitiser_unpacked_ptr, channelised_ptr));

    BOOST_LOG_TRIVIAL(debug) << "Detecting and accumulating";
    kernels::detect_and_accumulate<<<1024, 1024, 0, _proc_stream>>>(channelised_ptr, detected_ptr, _nchans, _nsamps/_fft_length, _naccumulate);
}


template <class HandlerType>
bool SimpleFFTSpectrometer<HandlerType>::operator()(RawBytes& block)
{
    BOOST_LOG_TRIVIAL(debug) << "SimpleFFTSpectrometer operator() called";
    int nsamps_in_block = 8 * block.used_bytes() / _nbits;
    if (_nsamps != nsamps_in_block)
    {
        throw std::runtime_error("Received expected number of samples");
    }
    BOOST_LOG_TRIVIAL(debug) << nsamps_in_block << " samples in RawBytes block";


    // Synchronize all streams
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
    std::swap(_detected_current, _detected_previous);

    CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));
    std::swap(_detected_host_current, _detected_host_previous);

    // Start host to device copy
    hipMemcpyAsync((char*) thrust::raw_pointer_cast(_edd_raw_current->data()),
        block.ptr(), block.used_bytes(), hipMemcpyHostToDevice, _h2d_stream);

    /*
    if (_first)
    {
        _first = false;
        return false;
    }*/

    // Guaranteed that the previous copy is completed here
    process(_edd_raw_previous, _detected_current);
    // If this is the first pass, start processing and exit
    /*
    if (_second)
    {
        _second = false;
        return false;
    }*/

    hipMemcpyAsync((char*) thrust::raw_pointer_cast(_detected_host_current->data()),
        (char*) thrust::raw_pointer_cast(_detected_previous->data()),
        _detected_previous->size() * sizeof(float),
        hipMemcpyDeviceToHost, _d2h_stream);

    /*
    if (_third)
    {
        _third = false;
        return false;
    }*/

    //Wrap _detected_host_previous in a RawBytes object here;
    RawBytes bytes((char*) thrust::raw_pointer_cast(_detected_host_previous->data()),
        _detected_host_previous->size() * sizeof(float),
        _detected_host_previous->size() * sizeof(float));
    BOOST_LOG_TRIVIAL(debug) << "Calling handler";


    CUDA_ERROR_CHECK(hipStreamSynchronize(_h2d_stream));
    std::swap(_edd_raw_current, _edd_raw_previous);

    return _handler(bytes);
}

} //edd
} //effelsberg
} //psrdada_cpp


