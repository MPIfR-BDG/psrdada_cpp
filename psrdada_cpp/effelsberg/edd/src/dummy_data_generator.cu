#include "hip/hip_runtime.h"
#include "boost/program_options.hpp"
#include "psrdada_cpp/cli_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/dada_client_base.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/dada_null_sink.hpp"
#include "psrdada_cpp/dada_output_stream.hpp"
#include "psrdada_cpp/multilog.hpp"
#include <thrust/extrema.h>
#include "psrdada_cpp/effelsberg/edd/DadaBufferLayout.hpp"
#include "psrdada_cpp/effelsberg/edd/Packer.cuh"

#include <unistd.h>
#include <iomanip>
#include <cstring>

#include <ctime>
#include <iostream>
#include <time.h>


using namespace psrdada_cpp;


namespace {
const size_t ERROR_IN_COMMAND_LINE = 1;
const size_t SUCCESS = 0;
const size_t ERROR_UNHANDLED_EXCEPTION = 2;
} // namespace


__device__ __forceinline__ uint64_t swap64(uint64_t x)
{
    uint64_t result;
    uint2 t;
    asm("mov.b64 {%0,%1},%2; \n\t"
        : "=r"(t.x), "=r"(t.y) : "l"(x));
    t.x = __byte_perm(t.x, 0, 0x0123);
    t.y = __byte_perm(t.y, 0, 0x0123);
    asm("mov.b64 %0,{%1,%2}; \n\t"
        : "=l"(result) : "r"(t.y), "r"(t.x));
    return result;
}

__global__ void toNetworkEndianess(uint64_t *s, size_t N)
{ 
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    s[i] = swap64(s[i]);
  }
}


int main(int argc, char **argv) {
  try {
    key_t output_key;
    unsigned int input_bit_depth;
    unsigned int delay;
    size_t nSideChannels;
    size_t nblocks;

    size_t speadHeapSize;
    std::string mode;

    /** Define and parse the program options
    */
    namespace po = boost::program_options;
    po::options_description desc("Options");

    desc.add_options()("help,h", "Print help messages");
    desc.add_options()(
        "output_key,o",
        po::value<std::string>()->default_value("dada")->notifier(
            [&output_key](std::string in) {
            output_key = string_to_key(in); }),
        "The shared memory key for the dada buffer to write to (hex "
        "string)");
    desc.add_options()("input_bit_depth,b", po::value<unsigned int>(&input_bit_depth)->required(),
                       "The number of bits per sample in the "
                       "packetiser output (8 or 12)");
//    desc.add_options()("mode,m", po::value<std::string >(&mode)->required(),
//      " Type of data to generate:\n "
//      "  gated: ");
    desc.add_options()("delay,d", po::value<unsigned int>(&delay)->required(),
                       "The delay between writing two consecutive blocks [ms].");

    desc.add_options()("nblocks,n",
                       po::value<size_t>()->default_value(0)->notifier(
                           [&nblocks](size_t in) { nblocks = in; }),
                       "Number of blocks to write in total. Default 0 means no-limit.");
    desc.add_options()("speadheap_size",
                       po::value<size_t>()->default_value(4096)->notifier(
                           [&speadHeapSize](size_t in) { speadHeapSize = in; }),
                       "size of the spead data heaps. The number of the "
                       "heaps in the dada block depends on the number of "
                       "side channel items.");

    desc.add_options()("nsidechannelitems,s",
                       po::value<size_t>()->default_value(1)->notifier(
                           [&nSideChannels](size_t in) { nSideChannels = in; }),
                       "Number of side channel items ( s >= 1)");
    desc.add_options()(
        "log_level", po::value<std::string>()->default_value("info")->notifier(
                         [](std::string level) { set_log_level(level); }),
        "The logging level to use "
        "(debug, info, warning, "
        "error)");

    po::variables_map vm;
    try {
      po::store(po::parse_command_line(argc, argv, desc), vm);
      if (vm.count("help")) {
        std::cout << "Fill dada buffer with dummy data"
                  << std::endl
                  << desc << std::endl;
        return SUCCESS;
      }

       po::notify(vm);

    } catch (po::error &e) {
      std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
      std::cerr << desc << std::endl;
      return ERROR_IN_COMMAND_LINE;
    }
    if (input_bit_depth != 8)
    {
      std::cerr << " Currently only 8 bit supported!\n";
      return ERROR_IN_COMMAND_LINE;
    }

    MultiLog log("edd::DummyDataGenerator");
    DadaOutputStream sink(output_key, log);
    char header[4096];

    std::strcpy(header, "HEADER       DADA\nHDR_VERSION  1.0\nHDR_SIZE     4096\nDADA_VERSION 1.0\nFILE_SIZE    2013265920\nNBIT           32\nNDIM         2\nNPOL         1\nNCHAN     4096\nRESOLUTION   1\nDSB 1\nSYNC_TIME    1234567890\nSAMPLE_CLOCK_START 175671842316288\n");




    RawBytes headerBlock(header, 4096, 4096);
    sink.init(headerBlock);

    effelsberg::edd::DadaBufferLayout dadaBufferLayout(output_key, speadHeapSize, nSideChannels);

    size_t n_samples = dadaBufferLayout.sizeOfData() * 8 / input_bit_depth;

    size_t nFreqs = n_samples/ 2 + 1;

    thrust::device_vector<hipfftComplex> input_dummy_data_freq(nFreqs);
    thrust::device_vector<float> tmp(dadaBufferLayout.sizeOfData() * 8 / input_bit_depth);
    thrust::device_vector<uint32_t> packed_data(tmp.size() * 8 / 32);
    input_dummy_data_freq[nFreqs / 3] = make_hipComplex(50.f, 0.0f);
    input_dummy_data_freq[nFreqs / 2] = make_hipComplex(20.f, 0.0f);

    hipfftHandle plan;
    hipfftPlan1d(&plan, tmp.size(), HIPFFT_C2R, 1);
    hipfftExecC2R(plan, (hipfftComplex*)thrust::raw_pointer_cast(input_dummy_data_freq.data()),(hipfftReal*)thrust::raw_pointer_cast(tmp.data()));


    float min = thrust::min_element(tmp.begin(), tmp.end())[0]; 
    float max = thrust::max_element(tmp.begin(), tmp.end())[0];

    effelsberg::edd::kernels::packNbit<8><<<128, 1024>>>
      (thrust::raw_pointer_cast(tmp.data()), (uint32_t*)thrust::raw_pointer_cast(packed_data.data()), tmp.size(), min, max);

    //toNetworkEndianess<<<64, 1024>>>((uint64_t*)thrust::raw_pointer_cast(packed_data.data()), packed_data.size() /2);
    thrust::host_vector<uint32_t> output(packed_data);

    // convert from 8 bit unsigned to 8 bit signed
    uint8_t *A_unsigned = reinterpret_cast<uint8_t*>(thrust::raw_pointer_cast(output.data()));
    int8_t *A_signed = reinterpret_cast<int8_t*>(thrust::raw_pointer_cast(output.data()));
    for(int i = 0; i < output.size() * 4; i++)
    {
      int f = A_unsigned[i];
      A_signed[i] = f - 128;
    }
    size_t counter = 0;
    while(true)
    {
      counter += 1;
      RawBytes dataBlock((char*) thrust::raw_pointer_cast(output.data()), output.size() * 32 / 8, output.size() * 32 / 8 );
      sink(dataBlock);
      std::cout << "Wrote " << counter << std::endl;
      if (counter == nblocks)
        break;
      usleep(delay * 1000);
    }


  } catch (std::exception &e) {
    std::cerr << "Unhandled Exception reached the top of main: " << e.what()
              << ", application will now exit" << std::endl;
    return ERROR_UNHANDLED_EXCEPTION;
  }
  return SUCCESS;
}

