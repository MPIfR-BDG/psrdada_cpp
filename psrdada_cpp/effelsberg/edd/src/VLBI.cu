#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/VLBI.cuh"
#include "psrdada_cpp/cuda_utils.hpp"

#define EDD_NTHREADS_PACK 1024 
#define NPACK 16

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {
namespace kernels {



__global__
void pack_edd_float32_to_2bit(const float * __restrict__ in, uint32_t* __restrict__ out, size_t n, float minV, float maxV)
{

    __shared__ uint32_t tmp_in[EDD_NTHREADS_PACK];
    //__shared__ uint32_t tmp_in[EDD_NTHREADS_PACK];
    //__shared__ volatile uint8_t tmp_out[EDD_NTHREADS_PACK / 4];

    const float s = (maxV - minV) / 3.;
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n ; idx += gridDim.x * blockDim.x)
    {
        const float delta = (in[idx] - minV);
        tmp_in[threadIdx.x] = 0;
        tmp_in[threadIdx.x] += (delta > 1 * s);
        tmp_in[threadIdx.x] += (delta > 2 * s);
        tmp_in[threadIdx.x] += (delta > 3 * s);
        __syncthreads();

        // can be improved by distributing work on more threads in tree
        // structure, but already at 60-70% memory utilization  
        if (threadIdx.x < EDD_NTHREADS_PACK / NPACK)
        {
          for (size_t i = 1; i < NPACK; i++)
          {
            tmp_in[threadIdx.x * NPACK] += (tmp_in[threadIdx.x * NPACK + i] << (i*2));
          }
          out[(idx - threadIdx.x) / NPACK + threadIdx.x] = tmp_in[threadIdx.x *NPACK];
        }

        __syncthreads();
    }
}


//__global__ void pack_edd_float32_to_2bit(const float* __restrict__ input, uint32_t* __restrict__ output, size_t inputSize, float minV, float maxV)
//{
//  float l = (maxV - minV) / 3;
//  for (size_t i = blockIdx.x * blockDim.x + 16 * threadIdx.x; (i < inputSize);
//       i += blockDim.x * gridDim.x * 16)
//  {
//    uint32_t out = 0;
//    for (size_t j =0; j < 16; j++)
//    {
//      //out = out << 2;
//
//      const float inp = input[i + j];
//      const uint32_t tmp = (inp > minV + l) + (inp > minV + 2 * l) + (inp > minV + 3 * l);
//      out += (tmp << (2 * j));
//      //input[i + j] = i + j;
//    }
//
//    output[i / 16] = out; 
//  }
//}



} //namespace kernels


void pack_2bit(thrust::device_vector<float> const& input, thrust::device_vector<uint8_t>& output, float minV, float maxV, hipStream_t _stream)
{
    BOOST_LOG_TRIVIAL(debug) << "Packing 2-bit data";
    assert(input.size() % NPACK == 0);
    output.resize(input.size() / NPACK * 4);
    BOOST_LOG_TRIVIAL(debug) << "Input size: " << input.size() << " elements";
    BOOST_LOG_TRIVIAL(debug) << "Resizing output buffer to " << output.size() << " elements";

    size_t nblocks = std::min(input.size() / EDD_NTHREADS_PACK, 4096uL);
    BOOST_LOG_TRIVIAL(debug) << "  using " << nblocks << " blocks of " << EDD_NTHREADS_PACK << " threads";

    float const* input_ptr = thrust::raw_pointer_cast(input.data());

    uint32_t* output_ptr = (uint32_t*) thrust::raw_pointer_cast(output.data());

    kernels::pack_edd_float32_to_2bit<<< nblocks, EDD_NTHREADS_PACK, 0, _stream>>>(
            input_ptr, output_ptr, input.size(), minV, maxV);
    CUDA_ERROR_CHECK(hipStreamSynchronize(_stream));
}


// Create  abit mask with 1 between first and lastBit (inclusive) and zero
/// otherwise;
uint32_t bitMask(uint32_t firstBit, uint32_t lastBit)
{
   uint32_t mask = 0U;
   for (uint32_t i=firstBit; i<=lastBit; i++)
       mask |= 1 << i;
   return mask;
}

/// Squeeze a value into the specified bitrange of the target
void setBitsWithValue(uint32_t &target, uint32_t firstBit, uint32_t lastBit, uint32_t value)
{
	// check if value is larger than bit range
	if (value > (1 << (lastBit + 1 - firstBit)))
	{
				std::cerr << "value: " << value << ", 1 << (last-bit - firstbit) " << (1 << (lastBit - firstBit)) << ", bitrange: " << lastBit-firstBit << std::endl;
				throw std::runtime_error("Value does not fit into bitrange");
	}

	uint32_t mask = bitMask(firstBit, lastBit);

	// zero out relevant bits in data
	target &= ~mask;

	// shift value to corerct position
	value = value << firstBit;

	// update target with value
	target |= value;
}

/// get numerical value from the specified bits in the target 
uint32_t getBitsValue(const uint32_t &target, uint32_t firstBit, uint32_t lastBit)
{
	uint32_t mask = bitMask(firstBit, lastBit);

	uint32_t res = target & mask;

	return res >> firstBit;
}


VDIFHeader::VDIFHeader()
{
  for (int i=0; i < 8; i++)
  {
    data[i] = 0U;
  }

  // set standard VDIF header
  setBitsWithValue(data[1], 30, 30, 0);
  setBitsWithValue(data[1], 30, 31, 0);

  // set Version Number to 1
  setBitsWithValue(data[2], 29, 31, 1);
}

uint32_t* VDIFHeader::getData()
{
  return data;
}

void VDIFHeader::setInvalid()
{
  setBitsWithValue(data[0], 31, 31, 1);
}

void VDIFHeader::setValid()
{
  setBitsWithValue(data[0], 31, 31, 0);
}

bool VDIFHeader::isValid() const
{
  return (getBitsValue(data[0], 31, 31) == 0);
}

void VDIFHeader::setSecondsFromReferenceEpoch(uint32_t value)
{
  setBitsWithValue(data[0], 0, 29, value);
}

uint32_t VDIFHeader::getSecondsFromReferenceEpoch() const
{
  return getBitsValue(data[0], 0, 29);
}

void VDIFHeader::setReferenceEpoch(uint32_t value)
{
  setBitsWithValue(data[1], 24, 29, value);
}

uint32_t VDIFHeader::getReferenceEpoch() const
{
  return getBitsValue(data[1], 24, 29);
}

void VDIFHeader::setDataFrameNumber(uint32_t value)
{
  setBitsWithValue(data[1], 0, 23, value);
}

uint32_t VDIFHeader::getDataFrameNumber() const
{
  return getBitsValue(data[1], 0, 23);
}

void VDIFHeader::setDataFrameLength(uint32_t value)
{
  setBitsWithValue(data[2], 0, 23, value);
}

uint32_t VDIFHeader::getDataFrameLength() const
{
  return getBitsValue(data[2], 0, 23);
}

uint32_t VDIFHeader::getVersionNumber() const
{
  return getBitsValue(data[2], 29, 31);
}

void VDIFHeader::setNumberOfChannels(uint32_t value)
{
  setBitsWithValue(data[2], 24, 28, value);
}

uint32_t VDIFHeader::getNumberOfChannels() const
{
  return getBitsValue(data[2], 24, 28);
}

bool VDIFHeader::isRealDataType() const
{
  return (getBitsValue(data[3], 31, 31) == 0);
}

bool VDIFHeader::isComplexDataType() const
{
  return (getBitsValue(data[3], 31, 31) == 1);
}

void VDIFHeader::setComplexDataType()
{
  setBitsWithValue(data[3], 31, 31, 1);
}

void VDIFHeader::setRealDataType()
{
  setBitsWithValue(data[0], 31, 31, 0);
}

void VDIFHeader::setBitsPerSample(uint32_t value)
{
  setBitsWithValue(data[3], 26, 30, value);
}

uint32_t VDIFHeader::getBitsPerSample() const
{
  return getBitsValue(data[3], 26, 30);
}

void VDIFHeader::setThreadId(uint32_t value)
{
  setBitsWithValue(data[3], 16, 25, value);
}

uint32_t VDIFHeader::getThreadId() const
{
  return getBitsValue(data[3], 16, 25);
}

void VDIFHeader::setStationId(uint32_t value)
{
  setBitsWithValue(data[3], 0, 15, value);
}

uint32_t VDIFHeader::getStationId() const
{
  return getBitsValue(data[3], 0, 15);
}





} //namespace edd
} //namespace effelsberg
} //namespace psrdada_cpp
