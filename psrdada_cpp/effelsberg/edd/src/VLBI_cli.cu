#include "hip/hip_runtime.h"
#include "boost/program_options.hpp"
#include "psrdada_cpp/cli_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/dada_client_base.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/dada_null_sink.hpp"
#include "psrdada_cpp/dada_output_stream.hpp"
#include "psrdada_cpp/multilog.hpp"
#include "psrdada_cpp/simple_file_writer.hpp"

#include "psrdada_cpp/effelsberg/edd/VLBI.cuh"

#include <ctime>
#include <iostream>
#include <time.h>


using namespace psrdada_cpp;


namespace {
const size_t ERROR_IN_COMMAND_LINE = 1;
const size_t SUCCESS = 0;
const size_t ERROR_UNHANDLED_EXCEPTION = 2;
} // namespace


int main(int argc, char **argv) {
  try {
    key_t input_key;
    unsigned int nbits;

    size_t speadHeapSize;

    std::time_t now = std::time(NULL);
    std::tm *ptm = std::localtime(&now);
    char buffer[32];
    std::strftime(buffer, 32, "%Y-%m-%d-%H:%M:%S.bp", ptm);
    std::string filename(buffer);
    std::string output_type = "file";

    /** Define and parse the program options
    */
    namespace po = boost::program_options;
    po::options_description desc("Options");

    desc.add_options()("help,h", "Print help messages");
    desc.add_options()(
        "input_key,i",
        po::value<std::string>()->default_value("dada")->notifier(
            [&input_key](std::string in) { input_key = string_to_key(in); }),
        "The shared memory key for the dada buffer to connect to (hex "
        "string)");
    desc.add_options()(
        "output_type", po::value<std::string>(&output_type)->default_value(output_type),
        "output type [dada, file]. Default is file."
        );
    desc.add_options()(
        "output_key,o", po::value<std::string>(&filename)->default_value(filename),
        "The key of the output bnuffer / name of the output file to write spectra "
        "to");
    desc.add_options()("nbits,b", po::value<unsigned int>(&nbits)->required(),
                       "The number of bits per sample in the "
                       "packetiser output (8 or 12)");
    desc.add_options()("speadheap_size",
                       po::value<size_t>()->default_value(4096)->notifier(
                           [&speadHeapSize](size_t in) { speadHeapSize = in; }),
                       "size of the spead data heaps. The number of the "
                       "heaps in the dada block depends on the number of "
                       "side channel items.");

    desc.add_options()(
        "log_level", po::value<std::string>()->default_value("info")->notifier(
                         [](std::string level) { set_log_level(level); }),
        "The logging level to use "
        "(debug, info, warning, "
        "error)");

    po::variables_map vm;
    try {
      po::store(po::parse_command_line(argc, argv, desc), vm);
      if (vm.count("help")) {
        std::cout << "VLBI -- Read EDD data from a DADA buffer "
                     "and convert it to 2 bit VLBI data in VDIF format"
                  << std::endl
                  << desc << std::endl;
        return SUCCESS;
      }

      po::notify(vm);
      if (vm.count("output_type") && (!(output_type == "dada" || output_type == "file") ))
      {
        throw po::validation_error(po::validation_error::invalid_option_value, "output_type", output_type);
      }

    } catch (po::error &e) {
      std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
      std::cerr << desc << std::endl;
      return ERROR_IN_COMMAND_LINE;
    }

    MultiLog log("edd::VLBI");
    DadaClientBase client(input_key, log);
    std::size_t buffer_bytes = client.data_buffer_size();

    // ToDo: Options to set values
    effelsberg::edd::VDIFHeader vdifHeader;
    vdifHeader.setThreadId(0);
    vdifHeader.setStationId(0);
    vdifHeader.setReferenceEpoch(123);
    vdifHeader.setSecondsFromReferenceEpoch(42); // for first block
    double sampleRate = 2.6E9;


    std::cout << "Running with output_type: " << output_type << std::endl;
    if (output_type == "file")
    {
      SimpleFileWriter sink(filename);
      effelsberg::edd::VLBI<decltype(sink)> vlbi(
          buffer_bytes, nbits,
          speadHeapSize, sampleRate, vdifHeader, sink);

      DadaInputStream<decltype(vlbi)> istream(input_key, log, vlbi);
      istream.start();
    }
    else if (output_type == "dada")
    {
      DadaOutputStream sink(string_to_key(filename), log);
      effelsberg::edd::VLBI<decltype(sink)> vlbi(
          buffer_bytes, nbits,
          speadHeapSize, sampleRate, vdifHeader, sink);
      DadaInputStream<decltype(vlbi)> istream(input_key, log, vlbi);
      istream.start();
    }
    else
    {
      throw std::runtime_error("Unknown oputput-type");
    }


  } catch (std::exception &e) {
    std::cerr << "Unhandled Exception reached the top of main: " << e.what()
              << ", application will now exit" << std::endl;
    return ERROR_UNHANDLED_EXCEPTION;
  }
  return SUCCESS;
}

