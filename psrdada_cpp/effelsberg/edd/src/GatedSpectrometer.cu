#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"


namespace psrdada_cpp {
namespace effelsberg {
namespace edd {


__global__ void mergeSideChannels(uint64_t* __restrict__ A, uint64_t*
        __restrict__ B, size_t N)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x)
  {
    uint64_t v = A[i] || B[i];
    A[i] = v;
    B[i] = v;
  }
}


__global__ void gating(float* __restrict__ G0,
        float* __restrict__ G1,
        const uint64_t* __restrict__ sideChannelData,
        size_t N, size_t heapSize, size_t bitpos,
        size_t noOfSideChannels, size_t selectedSideChannel,
        const float*  __restrict__ _baseLineG0,
        const float*  __restrict__ _baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1) {
  // statistics values for samopels to G0, G1
  uint32_t _G0stats = 0;
  uint32_t _G1stats = 0;

  const float baseLineG0 = _baseLineG0[0];
  const float baseLineG1 = _baseLineG1[0];

  float baselineUpdateG0 = 0;
  float baselineUpdateG1 = 0;

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    const float v = G0[i];

    const uint64_t sideChannelItem = sideChannelData[((i / heapSize) * (noOfSideChannels)) +
                        selectedSideChannel];

    const unsigned int bit_set = TEST_BIT(sideChannelItem, bitpos);
    const unsigned int heap_lost = TEST_BIT(sideChannelItem, 63);
    G1[i] = (v - baseLineG1) * bit_set * (!heap_lost) + baseLineG1;
    G0[i] = (v - baseLineG0) * (!bit_set) *(!heap_lost) + baseLineG0;

    _G0stats += (!bit_set) *(!heap_lost);
    _G1stats += bit_set * (!heap_lost);

    baselineUpdateG1 += v * bit_set * (!heap_lost);
    baselineUpdateG0 += v * (!bit_set) *(!heap_lost);
  }

  __shared__ uint32_t x[1024];

  // Reduce G0, G1
  sum_reduce<uint32_t>(x, _G0stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G0,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<uint32_t>(x, _G1stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G1,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  //reuse shared array
  float *y = (float*) x;
  //update the baseline array
  sum_reduce<float>(y, baselineUpdateG0);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG0, y[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<float>(y, baselineUpdateG1);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG1, y[threadIdx.x]);
  }
  __syncthreads();
}



// Updates the baselines of the gates for the polarization set for the next
// block
// only few output blocks per input block thus execution on only one thread.
// Important is that the execution is async on the GPU.
__global__ void update_baselines(float*  __restrict__ baseLineG0,
        float*  __restrict__ baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1,
        size_t N)
{
    size_t NG0 = 0;
    size_t NG1 = 0;

    for (size_t i =0; i < N; i++)
    {
       NG0 += stats_G0[i];
       NG1 += stats_G1[i];
    }

    baseLineG0[0] = baseLineNG0[0] / NG0;
    baseLineG1[0] = baseLineNG1[0] / NG1;
    baseLineNG0[0] = 0;
    baseLineNG1[0] = 0;
}







PolarizationData::PolarizationData(size_t fft_length, size_t batch, const DadaBufferLayout
        &dadaBufferLayout) : _fft_length(fft_length), _batch(batch), _dadaBufferLayout(dadaBufferLayout)
{
    _raw_voltage.resize(_dadaBufferLayout.sizeOfData() / sizeof(uint64_t));
    BOOST_LOG_TRIVIAL(debug) << "  Input voltages size (in 64-bit words): " << _raw_voltage.size();

    _baseLineG0.resize(1);
    _baseLineG0_update.resize(1);
    _baseLineG1.resize(1);
    _baseLineG1_update.resize(1);
    _channelised_voltage_G0.resize((_fft_length / 2 + 1) * _batch);
    _channelised_voltage_G1.resize((_fft_length / 2 + 1) * _batch);
    _sideChannelData.resize(_dadaBufferLayout.getNSideChannels() * _dadaBufferLayout.getNHeaps());
    BOOST_LOG_TRIVIAL(debug) << "  Channelised voltages size: " << _channelised_voltage_G0.size();
};


void PolarizationData::swap()
{
    _raw_voltage.swap();
    _sideChannelData.swap();
}


void PolarizationData::getFromBlock(RawBytes &block, hipStream_t &_h2d_stream)
{
  BOOST_LOG_TRIVIAL(debug) << "   block.used_bytes() = " << block.used_bytes()
                           << ", dataBlockBytes = " << _dadaBufferLayout.sizeOfData() << "\n";

  CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void *>(_raw_voltage.a_ptr()),
                                   static_cast<void *>(block.ptr()),
                                   _dadaBufferLayout.sizeOfData() , hipMemcpyHostToDevice,
                                   _h2d_stream));
  CUDA_ERROR_CHECK(hipMemcpyAsync(
      static_cast<void *>(_sideChannelData.a_ptr()),
      static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()),
      _dadaBufferLayout.sizeOfSideChannelData(), hipMemcpyHostToDevice, _h2d_stream));
  BOOST_LOG_TRIVIAL(debug) << "First side channel item: 0x" <<   std::setw(16)
      << std::setfill('0') << std::hex <<
      (reinterpret_cast<uint64_t*>(block.ptr() + _dadaBufferLayout.sizeOfData()
                                   + _dadaBufferLayout.sizeOfGap()))[0] <<
      std::dec;
}


DualPolarizationData::DualPolarizationData(size_t fft_length, size_t batch, const DadaBufferLayout
        &dadaBufferLayout) : polarization0(fft_length, batch, dadaBufferLayout),
                            polarization1(fft_length, batch, dadaBufferLayout),
                            _dadaBufferLayout(dadaBufferLayout)
{
};

void DualPolarizationData::swap()
{
    polarization0.swap(); polarization1.swap();
}

void DualPolarizationData::getFromBlock(RawBytes &block, hipStream_t &_h2d_stream)
{
// Copy the data with stride to the GPU:
// CPU: P1P2P1P2P1P2 ...
// GPU: P1P1P1 ... P2P2P2 ...
// If this is a bottleneck the gating kernel could sort the layout out
// during copy
int heapsize_bytes =  _dadaBufferLayout.getHeapSize();
CUDA_ERROR_CHECK(hipMemcpy2DAsync(
  static_cast<void *>(polarization0._raw_voltage.a_ptr()),
    heapsize_bytes,
    static_cast<void *>(block.ptr()),
    2 * heapsize_bytes,
    heapsize_bytes, _dadaBufferLayout.sizeOfData() / heapsize_bytes/ 2,
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
  static_cast<void *>(polarization1._raw_voltage.a_ptr()),
    heapsize_bytes,
    static_cast<void *>(block.ptr()) + heapsize_bytes,
    2 * heapsize_bytes,
    heapsize_bytes, _dadaBufferLayout.sizeOfData() / heapsize_bytes/ 2,
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
    static_cast<void *>(polarization0._sideChannelData.a_ptr()),
    sizeof(uint64_t),
    static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()),
    2 * sizeof(uint64_t),
    sizeof(uint64_t),
    _dadaBufferLayout.sizeOfSideChannelData() / 2 / sizeof(uint64_t),
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
    static_cast<void *>(polarization1._sideChannelData.a_ptr()),
    sizeof(uint64_t),
    static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap() + sizeof(uint64_t)),
    2 * sizeof(uint64_t),
    sizeof(uint64_t),
    _dadaBufferLayout.sizeOfSideChannelData() / 2 / sizeof(uint64_t), hipMemcpyHostToDevice, _h2d_stream));

BOOST_LOG_TRIVIAL(debug) << "First side channel item: 0x" <<   std::setw(16)
    << std::setfill('0') << std::hex <<
    (reinterpret_cast<uint64_t*>(block.ptr() + _dadaBufferLayout.sizeOfData()
                                 + _dadaBufferLayout.sizeOfGap()))[0] << std::dec;
}



PowerSpectrumOutput::PowerSpectrumOutput(size_t size, size_t blocks)
{
    BOOST_LOG_TRIVIAL(debug) << "Setting size of power spectrum output size = " << size << ", blocks =  " << blocks;
   data.resize(size * blocks);
   _noOfBitSets.resize(blocks);
}


void PowerSpectrumOutput::reset(hipStream_t &_proc_stream)
{
    thrust::fill(thrust::cuda::par.on(_proc_stream), data.a().begin(), data.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), _noOfBitSets.a().begin(), _noOfBitSets.a().end(), 0L);
}


void PowerSpectrumOutput::swap()
{
    data.swap();
    _noOfBitSets.swap();
}


GatedPowerSpectrumOutput::GatedPowerSpectrumOutput(size_t nchans, size_t
        blocks) : OutputDataStream(nchans, blocks), G0(nchans, blocks),
G1(nchans, blocks)
{
  // on the host both power are stored in the same data buffer together with
  // the number of bit sets
  _host_power.resize( 2 * ( _nchans * sizeof(IntegratedPowerType) + sizeof(size_t) ) * G0._noOfBitSets.size());
}


void GatedPowerSpectrumOutput::reset(hipStream_t &_proc_stream)
{
    G0.reset(_proc_stream);
    G1.reset(_proc_stream);
}


/// Swap output buffers
void GatedPowerSpectrumOutput::swap()
{
    G0.swap();
    G1.swap();
    _host_power.swap();
}


void GatedPowerSpectrumOutput::data2Host(hipStream_t &_d2h_stream)
{
    // copy data to host if block is finished
  CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));

  for (size_t i = 0; i < G0._noOfBitSets.size(); i++)
  {
    // size of individual spectrum + meta
    size_t memslicesize = (_nchans * sizeof(IntegratedPowerType));
    // number of spectra per output
    size_t memOffset = 2 * i * (memslicesize +  + sizeof(size_t));

    // copy 2x channel data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset) ,
                        static_cast<void *>(G0.data.b_ptr() + i * memslicesize),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 1 * memslicesize) ,
                        static_cast<void *>(G1.data.b_ptr() + i * memslicesize),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    // copy noOf bit set data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType)),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType) + sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
  }
}





}}} // namespace
