#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"


namespace psrdada_cpp {
namespace effelsberg {
namespace edd {


__global__ void mergeSideChannels(uint64_t* __restrict__ A, uint64_t*
        __restrict__ B, size_t N)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x)
  {
    uint64_t v = A[i] || B[i];
    A[i] = v;
    B[i] = v;
  }
}


__global__ void gating(float* __restrict__ G0,
        float* __restrict__ G1,
        const uint64_t* __restrict__ sideChannelData,
        size_t N, size_t heapSize, size_t bitpos,
        size_t noOfSideChannels, size_t selectedSideChannel,
        const float*  __restrict__ _baseLineG0,
        const float*  __restrict__ _baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1) {
  // statistics values for samopels to G0, G1
  uint32_t _G0stats = 0;
  uint32_t _G1stats = 0;

  const float baseLineG0 = _baseLineG0[0];
  const float baseLineG1 = _baseLineG1[0];

  float baselineUpdateG0 = 0;
  float baselineUpdateG1 = 0;

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < N);
       i += blockDim.x * gridDim.x) {
    const float v = G0[i];

    const uint64_t sideChannelItem = sideChannelData[((i / heapSize) * (noOfSideChannels)) +
                        selectedSideChannel];

    const unsigned int bit_set = TEST_BIT(sideChannelItem, bitpos);
    const unsigned int heap_lost = TEST_BIT(sideChannelItem, 63);
    G1[i] = (v - baseLineG1) * bit_set * (!heap_lost) + baseLineG1;
    G0[i] = (v - baseLineG0) * (!bit_set) *(!heap_lost) + baseLineG0;

    _G0stats += (!bit_set) *(!heap_lost);
    _G1stats += bit_set * (!heap_lost);

    baselineUpdateG1 += v * bit_set * (!heap_lost);
    baselineUpdateG0 += v * (!bit_set) *(!heap_lost);
  }

  __shared__ uint32_t x[1024];

  // Reduce G0, G1
  sum_reduce<uint32_t>(x, _G0stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G0,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<uint32_t>(x, _G1stats);
  if(threadIdx.x == 0) {
    atomicAdd(stats_G1,  (uint64_cu) x[threadIdx.x]);
  }
  __syncthreads();

  //reuse shared array
  float *y = (float*) x;
  //update the baseline array
  sum_reduce<float>(y, baselineUpdateG0);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG0, y[threadIdx.x]);
  }
  __syncthreads();

  sum_reduce<float>(y, baselineUpdateG1);
  if(threadIdx.x == 0) {
    atomicAdd(baseLineNG1, y[threadIdx.x]);
  }
  __syncthreads();
}



// Updates the baselines of the gates for the polarization set for the next
// block
// only few output blocks per input block thus execution on only one thread.
// Important is that the execution is async on the GPU.
__global__ void update_baselines(float*  __restrict__ baseLineG0,
        float*  __restrict__ baseLineG1,
        float* __restrict__ baseLineNG0,
        float* __restrict__ baseLineNG1,
        uint64_cu* stats_G0, uint64_cu* stats_G1,
        size_t N)
{
    size_t NG0 = 0;
    size_t NG1 = 0;

    for (size_t i =0; i < N; i++)
    {
       NG0 += stats_G0[i];
       NG1 += stats_G1[i];
    }

    baseLineG0[0] = baseLineNG0[0] / NG0;
    baseLineG1[0] = baseLineNG1[0] / NG1;
    baseLineNG0[0] = 0;
    baseLineNG1[0] = 0;
}



/**
 * @brief calculate stokes IQUV from two complex valuies for each polarization
 */
__host__ __device__ void stokes_IQUV(const float2 &p1, const float2 &p2, float &I, float &Q, float &U, float &V)
{
    I = fabs(p1.x*p1.x + p1.y * p1.y) + fabs(p2.x*p2.x + p2.y * p2.y);
    Q = fabs(p1.x*p1.x + p1.y * p1.y) - fabs(p2.x*p2.x + p2.y * p2.y);
    U = 2 * (p1.x*p2.x + p1.y * p2.y);
    V = -2 * (p1.y*p2.x - p1.x * p2.y);
}




/**
 * @brief calculate stokes IQUV spectra pol1, pol2 are arrays of naccumulate
 * complex spectra for individual polarizations
 */
__global__ void stokes_accumulate(float2 const __restrict__ *pol1,
        float2 const __restrict__ *pol2, float *I, float* Q, float *U, float*V,
        int nchans, int naccumulate)
{

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; (i < nchans);
       i += blockDim.x * gridDim.x)
  {
      float rI = 0;
      float rQ = 0;
      float rU = 0;
      float rV = 0;

      for (int k=0; k < naccumulate; k++)
      {
        const float2 p1 = pol1[i + k * nchans];
        const float2 p2 = pol2[i + k * nchans];

        rI += fabs(p1.x * p1.x + p1.y * p1.y) + fabs(p2.x * p2.x + p2.y * p2.y);
        rQ += fabs(p1.x * p1.x + p1.y * p1.y) - fabs(p2.x * p2.x + p2.y * p2.y);
        rU += 2.f * (p1.x * p2.x + p1.y * p2.y);
        rV += -2.f * (p1.y * p2.x - p1.x * p2.y);
      }
      I[i] += rI;
      Q[i] += rQ;
      U[i] += rU;
      V[i] += rV;
  }

}


void PolarizationData::resize(size_t rawVolttageBufferBytes, size_t nsidechannelitems, size_t channelized_samples)
{
    _raw_voltage.resize(rawVolttageBufferBytes / sizeof(uint64_t));
    BOOST_LOG_TRIVIAL(debug) << "  Input voltages size (in 64-bit words): " << _raw_voltage.size();

    _baseLineG0.resize(1);
    _baseLineG0_update.resize(1);
    _baseLineG1.resize(1);
    _baseLineG1_update.resize(1);
    _channelised_voltage_G0.resize(channelized_samples);
    _channelised_voltage_G1.resize(channelized_samples);
    _sideChannelData.resize(nsidechannelitems);
    BOOST_LOG_TRIVIAL(debug) << "  Channelised voltages size: " << _channelised_voltage_G0.size();
}


SinglePolarizationInput::SinglePolarizationInput(size_t fft_length, size_t nbits, const DadaBufferLayout
        &dadaBufferLayout) : PolarizationData(nbits), _fft_length(fft_length), _dadaBufferLayout(dadaBufferLayout)
{

  size_t nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;
  size_t _batch = nsamps_per_buffer / _fft_length;

    resize(_dadaBufferLayout.sizeOfData(), _dadaBufferLayout.getNSideChannels() * _dadaBufferLayout.getNHeaps(), (_fft_length / 2 + 1) * _batch);
};


size_t SinglePolarizationInput::getSamplesPerInputPolarization()
{
    return _dadaBufferLayout.sizeOfData() * 8 / _nbits;
}


void PolarizationData::swap()
{
    _raw_voltage.swap();
    _sideChannelData.swap();
}


void SinglePolarizationInput::getFromBlock(RawBytes &block, hipStream_t &_h2d_stream)
{
  BOOST_LOG_TRIVIAL(debug) << "   block.used_bytes() = " << block.used_bytes()
                           << ", dataBlockBytes = " << _dadaBufferLayout.sizeOfData() << "\n";

  CUDA_ERROR_CHECK(hipMemcpyAsync(static_cast<void *>(_raw_voltage.a_ptr()),
                                   static_cast<void *>(block.ptr()),
                                   _dadaBufferLayout.sizeOfData() , hipMemcpyHostToDevice,
                                   _h2d_stream));
  CUDA_ERROR_CHECK(hipMemcpyAsync(
      static_cast<void *>(_sideChannelData.a_ptr()),
      static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()),
      _dadaBufferLayout.sizeOfSideChannelData(), hipMemcpyHostToDevice, _h2d_stream));
  BOOST_LOG_TRIVIAL(debug) << "First side channel item: 0x" <<   std::setw(16)
      << std::setfill('0') << std::hex <<
      (reinterpret_cast<uint64_t*>(block.ptr() + _dadaBufferLayout.sizeOfData()
                                   + _dadaBufferLayout.sizeOfGap()))[0] <<
      std::dec;
}


DualPolarizationInput::DualPolarizationInput(size_t fft_length, size_t nbits, const DadaBufferLayout
        &dadaBufferLayout) : _fft_length(fft_length),
    polarization0(nbits),
    polarization1(nbits),
    _dadaBufferLayout(dadaBufferLayout)
{

  size_t nsamps_per_buffer = _dadaBufferLayout.sizeOfData() * 8 / nbits;
  size_t _batch = nsamps_per_buffer / _fft_length / 2;

    polarization0.resize(_dadaBufferLayout.sizeOfData() / 2, _dadaBufferLayout.getNSideChannels() * _dadaBufferLayout.getNHeaps() / 2, (_fft_length / 2 + 1) * _batch);
    polarization1.resize(_dadaBufferLayout.sizeOfData() / 2, _dadaBufferLayout.getNSideChannels() * _dadaBufferLayout.getNHeaps() / 2, (_fft_length / 2 + 1) * _batch);
};


void DualPolarizationInput::swap()
{
    polarization0.swap();
    polarization1.swap();
}


size_t DualPolarizationInput::getSamplesPerInputPolarization()
{
    return _dadaBufferLayout.sizeOfData() * 8 / polarization0._nbits / 2;
}


void DualPolarizationInput::getFromBlock(RawBytes &block, hipStream_t &_h2d_stream)
{
// Copy the data with stride to the GPU:
// CPU: P1P2P1P2P1P2 ...
// GPU: P1P1P1 ... P2P2P2 ...
// If this is a bottleneck the gating kernel could sort the layout out
// during copy
int heapsize_bytes =  _dadaBufferLayout.getHeapSize();
CUDA_ERROR_CHECK(hipMemcpy2DAsync(
  static_cast<void *>(polarization0._raw_voltage.a_ptr()),
    heapsize_bytes,
    static_cast<void *>(block.ptr()),
    2 * heapsize_bytes,
    heapsize_bytes, _dadaBufferLayout.sizeOfData() / heapsize_bytes/ 2,
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
  static_cast<void *>(polarization1._raw_voltage.a_ptr()),
    heapsize_bytes,
    static_cast<void *>(block.ptr() + heapsize_bytes),
    2 * heapsize_bytes,
    heapsize_bytes, _dadaBufferLayout.sizeOfData() / heapsize_bytes/ 2,
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
    static_cast<void *>(polarization0._sideChannelData.a_ptr()),
    sizeof(uint64_t),
    static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap()),
    2 * sizeof(uint64_t),
    sizeof(uint64_t),
    _dadaBufferLayout.sizeOfSideChannelData() / 2 / sizeof(uint64_t),
    hipMemcpyHostToDevice, _h2d_stream));

CUDA_ERROR_CHECK(hipMemcpy2DAsync(
    static_cast<void *>(polarization1._sideChannelData.a_ptr()),
    sizeof(uint64_t),
    static_cast<void *>(block.ptr() + _dadaBufferLayout.sizeOfData() + _dadaBufferLayout.sizeOfGap() + sizeof(uint64_t)),
    2 * sizeof(uint64_t),
    sizeof(uint64_t),
    _dadaBufferLayout.sizeOfSideChannelData() / 2 / sizeof(uint64_t), hipMemcpyHostToDevice, _h2d_stream));

BOOST_LOG_TRIVIAL(debug) << "First side channel item: 0x" <<   std::setw(16)
    << std::setfill('0') << std::hex <<
    (reinterpret_cast<uint64_t*>(block.ptr() + _dadaBufferLayout.sizeOfData()
                                 + _dadaBufferLayout.sizeOfGap()))[0] << std::dec;
}



PowerSpectrumOutput::PowerSpectrumOutput(size_t size, size_t blocks)
{
    BOOST_LOG_TRIVIAL(debug) << "Setting size of power spectrum output size = " << size << ", blocks =  " << blocks;
   data.resize(size * blocks);
   _noOfBitSets.resize(blocks);
}


void PowerSpectrumOutput::swap(hipStream_t &_proc_stream)
{
    data.swap();
    _noOfBitSets.swap();
    thrust::fill(thrust::cuda::par.on(_proc_stream), data.a().begin(), data.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), _noOfBitSets.a().begin(), _noOfBitSets.a().end(), 0L);
}


GatedPowerSpectrumOutput::GatedPowerSpectrumOutput(size_t nchans, size_t
        blocks) : OutputDataStream(nchans, blocks), G0(nchans, blocks),
G1(nchans, blocks)
{
  // on the host both power are stored in the same data buffer together with
  // the number of bit sets
  _host_power.resize( 2 * ( _nchans * sizeof(IntegratedPowerType) + sizeof(size_t) ) * G0._noOfBitSets.size());
}


/// Swap output buffers
void GatedPowerSpectrumOutput::swap(hipStream_t &_proc_stream)
{
    G0.swap(_proc_stream);
    G1.swap(_proc_stream);
    _host_power.swap();
}


void GatedPowerSpectrumOutput::data2Host(hipStream_t &_d2h_stream)
{
    // copy data to host if block is finished
  CUDA_ERROR_CHECK(hipStreamSynchronize(_d2h_stream));

  for (size_t i = 0; i < G0._noOfBitSets.size(); i++)
  {
    // size of individual spectrum + meta
    size_t memslicesize = (_nchans * sizeof(IntegratedPowerType));
    // number of spectra per output
    size_t memOffset = 2 * i * (memslicesize +  + sizeof(size_t));

    // copy 2x channel data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset) ,
                        static_cast<void *>(G0.data.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 1 * memslicesize) ,
                        static_cast<void *>(G1.data.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    // copy noOf bit set data
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType)),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 2 * _nchans * sizeof(IntegratedPowerType) + sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
  }
}


void FullStokesOutput::swap(hipStream_t &_proc_stream)
{
    I.swap();
    Q.swap();
    U.swap();
    V.swap();
    _noOfBitSets.swap();
    thrust::fill(thrust::cuda::par.on(_proc_stream), I.a().begin(), I.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), Q.a().begin(), Q.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), U.a().begin(), U.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), V.a().begin(), V.a().end(), 0.);
    thrust::fill(thrust::cuda::par.on(_proc_stream), _noOfBitSets.a().begin(), _noOfBitSets.a().end(), 0L);
}


FullStokesOutput::FullStokesOutput(size_t size, size_t blocks)
{
    I.resize(size * blocks);
    Q.resize(size * blocks);
    U.resize(size * blocks);
    V.resize(size * blocks);
    _noOfBitSets.resize(blocks);
}



GatedFullStokesOutput::GatedFullStokesOutput(size_t nchans, size_t blocks): OutputDataStream(nchans, blocks), G0(nchans, blocks),
G1(nchans, blocks)
{
    BOOST_LOG_TRIVIAL(debug) << "Output with " << _blocks << " blocks a " << _nchans << " channels";
    _host_power.resize( 8 * ( _nchans * sizeof(IntegratedPowerType) + sizeof(size_t) ) * _blocks);
    BOOST_LOG_TRIVIAL(debug) << "Allocated " << _host_power.size() << " bytes.";
};


void GatedFullStokesOutput::swap(hipStream_t &_proc_stream)
{
    G0.swap(_proc_stream);
    G1.swap(_proc_stream);
    _host_power.swap();
}


void GatedFullStokesOutput::data2Host(hipStream_t &_d2h_stream)
{
for (size_t i = 0; i < G0._noOfBitSets.size(); i++)
{
    size_t memslicesize = (_nchans * sizeof(IntegratedPowerType));
    size_t memOffset = 8 * i * (memslicesize + sizeof(size_t));
    // Copy  II QQ UU VV
    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset) ,
                        static_cast<void *>(G0.I.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 1 * memslicesize) ,
                        static_cast<void *>(G1.I.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 2 * memslicesize) ,
                        static_cast<void *>(G0.Q.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 3 * memslicesize) ,
                        static_cast<void *>(G1.Q.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 4 * memslicesize) ,
                        static_cast<void *>(G0.U.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 5 * memslicesize) ,
                        static_cast<void *>(G1.U.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 6 * memslicesize) ,
                        static_cast<void *>(G0.V.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    CUDA_ERROR_CHECK(
        hipMemcpyAsync(static_cast<void *>(_host_power.a_ptr() + memOffset + 7 * memslicesize) ,
                        static_cast<void *>(G1.V.b_ptr() + i * _nchans),
                        _nchans * sizeof(IntegratedPowerType),
                        hipMemcpyDeviceToHost, _d2h_stream));

    // Copy SCI
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 1 * sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 2 * sizeof(size_t)),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 3 * sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 4 * sizeof(size_t)),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 5 * sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 6 * sizeof(size_t)),
          static_cast<void *>(G0._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
    CUDA_ERROR_CHECK(
        hipMemcpyAsync( static_cast<void *>(_host_power.a_ptr() + memOffset + 8 * memslicesize + 7 * sizeof(size_t)),
          static_cast<void *>(G1._noOfBitSets.b_ptr() + i ),
            1 * sizeof(size_t),
            hipMemcpyDeviceToHost, _d2h_stream));
  }
}


}}} // namespace
