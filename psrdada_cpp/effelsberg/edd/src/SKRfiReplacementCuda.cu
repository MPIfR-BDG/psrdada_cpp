#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/SKRfiReplacementCuda.cuh"

namespace psrdada_cpp {
namespace effelsberg {
namespace edd {

struct get_real{
    __host__ __device__
    float operator() (thrust::complex<float> val) const{
        return val.real();
    }
};

struct get_imag{
    __host__ __device__
    float operator() (thrust::complex<float> val) const{
        return val.imag();
    }
};

struct mean_subtraction_square{
    const float mean;
    mean_subtraction_square(float _mean) :mean(_mean) {}
    __host__ __device__
    float operator() (float val) const{
        return ((val - mean) * (val - mean));
    }
};

SKRfiReplacementCuda::SKRfiReplacementCuda(const thrust::device_vector<int> &rfi_status)
    :  _rfi_status(rfi_status)
{
    BOOST_LOG_TRIVIAL(info) << "Creating new SKRfiReplacementCuda instance..\n";
}

SKRfiReplacementCuda::~SKRfiReplacementCuda()
{
    BOOST_LOG_TRIVIAL(info) << "Destroying SKRfiReplacementCuda instance..\n";
}

void SKRfiReplacementCuda::init()
{
    BOOST_LOG_TRIVIAL(info) << "initializing data_members of SKRfiReplacementCuda class..\n";
    _nwindows = _rfi_status.size();
    _rfi_window_indices.reserve(_nwindows);
    get_rfi_window_indices();
    _clean_window_indices.reserve(_nwindows);
    get_clean_window_indices();
}

void SKRfiReplacementCuda::get_rfi_window_indices()
{
    _nrfi_windows = thrust::count(_rfi_status.begin(), _rfi_status.end(), 1);
    _rfi_window_indices.resize(_nrfi_windows);
    std::size_t iter = 0;
    for(std::size_t index = 0; index < _nrfi_windows; index++){
        _rfi_window_indices[index] = thrust::distance(_rfi_status.begin(), 
                                     thrust::max_element((_rfi_status.begin() + iter), _rfi_status.end()));
        iter = _rfi_window_indices[index] + 1;
    }
}

void SKRfiReplacementCuda::get_clean_window_indices()
{
    _nclean_windows = thrust::count(_rfi_status.begin(), _rfi_status.end(), 0);
    _clean_window_indices.resize(DEFAULT_NUM_CLEAN_WINDOWS);
    std::size_t iter = 0;
    for(std::size_t index = 0; index < DEFAULT_NUM_CLEAN_WINDOWS; index++){
        _clean_window_indices[index] = thrust::distance(_rfi_status.begin(), 
                                       thrust::min_element((_rfi_status.begin() + iter), _rfi_status.end()));
        iter = _clean_window_indices[index] + 1;
    }
}

void SKRfiReplacementCuda::get_clean_data_statistics(const thrust::device_vector<thrust::complex<float>> &data,
                                                     DataStatistics &ref_data_statistics)
{
    _window_size = data.size() / _nwindows;
    thrust::device_vector<thrust::complex<float>> clean_data(DEFAULT_NUM_CLEAN_WINDOWS * _window_size);
    for(std::size_t ii = 0; ii < DEFAULT_NUM_CLEAN_WINDOWS; ii++){
        std::size_t window_index = _clean_window_indices[ii];
        std::size_t ibegin = window_index * _window_size;
        std::size_t iend = ibegin + _window_size - 1;
        std::size_t jj = ii * _window_size;
        std::copy((data.begin() + ibegin), (data.begin() + iend), (clean_data.begin() + jj));
        BOOST_LOG_TRIVIAL(debug) <<"clean_win_index = " << window_index
                                 << " ibegin = " << ibegin << " iend = " << iend;
    }
    compute_data_statistics(clean_data, ref_data_statistics);
}

void SKRfiReplacementCuda::compute_data_statistics(const thrust::device_vector<thrust::complex<float>> &data, 
                                                   DataStatistics &stats)
{
    std::size_t length = data.size();
    thrust::device_vector<float> d_vreal(length), d_vimag(length);
    thrust::transform(data.begin(), data.end(), d_vreal.begin(), get_real());
    thrust::transform(data.begin(), data.end(), d_vimag.begin(), get_imag());
    stats.r_mean = thrust::reduce(d_vreal.begin(), d_vreal.end(), 0.0f) / length;
    stats.i_mean = thrust::reduce(d_vimag.begin(), d_vimag.end(), 0.0f) / length;
    stats.r_sd = std::sqrt(thrust::transform_reduce(d_vreal.begin(), d_vreal.end(), mean_subtraction_square(stats.r_mean),
                           0.0f, thrust::plus<float> ()) / length);
    stats.i_sd = std::sqrt(thrust::transform_reduce(d_vimag.begin(), d_vimag.end(), mean_subtraction_square(stats.i_mean),
                           0.0f, thrust::plus<float> ()) / length);
    BOOST_LOG_TRIVIAL(debug) << "DataStatistics r_mean = " << stats.r_mean
                             << " r_sd =  " << stats.r_sd
                             << " i_mean = " << stats.i_mean
                             << " i_sd = " << stats.i_sd;
}

void SKRfiReplacementCuda::generate_replacement_data(const DataStatistics &stats, 
                                                     thrust::device_vector<thrust::complex<float>> &replacement_data)
{
    BOOST_LOG_TRIVIAL(info) << "generating replacement data..\n";
    thrust::host_vector<thrust::complex<float>> h_replacement_data(_window_size);
    replacement_data = h_replacement_data;
    thrust::minstd_rand gen;
    thrust::random::normal_distribution<float> rdist(stats.r_mean, stats.r_sd);
    thrust::random::normal_distribution<float> idist(stats.i_mean, stats.i_sd);
    for(std::size_t index = 0; index < _window_size; index++){
        replacement_data[index] = thrust::complex<float>(rdist(gen), idist(gen));
    }
}

void SKRfiReplacementCuda::replace_rfi_data(thrust::device_vector<thrust::complex<float>> &data)
{
    DataStatistics stats;
    thrust::device_vector<thrust::complex<float>> replacement_data;
    //initialize data members of the class
    init();
    //RFI present and not in all windows
    if((_nrfi_windows > 0) && (_nrfi_windows < _nwindows)){
        get_clean_data_statistics(data, stats);
        generate_replacement_data(stats, replacement_data);
	//Replacing RFI
	for(std::size_t ii = 0; ii < _nrfi_windows; ii++){
            std::size_t index = _rfi_window_indices[ii] * _window_size;
	    thrust::copy(replacement_data.begin(), replacement_data.end(), (data.begin() +index));
        }
    }
}

} //edd
} //effelsberg
} //psrdada_cpp
