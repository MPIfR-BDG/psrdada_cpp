#include "hip/hip_runtime.h"
#include "psrdada_cpp/multilog.hpp"
#include "psrdada_cpp/cli_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/dada_client_base.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/dada_output_stream.hpp"
#include "psrdada_cpp/simple_file_writer.hpp"
#include "psrdada_cpp/effelsberg/edd/FftSpectrometer.cuh"
#include "psrdada_cpp/dada_null_sink.hpp"
#include "boost/program_options.hpp"
#include <time.h>
#include <ctime>

using namespace psrdada_cpp;

namespace
{
  const size_t ERROR_IN_COMMAND_LINE = 1;
  const size_t SUCCESS = 0;
  const size_t ERROR_UNHANDLED_EXCEPTION = 2;
} // namespace


int main(int argc, char** argv)
{
    try
    {
        key_t input_key;
        int fft_length;
        int nsamps_per_block;
        int naccumulate;
        int nbits;
        float input_level;
        std::time_t now = std::time(NULL);
        std::tm * ptm = std::localtime(&now);
        char buffer[32];
        std::strftime(buffer, 32, "%Y-%m-%d-%H:%M:%S.bp", ptm);
        std::string filename(buffer);

        /** Define and parse the program options
        */
        namespace po = boost::program_options;
        po::options_description desc("Options");
        desc.add_options()

        ("help,h", "Print help messages")
        ("input_key,i", po::value<std::string>()
            ->default_value("dada")
            ->notifier([&input_key](std::string in)
                {
                    input_key = string_to_key(in);
                }),
           "The shared memory key for the dada buffer to connect to (hex string)")

        ("fft_length,n", po::value<int>(&fft_length)->required(),
            "The length of the FFT to perform on the data")

        ("naccumulate,a", po::value<int>(&naccumulate)->required(),
            "The number of samples to integrate in each channel")

        ("nbits,b", po::value<int>(&nbits)->required(),
            "The number of bits per sample in the packetiser output (8 or 12)")

        ("input_level", po::value<float>(&input_level)->required(),
            "The input power level (standard deviation, used for 8-bit conversion)")

        ("offset", po::value<float>(&offset)->required(),
            "The power offset for data produced by the spectrometer (used for conversion back to 8-bit)")

        ("outfile,o", po::value<std::string>(&filename)
            ->default_value(filename),
            "The output file to write spectra to")

        ("log_level", po::value<std::string>()
            ->default_value("info")
            ->notifier([](std::string level)
                {
                    set_log_level(level);
                }),
            "The logging level to use (debug, info, warning, error)");

        po::variables_map vm;
        try
        {
            po::store(po::parse_command_line(argc, argv, desc), vm);
            if ( vm.count("help")  )
            {
                std::cout << "FftSpectrometer -- Read EDD data from a DADA buffer and perform a simple FFT spectrometer"
                << std::endl << desc << std::endl;
                return SUCCESS;
            }
            po::notify(vm);
        }
        catch(po::error& e)
        {
            std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
            std::cerr << desc << std::endl;
            return ERROR_IN_COMMAND_LINE;
        }
        /**
         * All the application code goes here
         */
        MultiLog log("edd::FftSpectrometer");
        DadaClientBase client(input_key, log);
        std::size_t buffer_bytes = client.data_buffer_size()
        SimpleFileWriter sink(filename);
        //NullSink sink;
        effelsberg::edd::FftSpectrometer<decltype(sink)> spectrometer(buffer_bytes, fft_length, naccumulate, nbits, input_level, sink);
        DadaInputStream<decltype(spectrometer)> istream(input_key, log, spectrometer);
        istream.start();
        /**
         * End of application code
         */
    }
    catch(std::exception& e)
    {
        std::cerr << "Unhandled Exception reached the top of main: "
        << e.what() << ", application will now exit" << std::endl;
        return ERROR_UNHANDLED_EXCEPTION;
    }
    return SUCCESS;

}