#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/edd/GatedSpectrometer.cuh"

#include "psrdada_cpp/dada_null_sink.hpp"
#include "psrdada_cpp/multilog.hpp"
#include "gtest/gtest.h"

#include "thrust/device_vector.h"
#include "thrust/extrema.h"


TEST(GatedSpectrometer, BitManipulationMacros) {
  for (int i = 0; i < 64; i++) {
    uint64_t v = 0;
    SET_BIT(v, i);

    for (int j = 0; j < 64; j++) {
      if (j == i)
        EXPECT_EQ(TEST_BIT(v, j), 1);
      else
        EXPECT_EQ(TEST_BIT(v, j), 0);
    }
  }
}

//
//TEST(GatedSpectrometer, stokes_IQUV)
//{
//    float I,Q,U,V;
//    // No field
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){0.0f,0.0f}, (float2){0.0f,0.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 0);
//    EXPECT_FLOAT_EQ(Q, 0);
//    EXPECT_FLOAT_EQ(U, 0);
//    EXPECT_FLOAT_EQ(V, 0);
//
//    // For p1 = Ex, p2 = Ey
//    // horizontal polarized
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){1.0f,0.0f}, (float2){0.0f,0.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, 1);
//    EXPECT_FLOAT_EQ(U, 0);
//    EXPECT_FLOAT_EQ(V, 0);
//
//    // vertical polarized
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){0.0f,0.0f}, (float2){1.0f,0.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, -1);
//    EXPECT_FLOAT_EQ(U, 0);
//    EXPECT_FLOAT_EQ(V, 0);
//
//    //linear +45 deg.
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){1.0f/std::sqrt(2),0.0f}, (float2){1.0f/std::sqrt(2),0.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, 0);
//    EXPECT_FLOAT_EQ(U, 1);
//    EXPECT_FLOAT_EQ(V, 0);
//
//    //linear -45 deg.
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){-1.0f/std::sqrt(2),0.0f}, (float2){1.0f/std::sqrt(2),0.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, 0);
//    EXPECT_FLOAT_EQ(U, -1);
//    EXPECT_FLOAT_EQ(V, 0);
//
//    //left circular
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){.0f,1.0f/std::sqrt(2)}, (float2){1.0f/std::sqrt(2),.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, 0);
//    EXPECT_FLOAT_EQ(U, 0);
//    EXPECT_FLOAT_EQ(V, -1);
//
//    // right circular
//    psrdada_cpp::effelsberg::edd::stokes_IQUV((float2){.0f,-1.0f/std::sqrt(2)}, (float2){1.0f/std::sqrt(2),.0f}, I, Q, U, V);
//    EXPECT_FLOAT_EQ(I, 1);
//    EXPECT_FLOAT_EQ(Q, 0);
//    EXPECT_FLOAT_EQ(U, 0);
//    EXPECT_FLOAT_EQ(V, 1);
//}
//
//
//TEST(GatedSpectrometer, stokes_accumulate)
//{
//    CUDA_ERROR_CHECK(hipDeviceSynchronize());
//    size_t nchans = 8 * 1024 * 1024 + 1;
//    size_t naccumulate = 5;
//
//    thrust::device_vector<float2> P0(nchans * naccumulate);
//    thrust::device_vector<float2> P1(nchans * naccumulate);
//    thrust::fill(P0.begin(), P0.end(), (float2){0, 0});
//    thrust::fill(P1.begin(), P1.end(), (float2){0, 0});
//    thrust::device_vector<float> I(nchans);
//    thrust::device_vector<float> Q(nchans);
//    thrust::device_vector<float> U(nchans);
//    thrust::device_vector<float> V(nchans);
//    thrust::fill(I.begin(), I.end(), 0);
//    thrust::fill(Q.begin(), Q.end(), 0);
//    thrust::fill(U.begin(), U.end(), 0);
//    thrust::fill(V.begin(), V.end(), 0);
//
//    // This channel should be left circular polarized
//    size_t idx0 = 23;
//    for (int k = 0; k< naccumulate; k++)
//    {
//        size_t idx = idx0 + k * nchans;
//        P0[idx] = (float2){0.0f, 1.0f/std::sqrt(2)};
//        P1[idx] = (float2){1.0f/std::sqrt(2),0.0f};
//    }
//
//    psrdada_cpp::effelsberg::edd::stokes_accumulate<<<1024, 1024>>>(
//          thrust::raw_pointer_cast(P0.data()),
//          thrust::raw_pointer_cast(P1.data()),
//          thrust::raw_pointer_cast(I.data()),
//          thrust::raw_pointer_cast(Q.data()),
//          thrust::raw_pointer_cast(U.data()),
//          thrust::raw_pointer_cast(V.data()),
//          nchans,
//          naccumulate
//            );
//
//    CUDA_ERROR_CHECK(hipDeviceSynchronize());
//    thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator> minmax;
//
//    minmax = thrust::minmax_element(I.begin(), I.end());
//    EXPECT_FLOAT_EQ(*minmax.first, 0);
//    EXPECT_FLOAT_EQ(*minmax.second, naccumulate);
//
//    minmax = thrust::minmax_element(Q.begin(), Q.end());
//    EXPECT_FLOAT_EQ(*minmax.first, 0);
//    EXPECT_FLOAT_EQ(*minmax.second, 0);
//
//    minmax = thrust::minmax_element(U.begin(), U.end());
//    EXPECT_FLOAT_EQ(*minmax.first, 0);
//    EXPECT_FLOAT_EQ(*minmax.second, 0);
//
//    minmax = thrust::minmax_element(V.begin(), V.end());
//    EXPECT_FLOAT_EQ(*minmax.first, -1. * naccumulate);
//    EXPECT_FLOAT_EQ(*minmax.second, 0);
//}
//


TEST(GatedSpectrometer, GatingKernel)
{
  const size_t blockSize = 1024;
  const size_t nBlocks = 16 * 1024;

  thrust::device_vector<float> G0(blockSize * nBlocks);
  thrust::device_vector<float> G1(blockSize * nBlocks);
  thrust::device_vector<uint64_t> _sideChannelData(nBlocks);
  thrust::device_vector<psrdada_cpp::effelsberg::edd::uint64_cu> _nG0(nBlocks);
  thrust::device_vector<psrdada_cpp::effelsberg::edd::uint64_cu> _nG1(nBlocks);
  thrust::device_vector<float> baseLineG0(1);
  thrust::device_vector<float> baseLineG1(1);

  thrust::device_vector<float> baseLineG0_update(1);
  thrust::device_vector<float> baseLineG1_update(1);
  thrust::fill(G0.begin(), G0.end(), 42);
  thrust::fill(G1.begin(), G1.end(), 23);
  thrust::fill(_sideChannelData.begin(), _sideChannelData.end(), 0);

  // everything to G0
  {
    thrust::fill(_nG0.begin(), _nG0.end(), 0);
    thrust::fill(_nG1.begin(), _nG1.end(), 0);
    baseLineG0[0] = -3;
    baseLineG1[0] = -4;
    baseLineG0_update[0] = 0;
    baseLineG1_update[0] = 0;

    const uint64_t *sideCD =
        (uint64_t *)(thrust::raw_pointer_cast(_sideChannelData.data()));
    psrdada_cpp::effelsberg::edd::gating<<<1024 , 1024>>>(
          thrust::raw_pointer_cast(G0.data()),
          thrust::raw_pointer_cast(G1.data()), sideCD,
          G0.size(), blockSize, 0, 1,
          0,
          thrust::raw_pointer_cast(baseLineG0.data()),
          thrust::raw_pointer_cast(baseLineG1.data()),
          thrust::raw_pointer_cast(baseLineG0_update.data()),
          thrust::raw_pointer_cast(baseLineG1_update.data()),
          thrust::raw_pointer_cast(_nG0.data()),
          thrust::raw_pointer_cast(_nG1.data())
          );

    thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator> minmax;
    minmax = thrust::minmax_element(G0.begin(), G0.end());
    EXPECT_EQ(*minmax.first, 42);
    EXPECT_EQ(*minmax.second, 42);

    minmax = thrust::minmax_element(G1.begin(), G1.end());
    EXPECT_EQ(*minmax.first, -4.);
    EXPECT_EQ(*minmax.second, -4.);

    EXPECT_EQ(_nG0[0], G0.size());
    EXPECT_EQ(_nG1[0], 0u);

    EXPECT_FLOAT_EQ(42.f, baseLineG0_update[0] / (_nG0[0] + 1E-121));
    EXPECT_FLOAT_EQ(0.f, baseLineG1_update[0] / (_nG1[0] + 1E-121));
  }

  // everything to G1 // with baseline -5
  {
    thrust::fill(_nG0.begin(), _nG0.end(), 0);
    thrust::fill(_nG1.begin(), _nG1.end(), 0);
    baseLineG0[0] = 5.;
    baseLineG1[0] = -2;
    baseLineG0_update[0] = 0;
    baseLineG1_update[0] = 0;

    thrust::fill(_sideChannelData.begin(), _sideChannelData.end(), 1L);
    const uint64_t *sideCD =
        (uint64_t *)(thrust::raw_pointer_cast(_sideChannelData.data()));
    psrdada_cpp::effelsberg::edd::gating<<<1024, 1024>>>(
          thrust::raw_pointer_cast(G0.data()),
          thrust::raw_pointer_cast(G1.data()), sideCD,
          G0.size(), blockSize, 0, 1,
          0,
          thrust::raw_pointer_cast(baseLineG0.data()),
          thrust::raw_pointer_cast(baseLineG1.data()),
          thrust::raw_pointer_cast(baseLineG0_update.data()),
          thrust::raw_pointer_cast(baseLineG1_update.data()),
          thrust::raw_pointer_cast(_nG0.data()),
          thrust::raw_pointer_cast(_nG1.data())
          );
    thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator> minmax;
    minmax = thrust::minmax_element(G0.begin(), G0.end());
    EXPECT_EQ(*minmax.first, 5.);
    EXPECT_EQ(*minmax.second, 5.);

    minmax = thrust::minmax_element(G1.begin(), G1.end());
    EXPECT_EQ(*minmax.first, 42);
    EXPECT_EQ(*minmax.second, 42);

    EXPECT_EQ(_nG0[0], 0u);
    EXPECT_EQ(_nG1[0], G1.size());

    EXPECT_FLOAT_EQ(0.f, baseLineG0_update[0] / (_nG0[0] + 1E-121));
    EXPECT_FLOAT_EQ(42.f, baseLineG1_update[0] / (_nG1[0] + 1E-121));
  }
}

TEST(GatedSpectrometer, array_sum) {

  const size_t NBLOCKS = 16 * 32;
  const size_t NTHREADS = 1024;

  size_t inputLength = 1 << 22 + 1 ;
  size_t dataLength = inputLength;
  ////zero pad input array
  //if (inputLength % (2 * NTHREADS) != 0)
  //  dataLength = (inputLength / (2 * NTHREADS) + 1) * 2 * NTHREADS;
  thrust::device_vector<float> data(dataLength);
  thrust::fill(data.begin(), data.begin() + inputLength, 1);
  //thrust::fill(data.begin() + inputLength, data.end(), 0);
  thrust::device_vector<float> blr(NTHREADS * 2);

  thrust::fill(blr.begin(), blr.end(), 0);

  psrdada_cpp::effelsberg::edd::array_sum<<<NBLOCKS, NTHREADS, NTHREADS* sizeof(float)>>>(thrust::raw_pointer_cast(data.data()), data.size(), thrust::raw_pointer_cast(blr.data()));
  psrdada_cpp::effelsberg::edd::array_sum<<<1, NTHREADS, NTHREADS* sizeof(float)>>>(thrust::raw_pointer_cast(blr.data()), blr.size(), thrust::raw_pointer_cast(blr.data()));

  EXPECT_EQ(size_t(blr[0]), inputLength);
}
