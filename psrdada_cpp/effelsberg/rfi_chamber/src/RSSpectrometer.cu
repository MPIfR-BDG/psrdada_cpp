#include "hip/hip_runtime.h"
#include "psrdada_cpp/effelsberg/rfi_chamber/RSSpectrometer.cuh"
#include "psrdada_cpp/cuda_utils.hpp"
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/fill.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <cassert>
#include <fstream>
#include <iomanip>

#define PASSTHROUGH_MODE_IQ_SCALING 1.0f/(1<<15)
#define PFB_MODE_NSHIFTS 3
#define PFB_MODE_IQ_SCALING 1.0f/(1<<(30 - PFB_MODE_NSHIFTS))
#define FSW_IMPEDANCE 50.0f

namespace psrdada_cpp {
namespace effelsberg {
namespace rfi_chamber {
namespace kernels {

struct short2_be_to_float2_le
    : public thrust::unary_function<short2, float2>
{
    __host__ __device__
    float2 operator()(short2 in)
    {
        char4 swap;
        char4* in_ptr = (char4*)(&in);
        swap.x = in_ptr->y;
        swap.y = in_ptr->x;
        swap.z = in_ptr->w;
        swap.w = in_ptr->z;
        short2* swap_as_short2 = (short2*)(&swap);
        float2 out;
        out.x = (float) swap_as_short2->x;
        out.y = (float) swap_as_short2->y;
        return out;
    }
};

struct detect_scale
    : public thrust::unary_function<float2, float>
{
    detect_scale(float scale_factor=1)
    : _scale_factor(scale_factor){}

    __host__ __device__
    float operator()(float2 voltage)
    {
        float x = voltage.x * _scale_factor;
        float y = voltage.y * _scale_factor;
        float power = x * x + y * y;
        return power;
    }

    const float _scale_factor;
};

struct detect_magnitude
    : public thrust::unary_function<float2, float>
{
    detect_magnitude(float scale_factor=1)
    : _scale_factor(scale_factor){}

    __device__
    float operator()(float2 voltage)
    {
        float x = voltage.x * _scale_factor;
        float y = voltage.y * _scale_factor;
        float power = x * x + y * y;
        return sqrtf(power);
    }

    const float _scale_factor;
};

struct detect_accumulate
    : public thrust::binary_function<float2, float, float>
{
    detect_accumulate(float scale_factor=1)
    : _scale_factor(scale_factor){}

    __host__ __device__
    float operator()(float2 voltage, float power_accumulator)
    {
        float x = voltage.x * _scale_factor;
        float y = voltage.y * _scale_factor;
        float power = x * x + y * y;
        return power_accumulator + power;
    }

    const float _scale_factor;
};

struct convert_to_dBm
    : public thrust::unary_function<float, float>
{
    convert_to_dBm(float scale_factor=1, float offset=0)
    : _scale_factor(scale_factor)
    , _offset(offset){}

    __device__
    float operator()(float power)
    {
        // Typically _scale_factor here is 1000.0 / (50.0 * naccumulate);
        return 10 * __log10f(power * _scale_factor) + _offset;
    }

    const float _scale_factor;
    const float _offset;
};

} // namespace kernels

// dense histogram using binary search
void histogram(const thrust::device_vector<float2>& input,
    thrust::device_vector<int>& d_hist,
    float min_val,
    float max_val,
    std::size_t nbins)
{
    // sort data to bring equal elements together
    thrust::device_vector<float> magnitudes(input.size());
    thrust::transform(input.begin(), input.end(), magnitudes.begin(),
        kernels::detect_magnitude(PASSTHROUGH_MODE_IQ_SCALING));
    thrust::sort(magnitudes.begin(), magnitudes.end());
    thrust::device_vector<float> bins(nbins);
    float step = (max_val - min_val) / nbins;
    thrust::sequence(bins.begin(), bins.end(), min_val, step);
    // resize histogram storage
    d_hist.resize(nbins);
    // find the end of each bin of values
    thrust::upper_bound(magnitudes.begin(), magnitudes.end(),
                        bins.begin(), bins.end(),
                        d_hist.begin());
    // compute the histogram by taking differences of the cumulative histogram
    thrust::adjacent_difference(d_hist.begin(), d_hist.end(),
                                d_hist.begin());
}

RSSpectrometer::RSSpectrometer(
    std::size_t input_nchans, std::size_t fft_length,
    std::size_t naccumulate, std::size_t nskip,
    std::string filename, float reference_dbm)
    : _input_nchans(input_nchans)
    , _fft_length(fft_length)
    , _naccumulate(naccumulate)
    , _nskip(nskip)
    , _filename(filename)
    , _reference_dbm(reference_dbm)
    , _output_nchans(_fft_length * _input_nchans)
    , _bytes_per_input_spectrum(_input_nchans * sizeof(InputType))
    , _naccumulated(0)
{

    BOOST_LOG_TRIVIAL(info) << "Initialising RSSpectrometer";
    BOOST_LOG_TRIVIAL(info) << "Number of input channels: " << _input_nchans;
    BOOST_LOG_TRIVIAL(info) << "FFT length: " << _fft_length;
    BOOST_LOG_TRIVIAL(info) << "Number of spectra to accumulate: " << _naccumulate;
    BOOST_LOG_TRIVIAL(info) << "Number of DADA blocks to skip: " << _nskip;
    BOOST_LOG_TRIVIAL(info) << "Number of output channels: " << _output_nchans;

    std::size_t total_mem, free_mem;
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Total GPU memory: " << total_mem << " bytes";
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory: " << free_mem << " bytes";

    // Memory required for accumulation buffer
    std::size_t accumulator_buffer_bytes = _output_nchans * sizeof(OutputType);
    BOOST_LOG_TRIVIAL(debug) << "Memory required for accumulator buffer: " << accumulator_buffer_bytes << " bytes";
    if (accumulator_buffer_bytes > free_mem)
    {
        throw std::runtime_error("The requested FFT length exceeds the free GPU memory");
    }
    std::size_t mem_budget = static_cast<std::size_t>((free_mem - accumulator_buffer_bytes) * 0.8) ; // Make only 80% of memory available
    BOOST_LOG_TRIVIAL(debug) << "Memory budget: " << mem_budget << " bytes";
    // Memory required per input channel
    std::size_t mem_per_input_channel = (_fft_length *  (sizeof(FftType) * 2 + 2 * sizeof(InputType)));
    BOOST_LOG_TRIVIAL(debug) << "Memory required per input channel: " << mem_per_input_channel << " bytes";
    _chans_per_copy = min(_input_nchans, mem_budget / mem_per_input_channel);
    if (mem_per_input_channel > mem_budget)
    {
	 throw std::runtime_error("The requested FFT length exceeds the free GPU memory");
    }
    BOOST_LOG_TRIVIAL(debug) << "Max possible Nchans per copy: " << mem_budget / mem_per_input_channel;
    while (_input_nchans % _chans_per_copy != 0)
    {
        _chans_per_copy -= 1;
    }
    assert(_chans_per_copy > 0); /** Must be able to process at least 1 channel */
    BOOST_LOG_TRIVIAL(debug) << "Nchannels per GPU transfer: " << _chans_per_copy;
    mem_budget -= _chans_per_copy * mem_per_input_channel;
    BOOST_LOG_TRIVIAL(debug) << "Remaining memory budget: " << mem_budget << " bytes";

    // Resize all buffers.
    BOOST_LOG_TRIVIAL(debug) << "Resizing all memory buffers";
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory: " << free_mem << " bytes";
    _accumulation_buffer.resize(_output_nchans, 0.0f);
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory after acc buffer: " << free_mem << " bytes";
    _h_accumulation_buffer.resize(_output_nchans, 0.0f);
    BOOST_LOG_TRIVIAL(debug) << "Allocating " << _chans_per_copy * _fft_length * 8  * 2 << " bytes for FFT buffers";
    _fft_input_buffer.resize(_chans_per_copy * _fft_length);
    _fft_output_buffer.resize(_chans_per_copy * _fft_length);
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory after FFT buffer: " << free_mem << " bytes";
    _copy_buffer.resize(_chans_per_copy * _fft_length);
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory after copy buffer: " << free_mem << " bytes";

    // Allocate streams
    BOOST_LOG_TRIVIAL(debug) << "Allocating CUDA streams";
    CUDA_ERROR_CHECK(hipStreamCreate(&_copy_stream));
    CUDA_ERROR_CHECK(hipStreamCreate(&_proc_stream));

    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory pre-cufft: " << free_mem << " bytes";
    // Configure CUFFT for FFT execution

    BOOST_LOG_TRIVIAL(debug) << "Generating CUFFT plan";
    int n[] = {static_cast<int>(_fft_length)};
    int inembed[] = {static_cast<int>(_chans_per_copy)};
    int onembed[] = {static_cast<int>(_fft_length)};
    CUFFT_ERROR_CHECK(hipfftPlanMany(&_fft_plan, 1, n, inembed, _chans_per_copy, 1, onembed, 1, _fft_length,
        HIPFFT_C2C, _chans_per_copy));

    BOOST_LOG_TRIVIAL(debug) << "Setting CUFFT stream";
    CUFFT_ERROR_CHECK(hipfftSetStream(_fft_plan, _proc_stream));
    CUDA_ERROR_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    BOOST_LOG_TRIVIAL(debug) << "Free GPU memory post-cufft: " << free_mem << " bytes";
    BOOST_LOG_TRIVIAL(debug) << "RSSpectrometer instance initialised";
}

RSSpectrometer::~RSSpectrometer()
{
    BOOST_LOG_TRIVIAL(debug) << "Destroying RSSpectrometer instance";
    BOOST_LOG_TRIVIAL(debug) << "Destroying CUDA streams";
    CUDA_ERROR_CHECK(hipStreamDestroy(_copy_stream));
    CUDA_ERROR_CHECK(hipStreamDestroy(_proc_stream));
    BOOST_LOG_TRIVIAL(debug) << "Destroying CUFFT plan";
    CUFFT_ERROR_CHECK(hipfftDestroy(_fft_plan));
    BOOST_LOG_TRIVIAL(info) << "RSSpectrometer destroyed";
}

void RSSpectrometer::init(RawBytes &header)
{
    BOOST_LOG_TRIVIAL(debug) << "RSSpectrometer received header block";
}

bool RSSpectrometer::operator()(RawBytes &block)
{
    BOOST_LOG_TRIVIAL(debug) << "RSSpectrometer received data block";
    if (_nskip > 0)
    {
        BOOST_LOG_TRIVIAL(debug) << "Skipping block while stream stabilizes";
        --_nskip;
        return false;
    }
    assert(block.used_bytes() % _bytes_per_input_spectrum == 0); /** Block is not a multiple of the heap group size */
    std::size_t nspectra_in = block.used_bytes() / _bytes_per_input_spectrum;
    BOOST_LOG_TRIVIAL(debug) << "Number of input spectra per block: " << nspectra_in;
    assert(block.used_bytes() % _output_nchans * sizeof(InputType) == 0); /** Block is not a multiple of the spectrum size */
    std::size_t nspectra_out = block.used_bytes() / (_output_nchans * sizeof(InputType));
    BOOST_LOG_TRIVIAL(debug) << "Number of output spectra per block: " << nspectra_out;

    std::size_t n_to_accumulate;
    if (nspectra_out > _naccumulate)
    {
        n_to_accumulate = _naccumulate;
    }
    else
    {
        n_to_accumulate = nspectra_out;
    }
    BOOST_LOG_TRIVIAL(debug) << "Number of spectra to accumulate in current block: " << n_to_accumulate;
    BOOST_LOG_TRIVIAL(debug) << "Entering processing loop";
    std::size_t nchan_blocks = _input_nchans / _chans_per_copy;
    for (std::size_t spec_idx = 0; spec_idx < n_to_accumulate; ++spec_idx)
    {
        copy(block, spec_idx, 0, nspectra_in);
        for (std::size_t chan_block_idx = 1;
            chan_block_idx < nchan_blocks;
            ++chan_block_idx)
        {
            copy(block, spec_idx, chan_block_idx, nspectra_in);
            process(chan_block_idx - 1);
        }
        CUDA_ERROR_CHECK(hipStreamSynchronize(_copy_stream));
        _copy_buffer.swap();
        process(nchan_blocks - 1);
    }
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
    BOOST_LOG_TRIVIAL(debug) << "Processing loop complete";
    _naccumulate -= n_to_accumulate;
    _naccumulated += n_to_accumulate;
    BOOST_LOG_TRIVIAL(info) << "Accumulated " << n_to_accumulate
    << " spectra ("<< _naccumulate << " remaining)";
    if (_naccumulate == 0)
    {
        BOOST_LOG_TRIVIAL(debug) << "Processing loop complete";
        // Here we need to do the final scaling and conversion
        thrust::transform(_accumulation_buffer.begin(), _accumulation_buffer.end(),
            _accumulation_buffer.begin(),
            kernels::convert_to_dBm(1000.0f / (FSW_IMPEDANCE * _naccumulated), 0));
        write_spectrum();
        // Free up some memory for histogram calculation
        _fft_output_buffer.resize(0);

        // Here we can calculate the histogram of the last block
        thrust::device_vector<int> d_hist;
        histogram(_fft_input_buffer, d_hist, 0.0, 2.0, 1024);
        write_histogram(d_hist);

        return true;
    }
    return false;
}

void RSSpectrometer::process(std::size_t chan_block_idx)
{
    /** Note streams do not actually work as expected
     *  with Thrust. The code is synchronous with respect
     *  to the host. The Thrust 1.9.4 (CUDA 10.1) release
     *  includes thrust::async which alleviates this problem.
     *  This can be included here if need be, but as it is the
     *  H2D copy should still run in parallel to the FFT, so
     *  there is no performance cost to blocking on the host.
     */
    CUDA_ERROR_CHECK(hipStreamSynchronize(_proc_stream));
    // Convert shorts to floats
    BOOST_LOG_TRIVIAL(debug) << "Performing short2 to float2 conversion";

    thrust::transform(
        thrust::cuda::par.on(_proc_stream),
        _copy_buffer.b().begin(),
        _copy_buffer.b().end(),
        _fft_input_buffer.begin(),
        kernels::short2_be_to_float2_le());

    float scale_factor;
    if (_input_nchans == 1)
    {
        scale_factor = PASSTHROUGH_MODE_IQ_SCALING * sqrtf( powf(10.0f, 
				(_reference_dbm - 30.0f) / 10.0f) * 50.0f);
    }
    else if (_input_nchans == (1<<15))
    {
        scale_factor = PFB_MODE_IQ_SCALING;
    }
    else
    {
        BOOST_LOG_TRIVIAL(warning) << "No IQ scale factor known for " << _input_nchans << " channel input";
        scale_factor = 1.0f;
    }

    // Calculate RMS of data
    /*
    float sum = thrust::transform_reduce(
        thrust::cuda::par.on(_proc_stream),
        _fft_input_buffer.begin(),
        _fft_input_buffer.end(),
        kernels::detect_scale(scale_factor),
        0.0f,
        thrust::plus<float>());
    float rms = sqrtf(sum / _fft_input_buffer.size());
    BOOST_LOG_TRIVIAL(debug) << "RMS voltage of IQ data: " << rms << " V";
    */
    // Perform forward C2C transform
    BOOST_LOG_TRIVIAL(debug) << "Executing FFT";
    hipfftComplex* in_ptr = static_cast<hipfftComplex*>(
        thrust::raw_pointer_cast(_fft_input_buffer.data()));
    hipfftComplex* out_ptr = static_cast<hipfftComplex*>(
        thrust::raw_pointer_cast(_fft_output_buffer.data()));
    CUFFT_ERROR_CHECK(hipfftExecC2C(
        _fft_plan, in_ptr, out_ptr, HIPFFT_FORWARD));
    std::size_t chan_offset = chan_block_idx * _chans_per_copy * _fft_length;
    // Detect FFT output and accumulate
    BOOST_LOG_TRIVIAL(debug) << "Detecting and accumulating";

    thrust::transform(
        thrust::cuda::par.on(_proc_stream),
        _fft_output_buffer.begin(),
        _fft_output_buffer.end(),
        _accumulation_buffer.begin() + chan_offset,
        _accumulation_buffer.begin() + chan_offset,
        kernels::detect_accumulate(scale_factor/_fft_length));

}

void RSSpectrometer::copy(RawBytes& block, std::size_t spec_idx, std::size_t chan_block_idx, std::size_t nspectra_in)
{
    BOOST_LOG_TRIVIAL(debug) << "Copying block to GPU";
    std::size_t spitch = _input_nchans * sizeof(short2); // Width of a row in bytes (so number of channels total)
    std::size_t width = _chans_per_copy * sizeof(short2);; // Total number of samples in the input
    std::size_t dpitch = _chans_per_copy * sizeof(short2); // Width of row in bytes in the output
    std::size_t height = _fft_length; // Total number of samples to copy
    CUDA_ERROR_CHECK(hipStreamSynchronize(_copy_stream));
    _copy_buffer.swap();

    if (_input_nchans != 1)
    {
        char* src = block.ptr() + spec_idx * spitch * height + chan_block_idx * width;
        BOOST_LOG_TRIVIAL(debug) << "Calling hipMemcpy2DAsync with args: "
    	    << "dest=" << _copy_buffer.a_ptr() << ", "
    	    << "dpitch=" << dpitch << ", "
    	    << "src=" << (void*) src << ", "
    	    << "spitch=" << spitch << ", "
    	    << "width=" << width << ", "
    	    << "height=" << height << ", "
    	    << hipMemcpyHostToDevice << ", "
    	    << _copy_stream;
        CUDA_ERROR_CHECK(hipMemcpy2DAsync(_copy_buffer.a_ptr(),
            dpitch, src, spitch, width, height,
            hipMemcpyHostToDevice, _copy_stream));
    }
    else
    {
        std::size_t nbytes = _fft_length * sizeof(short2);
        char* src = block.ptr() + spec_idx * nbytes;
        CUDA_ERROR_CHECK(hipMemcpyAsync(_copy_buffer.a_ptr(), src, nbytes,
            hipMemcpyHostToDevice, _copy_stream));
    }
}

void RSSpectrometer::write_histogram(thrust::device_vector<int> const& histogram)
{
    // Copy histogeam buffer to host
    BOOST_LOG_TRIVIAL(debug) << "Copying histogram to host";
    thrust::host_vector<int> h_hist = histogram;
    BOOST_LOG_TRIVIAL(debug) << "Perparing output file";
    std::ofstream outfile;
    std::string _hist_filename(_filename + ".hist");
    outfile.open(_hist_filename.c_str(),std::ifstream::out | std::ifstream::binary);
    if (outfile.is_open())
    {
        BOOST_LOG_TRIVIAL(debug) << "Opened file " << _hist_filename;
    }
    else
    {
        std::stringstream stream;
        stream << "Could not open file " << _hist_filename;
        throw std::runtime_error(stream.str().c_str());
    }
    outfile.write((char*)h_hist.data(), h_hist.size() * sizeof(int));
    outfile.flush();
    outfile.close();
}

void RSSpectrometer::write_spectrum()
{
    // Copy accumulation buffer to host
    BOOST_LOG_TRIVIAL(debug) << "Copying accumulated spectrum to host";
    _h_accumulation_buffer = _accumulation_buffer;
    BOOST_LOG_TRIVIAL(debug) << "Perparing output file";
    std::ofstream outfile;
    outfile.open(_filename.c_str(),std::ifstream::out | std::ifstream::binary);
    if (outfile.is_open())
    {
        BOOST_LOG_TRIVIAL(debug) << "Opened file " << _filename;
    }
    else
    {
        std::stringstream stream;
        stream << "Could not open file " << _filename;
        throw std::runtime_error(stream.str().c_str());
    }
    BOOST_LOG_TRIVIAL(info) << "Writing output to " << _filename << " with FFT shifts included";
    // Here we are now doing a double FFT shift
    // We must first shift the contents of every coarse channel
    // The we write out the full spectrum with a shift
    // First write second half of the band
    std::size_t nsubbands = _h_accumulation_buffer.size() / _fft_length;
    for (std::size_t subband_idx=nsubbands/2; subband_idx < nsubbands; ++subband_idx)
    {
        std::size_t offset = subband_idx * _fft_length;
        //First write upper half of the band
        char* back = reinterpret_cast<char*>(&_h_accumulation_buffer[offset + _fft_length/2]);
        char* front = reinterpret_cast<char*>(&_h_accumulation_buffer[offset]);
        outfile.write(back, (_fft_length/2) * sizeof(decltype(_h_accumulation_buffer)::value_type));
        outfile.write(front, (_fft_length/2) * sizeof(decltype(_h_accumulation_buffer)::value_type));
    }
    // Second write out the first half of the band
    for (std::size_t subband_idx=0; subband_idx < nsubbands/2; ++subband_idx)
    {
        std::size_t offset = subband_idx * _fft_length;
        //First write upper half of the band
        char* back = reinterpret_cast<char*>(&_h_accumulation_buffer[offset + _fft_length/2]);
        char* front = reinterpret_cast<char*>(&_h_accumulation_buffer[offset]);
        outfile.write(back, (_fft_length/2) * sizeof(decltype(_h_accumulation_buffer)::value_type));
        outfile.write(front, (_fft_length/2) * sizeof(decltype(_h_accumulation_buffer)::value_type));
    }
    outfile.flush();
    outfile.close();
    BOOST_LOG_TRIVIAL(debug) << "File write complete";
}


} //namespace rfi_chamber
} //namespace effelsberg
} //namespace psrdada_cpp

