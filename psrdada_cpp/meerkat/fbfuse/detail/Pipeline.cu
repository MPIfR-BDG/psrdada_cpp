#include "hip/hip_runtime.h"
#include "ascii_header.h"
#include <stdexcept>
#include <exception>
#include <cstdlib>

#define SAMPLE_CLOCK_START_KEY "SAMPLE_CLOCK_START"

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {


Pipeline::Pipeline(PipelineConfig const& config,
    DadaWriteClient& cb_writer,
    DadaWriteClient& ib_writer)
    : _config(config)
    , _start_epoch(0)
    , _call_count(0)
    , _cb_writer(cb_writer),
    , _cb_header_stream(cb_writer.header_stream())
    , _cb_data_stream(cb_writer.data_stream())
    , _ib_writer(ib_writer),
    , _ib_header_stream(ib_writer.header_stream())
    , _ib_data_stream(ib_writer.data_stream())
{
    CUDA_SAFE_CALL(hipStreamCreate(&_h2d_copy_stream));
    CUDA_SAFE_CALL(hipStreamCreate(&_processing_stream));
    CUDA_SAFE_CALL(hipStreamCreate(&_d2h_copy_stream));
}

Pipeline::~Pipeline()
{
    try
    {
        _cb_data_stream.release();
        _ib_data_stream.release();
    }
    catch (std::exception& e)
    {
        BOOST_LOG_TRIVIAL(warn) << "Non-fatal error on pipeline destruction: "
        << e.what();
    }
    CUDA_SAFE_CALL(hipStreamDestroy(_h2d_copy_stream));
    CUDA_SAFE_CALL(hipStreamDestroy(_processing_stream));
    CUDA_SAFE_CALL(hipStreamDestroy(_d2h_copy_stream));
}

void Pipeline::set_header(RawBlock& header)
{
    std::memset(static_cast<void*>(header.ptr()), 0, header.total_bytes());
    header.used_bytes(header.total_bytes());
    ascii_header_set(header.ptr(), SAMPLE_CLOCK_START_KEY, "%ul", _start_epoch);
}

void Pipeline::init(RawBlock& header)
{
    BOOST_LOG_TRIVIAL(debug) << "Parsing DADA header";
    // Extract the time from the header and convert it to a double epoch
    char tmp[32];
    if (ascii_header_get(header.ptr(), SAMPLE_CLOCK_START_KEY, "%s", tmp) == -1)
    {
        throw std::runtime_error("Could not fine SAMPLE_CLOCK_START key in DADA header.")
    }
    _start_epoch = std::strtoul(tmp, NULL, 0);
    BOOST_LOG_TRIVIAL(info) << SAMPLE_CLOCK_START_KEY << " = " << _start_epoch;

    // Need to set the header information on the coherent beam output block
    auto& cb_header_block = _cb_header_stream.next();
    set_header(cb_header_block);
    _cb_header_stream.release();

    // Need to set the header information on the incoherent beam output block
    auto& ib_header_block = _ib_header_stream.next();
    set_header(ib_header_block);
    _ib_header_stream.release();
}

void Pipeline::process(char2* taftp_ptr, char* tbftf_ptr, char* tftf_ptr)
{

}

bool Pipeline::operator()(RawBlock& data)
{
    ++_call_count;
    if (_call_count == 1)
    {
        // Here we should check the size of all the input and output
        // and throw an error on incorrect buffer sizes.
        //
        // Input buffer checks:
        //
        std::size_t heap_group_size = (FBFUSE_TOTAL_ANTENNAS * FBFUSE_NCHANS
            * FBFUSE_NSAMPLES_PER_HEAP * FBFUSE_NPOL) * sizeof(char2);
        if (data.used_bytes() % heap_group_size != 0)
        {
            throw std::runtime_error("Input DADA buffer is not a multiple "
                "of the expected heap group size");
        }
        _nheap_groups_per_block = data.used_bytes() / heap_group_size;
        _nsamples_per_dada_block = _nheap_groups_per_block * FBFUSE_NSAMPLES_PER_HEAP;
        if (_nsamples_per_dada_block % FBFUSE_CB_NSAMPLES_PER_BLOCK != 0)
        {
            throw std::runtime_error("Input DADA buffer does not contain an integer "
                "multiple of the required number of samples per device block");
        }
        _taftp_db.resize(heap_group_size / sizeof(char2), 0);

        //
        // Output buffer checks:
        //
        std::size_t expected_cb_size = (FBFUSE_CB_NBEAMS * _nsamples_per_dada_block
            / FBFUSE_CB_TSCRUNCH * FBFUSE_NCHANS / FBFUSE_CB_FSCRUNCH) * sizeof(char);
        if (_cb_writer.data_buffer_size() != expected_cb_size)
        {
            throw std::runtime_error(
                std::string("Expected coherent beam output buffer to have a size of ")
                + std::to_string(expected_cb_size)
                + " bytes, but it instead had a size of "
                + std::to_string(_cb_writer.data_buffer_size())
                + " bytes");
        }
        _tbftf_db.resize(expected_cb_size, 0);

        std::size_t expected_ib_size = (FBFUSE_IB_NBEAMS * _nsamples_per_dada_block
            / FBFUSE_IB_TSCRUNCH * FBFUSE_NCHANS / FBFUSE_IB_FSCRUNCH) * sizeof(char);
        if (_ib_writer.data_buffer_size() != expected_ib_size)
        {
            throw std::runtime_error(
                std::string("Expected incoherent beam output buffer to have a size of ")
                + std::to_string(expected_ib_size)
                + " bytes, but it instead had a size of "
                + std::to_string(_ib_writer.data_buffer_size())
                + " bytes");
        }
        _tftf_db.resize(expected_ib_size, 0);
    }

    // We first need to synchronize the h2d copy stream to ensure that
    // last host to device copy has completed successfully. When this is
    // done we are free to call swap on the double buffer without affecting
    // any previous copy.
    CUDA_SAFE_CALL(hipStreamSynchronize(_h2d_copy_stream));
    _taftp_db.swap();
    CUDA_SAFE_CALL(hipMemcpyAsync(static_cast<void*>(_taftp_db.a()),
        static_cast<void*>(data.ptr()), data.used_bytes(),
        hipMemcpyHostToDevice, _h2d_copy_stream));


    // If we are on the first call we can exit here as there is no
    // data on the GPU yet to process.
    if (_call_count == 1)
    {
        return false;
    }

    // Here we block on the processing stream before swapping
    // the processing buffers
    CUDA_SAFE_CALL(hipStreamSynchronize(_processing_stream));
    _tbftf_db.swap();
    _tftf_db.swap();
    process(_taftp_db.b(), _tbftf_db.a(), _tftf_db.a());

    // If we are on the second call we can exit here as there is not data
    // that has completed processing at this stage.
    if (_call_count == 2)
    {
        return false;
    }

    CUDA_SAFE_CALL(hipStreamSynchronize(_d2h_copy_stream));
    // Only want to perform one copy per data block here, not d2h then h2h.
    // For this reason we need access to two DadaWriteClient instances in
    // this class.
    if (_call_count > 3)
    {
        // If the call count is >3 then we have already performed the first
        // output copy and we need to release and get the next dada blocks
        _cb_data_stream.release();
        _ib_data_stream.release();
    }
    auto& cb_block = _cb_data_stream.next();
    auto& ib_block = _ib_data_stream.next();
    CUDA_SAFE_CALL(hipMemcpyAsync(static_cast<void*>(cb_block.ptr()),
        static_cast<void*>(_tbftf_db.b()), cb_block.total_bytes(),
        hipMemcpyDeviceToHost, _d2h_copy_stream));
    CUDA_SAFE_CALL(hipMemcpyAsync(static_cast<void*>(ib_block.ptr()),
        static_cast<void*>(_tftf_db.b()), ib_block.total_bytes(),
        hipMemcpyDeviceToHost, _d2h_copy_stream));

    return false;
}

} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

unsigned long int strtoul (const char* str, char** endptr, int base);