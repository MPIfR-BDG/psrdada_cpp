#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/Pipeline.cuh"
#include "psrdada_cpp/multilog.hpp"
#include "psrdada_cpp/raw_bytes.hpp"
#include "psrdada_cpp/dada_write_client.hpp"
#include "psrdada_cpp/dada_client_base.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/cli_utils.hpp"

#include "boost/program_options.hpp"

#include <sys/types.h>
#include <iostream>
#include <string>
#include <sstream>
#include <ios>
#include <algorithm>

using namespace psrdada_cpp;

namespace
{
  const size_t ERROR_IN_COMMAND_LINE = 1;
  const size_t SUCCESS = 0;
  const size_t ERROR_UNHANDLED_EXCEPTION = 2;
} // namespace

int main(int argc, char** argv)
{
    try
    {
        meerkat::fbfuse::PipelineConfig config;

        /** Define and parse the program options
        */
        namespace po = boost::program_options;
        po::options_description desc("Options");
        desc.add_options()
        ("help,h", "Print help messages")
        ("input_key", po::value<std::string>()
            ->required()
            ->notifier([&config](std::string key)
                {
                    config.input_dada_key(string_to_key(key));
                }),
           "The shared memory key (hex string) for the dada buffer containing input data (in TAFTP order)")
        ("cb_key", po::value<std::string>()
            ->required()
            ->notifier([&config](std::string key)
                {
                    config.cb_dada_key(string_to_key(key));
                }),
           "The shared memory key (hex string) for the output coherent beam dada buffer")
        ("ib_key", po::value<std::string>()
            ->required()
            ->notifier([&config](std::string key)
                {
                    config.ib_dada_key(string_to_key(key));
                }),
           "The shared memory key (hex string) for the output incoherent beam dada buffer")
        ("delay_key_root", po::value<std::string>()
            ->required()
            ->notifier([&config](std::string key)
                {
                    config.delay_buffer_shm(key);
                    config.delay_buffer_mutex(key + "_mutex");
                    config.delay_buffer_sem(key + "_count");
                }),
           "The root of the POSIX key for the delay buffer shared memory and semaphores")
        ("delay_engine_socket", po::value<std::string>()
            ->notifier([&config](std::string addr)
                {
                    config.delay_engine_socket(addr);
                }),
           "The address for the control socket of the delay engine. Setting this parameter"
           " enables 'offline' processing mode where explicit requests are made to the delay"
           " engine for new delay models. This reduces performance compared with the free-running"
           " 'online' mode (which is the default)")
        ("bandwidth", po::value<float>()
            ->required()
            ->notifier([&config](float value)
                {
                    config.bandwidth(value);
                }),
           "The bandwidth (Hz) of the subband this instance will process")
        ("cfreq", po::value<float>()
            ->required()
            ->notifier([&config](float value)
                {
                    config.centre_frequency(value);
                }),
           "The centre frequency (Hz) of the subband this instance will process")
        ("input_level", po::value<float>()
            ->notifier([&config](float value)
                {
                    config.input_level(value);
                }),
           "The standard deviation of the input data (used for calculating scaling factors)")
        ("output_level", po::value<float>()
            ->notifier([&config](float value)
                {
                    config.output_level(value);
                }),
           "The desired standard deviation of the output data (used for calculating scaling factors)")
        ("log_level", po::value<std::string>()
            ->default_value("info")
            ->notifier([](std::string level)
                {
                    set_log_level(level);
                }),
            "The logging level to use (debug, info, warning, error)");

        po::variables_map vm;
        try
        {
            po::store(po::parse_command_line(argc, argv, desc), vm);
            if ( vm.count("help")  )
            {
                std::cout << "fbfuse -- The fbfuse beamformer implementations" << std::endl
                << desc << std::endl;
                return SUCCESS;
            }
            po::notify(vm);
        }
        catch(po::error& e)
        {
            std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
            std::cerr << desc << std::endl;
            return ERROR_IN_COMMAND_LINE;
        }

        //

        /**
         * All the application code goes here
         */
        MultiLog log("fbfuse");
        DadaWriteClient cb_writer(config.cb_dada_key(), log);
        DadaWriteClient ib_writer(config.ib_dada_key(), log);
        // Need to setup a base client to retrive the block size
        // for the beamformer and register the host memory.
        DadaClientBase client(config.input_dada_key(), log);
        client.cuda_register_memory();
        cb_writer.cuda_register_memory();
        ib_writer.cuda_register_memory();
        meerkat::fbfuse::Pipeline pipeline(config, cb_writer, ib_writer,
            client.data_buffer_size());
        DadaInputStream<decltype(pipeline)> stream(config.input_dada_key(), log, pipeline);
        stream.start();
        /**
         * End of application code
         */
    }
    catch(std::exception& e)
    {
        std::cerr << "Unhandled Exception reached the top of main: "
        << e.what() << ", application will now exit" << std::endl;
        return ERROR_UNHANDLED_EXCEPTION;
    }
    return SUCCESS;

}
