#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/WeightsManager.cuh"
#include "psrdada_cpp/meerkat/fbfuse/PipelineConfig.hpp"
#include "psrdada_cpp/meerkat/fbfuse/DelayManager.cuh"
#include "psrdada_cpp/cuda_utils.hpp"
#include <thrust/device_vector.h>

#define TWOPI 6.283185307179586

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace kernels {

__global__
void generate_weights_k(
    float2 const * __restrict__ delay_models,
    char2 * __restrict__ weights,
    double const * __restrict__ channel_frequencies,
    int nantennas,
    int nbeams,
    int nchans,
    double current_epoch,
    double delay_epoch,
    double tstep,
    int ntsteps)
{

    //for each loaded delay poly we can produce multiple epochs for one antenna, one beam, all frequencies and both pols
    //Different blocks should handle different beams (as antennas are on the inner dimension of the output product)

    //Basics of this kernel:
    //
    // gridDim.x is used for beams (there is a loop if you want to limit the grid size)
    // gridDim.y is used for channels (there is a loop if you want to limit the grid size)
    // blockDim.x is used for antennas (there is a loop if you want to limit the grid size)
    //
    // Time steps are handled in a the inner loop. As antennas are on the inner dimension of
    // both the input and the output array, all reads and writes should be coalesced.
    const int weights_per_beam = nantennas;
    const int weights_per_channel = weights_per_beam * nbeams;
    const int weights_per_time_step = weights_per_channel * nchans;

    double2 weight;
    char2 compressed_weight;
    //This isn't really needed as there will never be more than 64 antennas
    //However this makes this fucntion more flexible with smaller blocks

    for (int chan_idx = blockIdx.y; chan_idx < nchans; chan_idx += gridDim.y)
    {
        double frequency = channel_frequencies[chan_idx];
        int chan_offset = chan_idx * weights_per_channel; // correct

        for (int beam_idx = blockIdx.x; beam_idx < nbeams; beam_idx += gridDim.x)
        {
            int beam_offset = chan_offset + beam_idx * weights_per_beam; // correct

            for (int antenna_idx = threadIdx.x; antenna_idx < nantennas; antenna_idx+=blockDim.x)
            {
                float2 delay_model = delay_models[beam_idx * nantennas + antenna_idx]; // correct
                double delay_offset = (double) delay_model.x;
                double delay_rate = (double) delay_model.y;
                int antenna_offset = beam_offset + antenna_idx;
                for (int time_idx = threadIdx.y; time_idx < ntsteps; time_idx+=blockDim.y)
                {
                    //Calculates epoch offset
                    double t = (current_epoch - delay_epoch) + time_idx * tstep;
                    double phase = (t * delay_rate + delay_offset) * frequency;
                    //This is possible as the magnitude of the weight is 1
                    //If we ever have to implement scalar weightings, this
                    //must change.
                    sincos(TWOPI * phase, &weight.y, &weight.x);
                    compressed_weight.x = (char) __double2int_rn(weight.x * 127.0);
                    compressed_weight.y = (char) __double2int_rn(-1.0 * weight.y * 127.0);
                    int output_idx = time_idx * weights_per_time_step + antenna_offset;
                    weights[output_idx] = compressed_weight;
                }
            }
        }
    }
}

} //namespace kernels

WeightsManager::WeightsManager(PipelineConfig const& config,
    hipStream_t stream)
    : _config(config)
    , _stream(stream)
{
    std::size_t nbeams = _config.cb_nbeams();
    std::size_t nantennas = _config.cb_nantennas();
    BOOST_LOG_TRIVIAL(debug) << "Constructing WeightsManager instance to hold weights for "
    << nbeams << " beams and " << nantennas << " antennas";
    _weights.resize(nbeams * nantennas * _config.nchans());
    // This should be an implicit copy to the device
    BOOST_LOG_TRIVIAL(debug) << "Copying channel frequencies to the GPU";
    _channel_frequencies = _config.channel_frequencies();
}

WeightsManager::~WeightsManager()
{
}

WeightsManager::WeightsVectorType const& WeightsManager::weights(
    DelayVectorType const& delays, TimeType current_epoch, TimeType delay_epoch)
{
    // First we retrieve new delays if there are any.
    BOOST_LOG_TRIVIAL(debug) << "Requesting weights: current epoch = " << current_epoch
                             << ", delay mode epoch = " << delay_epoch << " (difference = "
                             << (current_epoch - delay_epoch) << ")";
    DelayManager::DelayType const* delays_ptr = thrust::raw_pointer_cast(delays.data());
    WeightsType* weights_ptr = thrust::raw_pointer_cast(_weights.data());
    FreqType const* frequencies_ptr = thrust::raw_pointer_cast(_channel_frequencies.data());
    dim3 grid(_config.cb_nbeams(),
        _channel_frequencies.size(), 1);
    dim3 block(32, 32, 1);
    BOOST_LOG_TRIVIAL(debug) << "Launching weights generation kernel";
    kernels::generate_weights_k<<< grid, block, 0, _stream >>>(delays_ptr,
        weights_ptr, frequencies_ptr,
        _config.cb_nantennas(),
        _config.cb_nbeams(),
        _channel_frequencies.size(),
        current_epoch, delay_epoch,
        0.0, 1);
    CUDA_ERROR_CHECK(hipStreamSynchronize(_stream));
    BOOST_LOG_TRIVIAL(debug) << "Weights successfully generated";
    return _weights;
}

} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

