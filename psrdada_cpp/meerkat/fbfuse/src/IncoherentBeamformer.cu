#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/IncoherentBeamformer.cuh"
#include "psrdada_cpp/cuda_utils.hpp"
#include <cassert>

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace kernels {

__global__
void icbf_taftp_general_k(
    char4 const* __restrict__ taftp_voltages,
    int8_t* __restrict__ tf_powers,
    float output_scale,
    float output_offset,
    int ntimestamps)
{

    // What are the dimensions...
    // blockDim.x doesn't matter
    // blockDim.y == nchans / fscrunch
    // blockDim.z unused
    // gridDim.x == up to number of timestamps
    // gridDim.y is unused
    // gridDim.z is unused

    static_assert(FBFUSE_NSAMPLES_PER_HEAP % FBFUSE_IB_TSCRUNCH == 0,
        "tscrunch must divide 256");
    static_assert(FBFUSE_NCHANS % FBFUSE_IB_FSCRUNCH == 0,
        "Fscrunch must divide nchannels");

    const int output_size = FBFUSE_NSAMPLES_PER_HEAP/FBFUSE_IB_TSCRUNCH * FBFUSE_NCHANS/FBFUSE_IB_FSCRUNCH;
    volatile __shared__ float accumulation_buffer[FBFUSE_NCHANS/FBFUSE_IB_FSCRUNCH][FBFUSE_NSAMPLES_PER_HEAP];
    volatile __shared__ int8_t output_staging[FBFUSE_NSAMPLES_PER_HEAP/FBFUSE_IB_TSCRUNCH][FBFUSE_NCHANS/FBFUSE_IB_FSCRUNCH];

    //TAFTP
    const int tp = FBFUSE_NSAMPLES_PER_HEAP;
    const int ftp = FBFUSE_NCHANS * tp;
    const int aftp = FBFUSE_IB_NANTENNAS * ftp;
    const int channel_offset = blockIdx.y * FBFUSE_NCHANS;

    for (int timestamp_idx = blockIdx.x; timestamp_idx < ntimestamps; timestamp_idx += gridDim.x)
    {
        for (int sample_idx = threadIdx.x; sample_idx < FBFUSE_NSAMPLES_PER_HEAP; sample_idx += blockDim.x)
        {
            float xx = 0.0f, yy = 0.0f, zz = 0.0f, ww = 0.0f;

            // Must start with the right number of threads in the y dimension
            // blockDim.y = nchans / fscrunch
            for (int channel_idx = FBFUSE_IB_FSCRUNCH * threadIdx.y + channel_offset;
                channel_idx < min(channel_idx + FBFUSE_IB_FSCRUNCH + channel_offset, FBFUSE_NCHANS);
                ++channel_idx)
            {
                for (int antenna_idx = 0; antenna_idx < FBFUSE_IB_NANTENNAS; ++antenna_idx)
                {
                    int input_index = timestamp_idx * aftp + antenna_idx * ftp + channel_idx * tp + sample_idx;
                    char4 ant = taftp_voltages[input_index];
                    xx += ((float) ant.x) * ant.x;
                    yy += ((float) ant.y) * ant.y;
                    zz += ((float) ant.z) * ant.z;
                    ww += ((float) ant.w) * ant.w;
                }
            }
            accumulation_buffer[threadIdx.y][sample_idx] = (xx + yy + zz + ww);
        }
        __threadfence_block();
        if (threadIdx.x < FBFUSE_NSAMPLES_PER_HEAP/FBFUSE_IB_TSCRUNCH)
        {
            float val = 0.0f;
            for (int sample_idx = threadIdx.x * FBFUSE_IB_TSCRUNCH; sample_idx < (threadIdx.x + 1) * FBFUSE_IB_TSCRUNCH; ++sample_idx)
            {
                val += accumulation_buffer[threadIdx.y][sample_idx];
            }
            output_staging[threadIdx.x][threadIdx.y] = (int8_t)((val - output_offset)/output_scale);
        }
        __threadfence_block();
        for (int idx = threadIdx.x; idx < output_size; idx += gridDim.x)
        {
            tf_powers[idx * gridDim.y + threadIdx.y] = output_staging[idx][threadIdx.y];
        }
    }
}
} //namespace kernels


IncoherentBeamformer::IncoherentBeamformer(PipelineConfig const& config)
    : _config(config)
    , _size_per_aftp_block(0)
{
    BOOST_LOG_TRIVIAL(debug) << "Constructing IncoherentBeamformer instance";
    _size_per_aftp_block = (_config.npol() * _config.ib_nantennas()
        * _config.nchans() * _config.nsamples_per_heap());
    BOOST_LOG_TRIVIAL(debug) << "Size per AFTP block: " << _size_per_aftp_block;
}

IncoherentBeamformer::~IncoherentBeamformer()
{

}

void IncoherentBeamformer::beamform(VoltageVectorType const& input,
    PowerVectorType& output,
    hipStream_t stream)
{
    // First work out nsamples and resize output if not done already
    BOOST_LOG_TRIVIAL(debug) << "Executing coherent beamforming";
    assert(input.size() % _size_per_aftp_block == 0 /* Input is not a multiple of AFTP blocks*/);
    std::size_t ntimestamps = input.size() / _size_per_aftp_block;
    std::size_t output_size = (input.size() / _config.ib_nantennas()
	/ _config.npol() / _config.ib_tscrunch() / _config.ib_fscrunch());
    BOOST_LOG_TRIVIAL(debug) << "Resizing output buffer from "
    << output.size() << " to " << output_size
    << " elements";
    output.resize(output_size);
    int nthreads_y = _config.nchans() / _config.ib_fscrunch();
    int nthreads_x = 1024 / nthreads_y;
    dim3 block(nthreads_x, nthreads_y);
    dim3 grid(ntimestamps);
    char2 const* taftp_voltages_ptr = thrust::raw_pointer_cast(input.data());
    int8_t* tf_powers_ptr = thrust::raw_pointer_cast(output.data());
    BOOST_LOG_TRIVIAL(debug) << "Executing incoherent beamforming kernel";
    kernels::icbf_taftp_general_k<<<grid, block, 0, stream>>>(
        (char4 const*) taftp_voltages_ptr,
        tf_powers_ptr,
        _config.ib_power_scaling(),
        _config.ib_power_offset(),
        static_cast<int>(ntimestamps));
    CUDA_ERROR_CHECK(hipStreamSynchronize(stream));
    BOOST_LOG_TRIVIAL(debug) << "Incoherent beamforming kernel complete";
}

} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

