#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/IncoherentBeamformer.cuh"
#include "psrdada_cpp/cuda_utils.hpp"
#include <cassert>

#define FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK 16

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace kernels {

__global__
void icbf_taftp_general_k(
    char4 const* __restrict__ taftp_voltages,
    int8_t* __restrict__ tf_powers,
    float output_scale,
    float output_offset,
    int ntimestamps)
{

    // What are the dimensions...
    // blockDim.x doesn't matter
    // blockDim.y == nchans / fscrunch
    // blockDim.z unused
    // gridDim.x == up to number of timestamps
    // gridDim.y // Could use this for channel groups to keep shared memory size down
    // gridDim.z is unused

    static_assert(FBFUSE_NSAMPLES_PER_HEAP % FBFUSE_IB_TSCRUNCH == 0,
        "tscrunch must divide 256");
    static_assert(FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK % FBFUSE_IB_FSCRUNCH == 0,
        "Fscrunch must divide nchannels");

    const int nchans_output_total = FBFUSE_NCHANS / FBFUSE_IB_FSCRUNCH;
    const int nchans_output_block = FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK/FBFUSE_IB_FSCRUNCH;
    const int nsamps_output = FBFUSE_NSAMPLES_PER_HEAP/FBFUSE_IB_TSCRUNCH;
    volatile __shared__ float accumulation_buffer[nchans_output_block][FBFUSE_NSAMPLES_PER_HEAP];
    volatile __shared__ int8_t output_staging[nsamps_output][nchans_output_block];

    //TAFTP
    const int tp = FBFUSE_NSAMPLES_PER_HEAP;
    const int ftp = FBFUSE_NCHANS * tp;
    const int aftp = FBFUSE_IB_NANTENNAS * ftp;
    const int channel_offset = blockIdx.y * FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK;

    for (int timestamp_idx = blockIdx.x; timestamp_idx < ntimestamps; timestamp_idx += gridDim.x)
    {
        for (int sample_idx = threadIdx.x; sample_idx < FBFUSE_NSAMPLES_PER_HEAP; sample_idx += blockDim.x)
        {
            float xx = 0.0f, yy = 0.0f, zz = 0.0f, ww = 0.0f;

            // Must start with the right number of threads in the y dimension
            // blockDim.y = nchans / fscrunch
	    const int start_chan = FBFUSE_IB_FSCRUNCH * threadIdx.y + channel_offset;
            for (int channel_idx = start_chan; channel_idx < start_chan + FBFUSE_IB_FSCRUNCH; ++channel_idx)
            {
                for (int antenna_idx = 0; antenna_idx < FBFUSE_IB_NANTENNAS; ++antenna_idx)
                {
                    int input_index = timestamp_idx * aftp + antenna_idx * ftp + channel_idx * tp + sample_idx;
                    char4 ant = taftp_voltages[input_index];
                    xx += ((float) ant.x) * ant.x;
                    yy += ((float) ant.y) * ant.y;
                    zz += ((float) ant.z) * ant.z;
                    ww += ((float) ant.w) * ant.w;
                }
            }
            accumulation_buffer[threadIdx.y][sample_idx] = (xx + yy + zz + ww);
        }
        __threadfence_block();
        for (int output_sample_idx = threadIdx.x; output_sample_idx < nsamps_output; output_sample_idx += blockDim.x)
        {
            float val = 0.0f;
            for (int sample_idx = output_sample_idx * FBFUSE_IB_TSCRUNCH; sample_idx < (output_sample_idx + 1) * FBFUSE_IB_TSCRUNCH; ++sample_idx)
            {
                val += accumulation_buffer[threadIdx.y][sample_idx];
            }
            output_staging[output_sample_idx][threadIdx.y] = (int8_t)((val - output_offset)/output_scale);
        }
        __syncthreads();
	const int output_offset = timestamp_idx * nsamps_output * nchans_output_total;
        for (int idx = threadIdx.y; idx < nsamps_output; idx += blockDim.y)
        {
            for (int output_chan_idx = threadIdx.x; output_chan_idx < nchans_output; output_chan_idx += blockDim.x)
            {
                tf_powers[output_offset + idx * nchans_output_total + output_chan_idx + channel_offset] = output_staging[idx][output_chan_idx];
	    }
        }
    }
}
} //namespace kernels


IncoherentBeamformer::IncoherentBeamformer(PipelineConfig const& config)
    : _config(config)
    , _size_per_aftp_block(0)
{
    BOOST_LOG_TRIVIAL(debug) << "Constructing IncoherentBeamformer instance";
    _size_per_aftp_block = (_config.npol() * _config.ib_nantennas()
        * _config.nchans() * _config.nsamples_per_heap());
    BOOST_LOG_TRIVIAL(debug) << "Size per AFTP block: " << _size_per_aftp_block;
}

IncoherentBeamformer::~IncoherentBeamformer()
{

}

void IncoherentBeamformer::beamform(VoltageVectorType const& input,
    PowerVectorType& output,
    hipStream_t stream)
{
    // First work out nsamples and resize output if not done already
    BOOST_LOG_TRIVIAL(debug) << "Executing coherent beamforming";
    assert(input.size() % _size_per_aftp_block == 0 /* Input is not a multiple of AFTP blocks*/);
    std::size_t ntimestamps = input.size() / _size_per_aftp_block;
    std::size_t output_size = (input.size() / _config.ib_nantennas()
	/ _config.npol() / _config.ib_tscrunch() / _config.ib_fscrunch());
    BOOST_LOG_TRIVIAL(debug) << "Resizing output buffer from "
    << output.size() << " to " << output_size
    << " elements";
    output.resize(output_size);
    assert(FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK % _config.ib_fscrunch() == 0 /* IB fscrunch must divide the number of output channels per block*/);

    // The incoherent beamforming kernel can only handle 32 output channels per
    // block. As such we use the gridDim.y to handle blocks of 32 channels.
    int nchans_out_total = _config.nchans() / _config.ib_fscrunch();
    int nchans_groups = 1;
    if (nchans_out_total > FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK)
    {
        // Assumes that nchans is always a power of two.
        nchans_groups = nchans_out_total / FBFUSE_IB_MAX_NCHANS_OUT_PER_BLOCK;
    }
    BOOST_LOG_TRIVIAL(debug) << "IB kernel using " << nchans_groups << " channel groups";
    int nthreads_y = nchans_out_total / nchans_groups;
    int nthreads_x = 1024 / nthreads_y;
    dim3 block(nthreads_x, nthreads_y);
    dim3 grid(ntimestamps, nchans_groups);
    char2 const* taftp_voltages_ptr = thrust::raw_pointer_cast(input.data());
    int8_t* tf_powers_ptr = thrust::raw_pointer_cast(output.data());
    BOOST_LOG_TRIVIAL(debug) << "Executing incoherent beamforming kernel";
    kernels::icbf_taftp_general_k<<<grid, block, 0, stream>>>(
        (char4 const*) taftp_voltages_ptr,
        tf_powers_ptr,
        _config.ib_power_scaling(),
        _config.ib_power_offset(),
        static_cast<int>(ntimestamps));
    CUDA_ERROR_CHECK(hipStreamSynchronize(stream));
    BOOST_LOG_TRIVIAL(debug) << "Incoherent beamforming kernel complete";
}

} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

