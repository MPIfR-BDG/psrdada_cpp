#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/test/IncoherentBeamformerTester.cuh"
#include "psrdada_cpp/meerkat/fbfuse/fbfuse_constants.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include <random>
#include <cmath>
#include <complex>

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace test {

IncoherentBeamformerTester::IncoherentBeamformerTester()
    : ::testing::Test()
    , _stream(0)
{

}

IncoherentBeamformerTester::~IncoherentBeamformerTester()
{

}

void IncoherentBeamformerTester::SetUp()
{
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void IncoherentBeamformerTester::TearDown()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void IncoherentBeamformerTester::beamformer_c_reference(
    HostVoltageVectorType const& taftp_voltages,
    HostPowerVectorType& tf_powers,
    int nchannels,
    int tscrunch,
    int fscrunch,
    int ntimestamps,
    int nantennas,
    int npol,
    int nsamples_per_timestamp,
    float scale,
    float offset)
{
    const int tp = nsamples_per_timestamp * npol;
    const int ftp = nchannels * tp;
    const int aftp = nantennas * ftp;
    double power_sum = 0.0;
    double power_sq_sum = 0.0;
    std::size_t count = 0;

    for (int timestamp_idx = 0; timestamp_idx < ntimestamps; ++timestamp_idx)
    {
        for (int antenna_idx = 0; antenna_idx < nantennas; ++antenna_idx)
        {
            for (int subband_idx = 0; subband_idx < nchannels/fscrunch; ++subband_idx)
            {
                int subband_start = subband_idx * fscrunch;
                for (int subint_idx = 0; subint_idx < nsamples_per_timestamp/tscrunch; ++subint_idx)
                {
                    int subint_start = subint_idx * tscrunch;
                    float xx = 0.0f, yy = 0.0f;
                    for (int channel_idx = subband_start; channel_idx < subband_start + fscrunch;  ++channel_idx)
                    {
                        for (int sample_idx = subint_start; sample_idx < subint_start + tscrunch; ++sample_idx)
                        {
                            for (int pol_idx = 0; pol_idx < npol; ++pol_idx)
                            {
                                int input_index = timestamp_idx * aftp + antenna_idx * ftp + channel_idx * tp + sample_idx * npol + pol_idx;
                                char2 ant = taftp_voltages[input_index];
                                xx += ((float) ant.x) * ant.x;
                                yy += ((float) ant.y) * ant.y;
                            }
                        }
                    }
                    int time_idx = timestamp_idx * nsamples_per_timestamp/tscrunch + subint_idx;
                    int output_idx = time_idx * nchannels/fscrunch + subband_idx;
                    float power = (xx + yy);
                    power_sum += power;
                    power_sq_sum += power * power;
                    ++count;
                    tf_powers[output_idx] = (int8_t)(power - offset) / scale;
                }
            }
        }
    }
    double power_mean = power_sum / count;
    BOOST_LOG_TRIVIAL(debug) << "Average power level: " << power_mean;
    BOOST_LOG_TRIVIAL(debug) << "Power variance: " << power_sq_sum / count - power_mean * power_mean;
}

void IncoherentBeamformerTester::compare_against_host(
    DeviceVoltageVectorType const& taftp_voltages_gpu,
    DevicePowerVectorType& tf_powers_gpu,
    int ntimestamps)
{
    HostVoltageVectorType taftp_voltages_host = taftp_voltages_gpu;
    HostPowerVectorType tf_powers_cuda = tf_powers_gpu;
    HostPowerVectorType tf_powers_host(tf_powers_gpu.size());
    beamformer_c_reference(taftp_voltages_host,
        tf_powers_host,
        _config.nchans(),
        _config.ib_tscrunch(),
        _config.ib_fscrunch(),
        ntimestamps,
        _config.ib_nantennas(),
        _config.npol(),
        _config.nsamples_per_heap(),
        _config.ib_power_scaling(),
        _config.ib_power_offset());
    for (int ii = 0; ii < tf_powers_host.size(); ++ii)
    {
        std::cout << (int) tf_powers_cuda[ii] << ", ";
	std::cout << (int) tf_powers_host[ii] << ", " << (int) tf_powers_cuda[ii]
	    << ", (" << (int)tf_powers_host[ii] - (int)tf_powers_cuda[ii] << ");" << std::endl ;
        ASSERT_TRUE(std::abs(static_cast<int>(tf_powers_host[ii]) - tf_powers_cuda[ii]) <= 1);
    }
    //std::cout << "\n";
}

TEST_F(IncoherentBeamformerTester, ib_representative_noise_test)
{
    const float input_level = 32.0f;
    _config.input_level(input_level);
    _config.output_level(32.0f);
    std::default_random_engine generator;
    std::normal_distribution<float> normal_dist(0.0, input_level);
    IncoherentBeamformer incoherent_beamformer(_config);
    std::size_t ntimestamps = 1;
    std::size_t input_size = (ntimestamps * _config.ib_nantennas()
        * _config.nchans() * _config.nsamples_per_heap() * _config.npol());
    HostVoltageVectorType taftp_voltages_host(input_size);
    for (int ii = 0; ii < taftp_voltages_host.size(); ++ii)
    {
        taftp_voltages_host[ii].x = static_cast<int8_t>(std::lround(normal_dist(generator)));
        taftp_voltages_host[ii].y = static_cast<int8_t>(std::lround(normal_dist(generator)));
    }
    DeviceVoltageVectorType taftp_voltages_gpu = taftp_voltages_host;
    DevicePowerVectorType tf_powers_gpu;
    incoherent_beamformer.beamform(taftp_voltages_gpu, tf_powers_gpu, _stream);
    compare_against_host(taftp_voltages_gpu, tf_powers_gpu, ntimestamps);
}

} //namespace test
} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

