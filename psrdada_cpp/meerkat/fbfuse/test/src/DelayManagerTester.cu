#include "psrdada_cpp/meerkat/fbfuse/test/DelayManagerTester.cuh"
#include "psrdada_cpp/meerkat/fbfuse/fbfuse_constants.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "thrust/host_vector.hpp"
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <errno.h>
#include <cstring>
#include <sys/mman.h>

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace test {

DelayManagerTester::DelayManagerTester()
    : ::testing::Test()
    , _shm_fd(0)
    , _shm_ptr(nullptr)
    , _sem_id(nullptr)
    , _mutex_id(nullptr)
    , _delay_model(nullptr)
    , _stream(0)
{
    _config.delay_buffer_shm("test_delay_buffer_shm");
    _config.delay_buffer_sem("test_delay_buffer_sem");
    _config.delay_buffer_mutex("test_delay_buffer_mutex");
}

DelayManagerTester::~DelayManagerTester()
{

}

void DelayManagerTester::SetUp()
{
    _shm_fd = shm_open(_config.delay_buffer_shm().c_str(), O_CREAT | O_RDWR, 0666);
    if (_shm_fd == -1)
    {
        FAIL() << "Failed to open shared memory named "
        << _config.delay_buffer_shm() << " with error: "
        << std::strerror(errno);
    }
    if (ftruncate(_shm_fd, sizeof(DelayModel)) == -1)
    {
        FAIL() << "Failed to ftruncate shared memory named "
        << _config.delay_buffer_shm() << " with error: "
        << std::strerror(errno);
    }
    _shm_ptr = mmap(0, sizeof(DelayModel), PROT_WRITE, MAP_SHARED, _shm_fd, 0);
    if (_shm_ptr == NULL)
    {
        FAIL() << "Failed to mmap shared memory named "
        << _config.delay_buffer_shm() << " with error: "
        << std::strerror(errno);
    }
    _delay_model = static_cast<DelayModel*>(_shm_ptr);
    _sem_id = sem_open(_config.delay_buffer_sem().c_str(), O_CREAT, 0666, 0);
    if (_sem_id == SEM_FAILED)
    {
        FAIL() << "Failed to open delay buffer semaphore "
        << _config.delay_buffer_sem() << " with error: "
        << std::strerror(errno);
    }
    _mutex_id = sem_open(_config.delay_buffer_mutex().c_str(), O_CREAT, 0666, 0);
    if (_mutex_id == SEM_FAILED)
    {
        FAIL() << "Failed to open delay buffer mutex "
        << _config.delay_buffer_mutex() << " with error: "
        << std::strerror(errno);
    }
    // Here we post once so that the mutex has a value of 1
    // and can so be safely acquired by the DelayManger
    sem_post(_mutex_id);
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void DelayManagerTester::TearDown()
{
    if (munmap(_shm_ptr, sizeof(DelayModel)) == -1)
    {
        FAIL() << "Failed to unmap shared memory "
        << _config.delay_buffer_shm() << " with error: "
        << std::strerror(errno);
    }

    if (close(_shm_fd) == -1)
    {
        FAIL() << "Failed to close shared memory file descriptor "
        << _shm_fd << " with error: "
        << std::strerror(errno);
    }

    if (shm_unlink(_config.delay_buffer_shm().c_str()) == -1)
    {
        FAIL() << "Failed to unlink shared memory "
        << _config.delay_buffer_shm() << " with error: "
        << std::strerror(errno);
    }

    if (sem_close(_sem_id) == -1)
    {
        FAIL() << "Failed to close semaphore "
        << _config.delay_buffer_sem() << " with error: "
        << std::strerror(errno);
    }

    if (sem_close(_mutex_id) == -1)
    {
        FAIL() << "Failed to close mutex "
        << _config.delay_buffer_mutex() << " with error: "
        << std::strerror(errno);
    }
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void DelayManagerTester::compare_against_host(DelayManager::DelayVectorType const& delays)
{
    // Implicit sync copy back to host
    thrust::host_vector<DelayManager::DelayType> host_delays = delays;
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    for (int ii=0; ii < FBFUSE_CB_NBEAMS * FBFUSE_CB_NANTENNAS; ++ii)
    {
        ASSERT_EQ(_delay_model->delays[ii].x, _dhost_delays[ii].x);
        ASSERT_EQ(_delay_model->delays[ii].y, _dhost_delays[ii].y);
    }
}

TEST_F(DelayManagerTester, test_updates)
{
    DelayManager delay_manager(_config, _stream);
    sem_post(_sem_id);
    auto const& delay_vector = delay_manager.delays();
    compare_against_host(delay_vector);
    std::memset(static_cast<void*>(_delay_model->delays), 1, sizeof(_delay_model->delays));
    sem_post(_sem_id);
    compare_against_host(delay_vector);
}

TEST_F(DelayManagerTester, test_bad_keys)
{
    PipelineConfig config;
    config.delay_buffer_shm("bad_test_delay_buffer_shm");
    config.delay_buffer_sem("bda_test_delay_buffer_sem");
    config.delay_buffer_mutex("bad_test_delay_buffer_mutex");
    ASSERT_THROW(DelayManager(config, _stream));
}

} //namespace test
} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

