#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/tools/feng_to_bandpass.cuh"
#include "psrdada_cpp/meerkat/constants.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "hip/hip_complex.h"
#include <fstream>
#include <iomanip>

namespace psrdada_cpp {
namespace meerkat {
namespace tools {
namespace kernels {

    __global__ void feng_heaps_to_bandpass(
        char2* __restrict__ in, float* __restrict__ out,
        int nchans, int nants,
        int ntimestamps)
    {
        __shared__ float time_ar[MEERKAT_FENG_NSAMPS_PER_HEAP];

        float total_sum = 0.0f;
        int antenna_idx = blockIdx.x;
        int channel_idx = blockIdx.y;
        int poln_idx = blockIdx.z;
        for (int heap_idx=0; heap_idx<ntimestamps; ++heap_idx)
        {

            int offset = MEERKAT_FENG_NSAMPS_PER_HEAP * MEERKAT_FENG_NPOL_PER_HEAP * (
                nchans * (heap_idx * nants + antenna_idx)
                + channel_idx);

            char2 tmp = in[offset + threadIdx.x*MEERKAT_FENG_NPOL_PER_HEAP + poln_idx];
            hipComplex voltage = make_hipComplex(tmp.x,tmp.y);
            float val = voltage.x * voltage.x + voltage.y * voltage.y;
            time_ar[threadIdx.x] = val;
            __syncthreads();

            for (int ii=0; ii<8; ++ii)
            {
                if ((threadIdx.x + (1<<ii)) < MEERKAT_FENG_NSAMPS_PER_HEAP)
                {
                    val += time_ar[threadIdx.x + (1<<ii)];
                }
                __syncthreads();
                time_ar[threadIdx.x] = val;
                __syncthreads();
            }
            total_sum += val;
        }
        if (threadIdx.x == 0)
        {
            out[antenna_idx * nchans * MEERKAT_FENG_NPOL_PER_HEAP + antenna_idx * nchans * poln_idx + channel_idx] = total_sum;
        }
    }
}

}
}
}