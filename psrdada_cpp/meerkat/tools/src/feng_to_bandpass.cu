#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/tools/feng_to_bandpass.cuh"
#include "psrdada_cpp/meerkat/constants.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "hip/hip_complex.h"
#include <fstream>
#include <iomanip>

namespace psrdada_cpp {
namespace meerkat {
namespace tools {
namespace kernels {

    __global__ void feng_heaps_to_bandpass(
        char2* __restrict__ in, float* __restrict__ out,
        int nchans, int nants,
        int ntimestamps)
    {
        __shared__ float time_pol_ar[MEERKAT_FENG_NPOL_PER_HEAP*MEERKAT_FENG_NSAMPS_PER_HEAP];

        float total_sum = 0.0f;
        int antenna_idx = blockIdx.x;
        int channel_idx = blockIdx.y;
        for (int heap_idx=0; heap_idx<ntimestamps; ++heap_idx)
        {
            int offset = MEERKAT_FENG_NSAMPS_PER_HEAP * MEERKAT_FENG_NPOL_PER_HEAP * (
                nchans * (heap_idx * nants + antenna_idx)
                + channel_idx);

            char2 tmp = in[offset + threadIdx.x];
            hipComplex voltage = make_hipComplex(tmp.x,tmp.y);
            float val = voltage.x * voltage.x + voltage.y * voltage.y;
            time_pol_ar[threadIdx.x] = val;
            __syncthreads();

            for (int ii=1; ii<9; ++ii)
            {
                if ((threadIdx.x + (1<<ii)) < (MEERKAT_FENG_NSAMPS_PER_HEAP*MEERKAT_FENG_NPOL_PER_HEAP))
                {
                    val += time_pol_ar[threadIdx.x + (1<<ii)];
                }
                __syncthreads();
                time_pol_ar[threadIdx.x] = val;
                __syncthreads();
            }
            total_sum += val;
        }
        if (threadIdx.x == 0 || threadIdx.x == 1)
        {
            out[antenna_idx * nchans * MEERKAT_FENG_NPOL_PER_HEAP
                + antenna_idx * nchans * threadIdx.x
                + channel_idx] = total_sum;
        }
    }
}

}
}
}