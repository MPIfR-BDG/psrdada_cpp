#include "hip/hip_runtime.h"
#include "psrdada_cpp/multilog.hpp"
#include "psrdada_cpp/cli_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/dada_output_stream.hpp"
#include "psrdada_cpp/simple_file_writer.hpp"
#include "psrdada_cpp/dada_read_client.hpp"
#include "psrdada_cpp/meerkat/tools/feng_to_dada.cuh"
#include "psrdada_cpp/meerkat/tools/feng_header_inserter.cuh"
#include "boost/program_options.hpp"
#include <time.h>
#include <ctime>

using namespace psrdada_cpp;

namespace
{
  const size_t ERROR_IN_COMMAND_LINE = 1;
  const size_t SUCCESS = 0;
  const size_t ERROR_UNHANDLED_EXCEPTION = 2;
} // namespace


int main(int argc, char** argv)
{
    try
    {
        key_t input_key, output_key;
        std::size_t nchannels;
        double cfreq;
        double bw;
        double sync_epoch;
        std::string obs_id;
        /** Define and parse the program options
        */
        namespace po = boost::program_options;
        po::options_description desc("Options");
        desc.add_options()

        ("help,h", "Print help messages")

        ("input_key,i", po::value<std::string>()
            ->default_value("dada")
            ->notifier([&input_key](std::string in)
                {
                    input_key = string_to_key(in);
                }),
           "The shared memory key for the dada buffer to connect to (hex string)")

        ("output_key,o", po::value<std::string>()
            ->default_value("caca")
            ->notifier([&output_key](std::string in)
                {
                    output_key = string_to_key(in);
                }),
           "The shared memory key for the dada buffer to connect to (hex string)")

        ("nchannels,c", po::value<std::size_t>(&nchannels)->required(),
            "The number of frequency channels in the stream")

        ("cfreq,f", po::value<double>(&cfreq)->required(),
            "The centre frequency of the band being processed")

        ("bw,b", po::value<double>(&bw)->required(),
            "The bandwidth of the band being processed")

        ("obs_id", po::value<std::string>(&obs_id)
            ->default_value("default_id"),
            "ID to insert into header")

        ("sync_epoch,s", po::value<double>(&sync_epoch)
            ->default_value(0.0),
            "The synchronisation epoch of the packetiser")

        ("log_level", po::value<std::string>()
            ->default_value("info")
            ->notifier([](std::string level)
                {
                    set_log_level(level);
                }),
            "The logging level to use (debug, info, warning, error)");

        po::variables_map vm;
        try
        {
            po::store(po::parse_command_line(argc, argv, desc), vm);
            if ( vm.count("help")  )
            {
                std::cout << "Feng2Dada -- read MeerKAT F-engine from DADA ring buffer and convert it to TFP order DADA data"
                << std::endl << desc << std::endl;
                return SUCCESS;
            }
            po::notify(vm);
        }
        catch(po::error& e)
        {
            std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
            std::cerr << desc << std::endl;
            return ERROR_IN_COMMAND_LINE;
        }
        /**
         * All the application code goes here
         */
        MultiLog log("feng2dada");
        DadaReadClient reader(input_key, log);
        DadaOutputStream ostream(output_key, log);
        meerkat::tools::FengToDada<decltype(ostream)> feng2dada(nchannels, ostream);
        meerkat::tools::FengHeaderInserter<decltype(feng2dada)> header_inserter(
            feng2dada, obs_id, cfreq, bw, nchannels, sync_epoch);
        DadaInputStream<decltype(header_inserter)> istream(reader, header_inserter);
        istream.start();
        /**
         * End of application code
         */
    }
    catch(std::exception& e)
    {
        std::cerr << "Unhandled Exception reached the top of main: "
        << e.what() << ", application will now exit" << std::endl;
        return ERROR_UNHANDLED_EXCEPTION;
    }
    return SUCCESS;

}