#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/tools/feng_to_bandpass.cuh"
#include "psrdada_cpp/meerkat/constants.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include "psrdada_cpp/common.hpp"
#include "hip/hip_complex.h"
#include <fstream>
#include <iomanip>

namespace psrdada_cpp {
namespace meerkat {
namespace tools {

    template <class HandlerType>
    FengToBandpass<HandlerType>::FengToBandpass(std::size_t nchans, std::size_t nants, HandlerType& handler)
    : _nchans(nchans)
    , _natnennas(nants)
    , _handler(handler)
    {
        //Will output data as an array of bandpasses for each
        //polarisation and antenna
        _output.resize(_nchans * _natnennas * MEERKAT_FENG_NPOL_PER_HEAP);
    }

    template <class HandlerType>
    FengToBandpass<HandlerType>::~FengToBandpass()
    {
    }

    template <class HandlerType>
    void FengToBandpass<HandlerType>::init(RawBytes& block)
    {
        _handler.init(block);
    }

    template <class HandlerType>
    bool FengToBandpass<HandlerType>::operator()(RawBytes& block)
    {
        std::size_t used = block.used_bytes();
        std::size_t nbytes_per_timestamp =
            _natnennas * _nchans * MEERKAT_FENG_NSAMPS_PER_HEAP
            * MEERKAT_FENG_NPOL_PER_HEAP * sizeof(char2);
        if (used%nbytes_per_timestamp != 0)
        {
            throw std::runtime_error("Number of bytes in buffer is not an integer "
                "muliple of the number of bytes per timestamp");
        }
        std::size_t size = used/sizeof(char2);
        int ntimestamps = used/nbytes_per_timestamp;
        _input.resize(size);
        char2* d_input_ptr = thrust::raw_pointer_cast(_input.data());
        float* d_output_ptr = thrust::raw_pointer_cast(_output.data());
        CUDA_ERROR_CHECK(hipMemcpy(d_input_ptr, block.ptr(), used, hipMemcpyHostToDevice));
        dim3 grid(_natnennas,_nchans,MEERKAT_FENG_NPOL_PER_HEAP);
        kernels::feng_heaps_to_bandpass<<<grid,MEERKAT_FENG_NSAMPS_PER_HEAP>>>
            (d_input_ptr, d_output_ptr, _nchans, _natnennas, ntimestamps);
        CUDA_ERROR_CHECK(hipDeviceSynchronize());
        thrust::copy(_output.begin(),_output.end(),(float*) block.ptr());
        block.used_bytes(_output.size() * sizeof(float));
        _handler(block);
        return false;
    }
}
}
}